#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/runtime/utils/debugUtils.h"

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include <cfloat>

namespace
{
template <typename T>
__global__ void checkTensorNanKernel(T const* data, std::size_t size, int* foundNan)
{
    auto tidx = blockIdx.x * blockDim.x + threadIdx.x;

    int32_t found = 0;

    for (auto idx = tidx; idx < size; idx += blockDim.x * gridDim.x)
    {
        auto value = static_cast<float>(data[idx]);
        if (isnan(value))
        {
            found = 1;
            break;
        }
    }
    atomicCAS(foundNan, 0, found);
}
} // namespace

using namespace tensorrt_llm::runtime;
namespace tc = tensorrt_llm::common;

namespace tensorrt_llm::runtime::utils
{

template <typename T>
void invokeCheckTensorNanKernel(T const* data, std::size_t size, int* foundNan, hipStream_t stream)
{
    constexpr uint32_t kThreadsPerCta = 256;
    checkTensorNanKernel<<<tc::ceilDiv(size, kThreadsPerCta), kThreadsPerCta, 0, stream>>>(data, size, foundNan);
}

template void invokeCheckTensorNanKernel(float const* data, std::size_t size, int* foundNan, hipStream_t stream);
template void invokeCheckTensorNanKernel(half const* data, std::size_t size, int* foundNan, hipStream_t stream);
template void invokeCheckTensorNanKernel(
    __hip_bfloat16 const* data, std::size_t size, int* foundNan, hipStream_t stream);
template void invokeCheckTensorNanKernel(
    __hip_fp8_e4m3_fnuz const* data, std::size_t size, int* foundNan, hipStream_t stream);

template <typename T>
void printLogitsKeyInfo(ITensor const& tensor, std::string const& infoStr)
{
    auto const& shape = tensor.getShape();
    auto const volume = ITensor::volume(shape);

    BufferManager::ITensorPtr host{};
    T const* hostData;
    if (tensor.getMemoryType() == MemoryType::kGPU)
    {
        auto streamPtr = std::make_shared<CudaStream>();
        BufferManager manager{streamPtr};
        host = manager.copyFrom(tensor, MemoryType::kCPU);
        streamPtr->synchronize();
        hostData = bufferCast<T>(*host);
    }
    else
    {
        hostData = bufferCast<T>(tensor);
    }

    std::stringstream ss;
    ss << infoStr;
    ss << " Shape: " << shape;
    ss << "; Top 5: ";
    for (size_t ki = 0; ki < 5; ++ki)
    {
        ss << static_cast<float>(hostData[ki]) << ", ";
    }

    ss << " Last 5: ";
    for (size_t ki = volume - 6; ki < volume; ++ki)
    {
        ss << static_cast<float>(hostData[ki]) << ", ";
    }

    // find max, min, avg
    double mSum = 0.f;
    float mMax = -FLT_MAX;
    float mMin = FLT_MAX;

    for (size_t ki = 0; ki < volume; ++ki)
    {
        float value = static_cast<float>(hostData[ki]);
        mSum += value;
        if (value > mMax)
        {
            mMax = value;
        }
        if (value < mMin)
        {
            mMin = value;
        }
    }
    float mAvg = mSum / volume;

    ss << " avg: " << mAvg << ", min: " << mMin << ", max: " << mMax << std::endl;

    TLLM_LOG_TRACE(ss.str());
}

template void printLogitsKeyInfo<float>(ITensor const& tensor, std::string const& infoStr);
template void printLogitsKeyInfo<half>(ITensor const& tensor, std::string const& infoStr);
template void printLogitsKeyInfo<__hip_bfloat16>(ITensor const& tensor, std::string const& infoStr);
template void printLogitsKeyInfo<__hip_fp8_e4m3_fnuz>(ITensor const& tensor, std::string const& infoStr);

template <typename T>
bool tensorHasNan(ITensor const& tensor, BufferManager const& manager, std::string const& infoStr)
{
    printLogitsKeyInfo<T>(tensor, infoStr);
    auto foundNan = BufferManager::pinnedPool(ITensor::makeShape({1}), nvinfer1::DataType::kINT32);
    auto foundNanPtr = bufferCast<int32_t>(*foundNan);
    foundNanPtr[0] = 0;
    auto const size = tensor.getSize();
    invokeCheckTensorNanKernel(bufferCast<T>(tensor), size, foundNanPtr, manager.getStream().get());
    manager.getStream().synchronize();
    return static_cast<bool>(foundNanPtr[0]);
}

template bool tensorHasNan<float>(ITensor const& tensor, BufferManager const& manager, std::string const& infoStr);
template bool tensorHasNan<half>(ITensor const& tensor, BufferManager const& manager, std::string const& infoStr);
template bool tensorHasNan<__hip_bfloat16>(
    ITensor const& tensor, BufferManager const& manager, std::string const& infoStr);
template bool tensorHasNan<__hip_fp8_e4m3_fnuz>(
    ITensor const& tensor, BufferManager const& manager, std::string const& infoStr);

bool tensorHasNan(
    size_t M, size_t K, nvinfer1::DataType type, void const* data, hipStream_t stream, std::string const& infoStr)
{
    auto tensorView = ITensor::wrap(
        const_cast<void*>(data), type, ITensor::makeShape({static_cast<int32_t>(M), static_cast<int32_t>(K)}));
    auto manager = BufferManager(std::make_shared<CudaStream>(stream));
    if (type == nvinfer1::DataType::kFLOAT)
    {
        return tensorHasNan<float>(*tensorView, manager, infoStr);
    }
    else if (type == nvinfer1::DataType::kHALF)
    {
        return tensorHasNan<half>(*tensorView, manager, infoStr);
    }
    else if (type == nvinfer1::DataType::kBF16)
    {
        return tensorHasNan<__hip_bfloat16>(*tensorView, manager, infoStr);
    }
    else if (type == nvinfer1::DataType::kFP8)
    {
        return tensorHasNan<__hip_fp8_e4m3_fnuz>(*tensorView, manager, infoStr);
    }
    else
    {
        TLLM_THROW("Not supported type for Nan check");
    }
}

} // namespace tensorrt_llm::runtime::utils
