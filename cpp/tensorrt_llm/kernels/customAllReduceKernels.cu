#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "customAllReduceKernels.h"
#include "tensorrt_llm/common/cudaBf16Fallbacks.cuh"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/dataType.h"
#include "tensorrt_llm/common/envUtils.h"
#include <tuple>
#include <type_traits>

namespace tensorrt_llm::kernels
{

using tensorrt_llm::common::divUp;
using tensorrt_llm::common::roundUp;

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ void st_flag_release(uint32_t const& flag, uint32_t* flag_addr)
{
#if __CUDA_ARCH__ >= 700
    asm volatile("st.global.release.sys.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#else
    __threadfence_system();
    asm volatile("st.global.volatile.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ uint32_t ld_flag_acquire(uint32_t* flag_addr)
{
    uint32_t flag;
#if __CUDA_ARCH__ >= 700
    asm volatile("ld.global.acquire.sys.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#else
    asm volatile("ld.global.volatile.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#endif
    return flag;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// Type Converter that packs data format to 128 bits data type
//
using PackedFloat = union
{
    int4 packed;
    float unpacked[4];
};

using PackedHalf = union
{
    int4 packed;
    half2 unpacked[4];
};

template <typename T>
struct PackedOn16Bytes
{
};

template <>
struct PackedOn16Bytes<float>
{
    using Type = PackedFloat;
};

template <>
struct PackedOn16Bytes<half>
{
    using Type = PackedHalf;
};

#ifdef ENABLE_BF16
using PackedBFloat16 = union
{
    int4 packed;
    __hip_bfloat162 unpacked[4];
};

template <>
struct PackedOn16Bytes<__hip_bfloat16>
{
    using Type = PackedBFloat16;
};

#endif

// add two 128b data
template <typename T>
inline __device__ int4 add128b(T& a, T& b)
{
    T c;
    c.unpacked[0] = a.unpacked[0] + b.unpacked[0];
    c.unpacked[1] = a.unpacked[1] + b.unpacked[1];
    c.unpacked[2] = a.unpacked[2] + b.unpacked[2];
    c.unpacked[3] = a.unpacked[3] + b.unpacked[3];
    return c.packed;
}

__inline__ __device__ void multi_gpu_barrier(uint32_t** signals, uint32_t const flag, size_t const local_rank,
    size_t const world_size, int const tidx, int const bidx)
{
    // After this function, at least one block in each GPU has reached the barrier
    if (tidx < world_size)
    {
        // we can think of signals having the shape [world_size, world_size]
        // Dimension 0 is the "listening" dimension, dimension 1 is "emitting" dimension

        // Block 0 broadcasts its flag (local_rank on emitting dimension) to all receivers
        size_t offset = (flag % 2) ? world_size : 0;

        if (bidx == 0)
        {
            st_flag_release(flag, signals[tidx] + offset + local_rank);
        }

        // All blocks check that corresponding block 0 on other GPUs have set the flag
        // No deadlock because block #0 is always the first block started
        uint32_t* peer_barrier_d = signals[local_rank] + offset + tidx;
        while (ld_flag_acquire(peer_barrier_d) != flag)
        {
        }
    }

    __syncthreads();
}

__inline__ __device__ void block_barrier(uint32_t** signals, uint32_t const flag, size_t const local_rank,
    size_t const world_size, int const tidx, int const bidx, int const grid_size)
{
    // After this function, the block of id == bidx of each GPU has reached the barrier
    if (tidx < world_size)
    {
        // we can think of signals having the shape [world_size, 2, num_blocks, world_size]
        // (+ an offset on dim 2 to account for flags used in multi_gpu_barrier)
        // Dimension 0 is the "listening" dimension, dimension 3 is "emitting" dimension

        // Block broadcast its flag (local_rank on emitting dimension) to all receivers
        uint32_t flag_block_offset = world_size + bidx * world_size;

        if (flag % 2 == 1)
        {
            flag_block_offset += (grid_size + 1) * world_size;
        }

        st_flag_release(flag, signals[tidx] + flag_block_offset + local_rank);

        // Blocks check that corresponding blocks on other GPUs have also set the flag
        uint32_t* peer_barrier_d = signals[local_rank] + flag_block_offset + tidx;

        while (ld_flag_acquire(peer_barrier_d) != flag)
        {
        }
    }

    __syncthreads();
}

namespace reduce_fusion
{
namespace details
{
static constexpr int kBytesPerAccess = 16;
static constexpr int kWarpSize = 32;
static constexpr int kMaxCtaSize = 1024;
}; // namespace details

inline __device__ float warp_reduce_sum(float val)
{
    val += __shfl_xor_sync(~0, val, 16);
    val += __shfl_xor_sync(~0, val, 8);
    val += __shfl_xor_sync(~0, val, 4);
    val += __shfl_xor_sync(~0, val, 2);
    val += __shfl_xor_sync(~0, val, 1);
    return val;
}

inline __device__ float block_reduce_sum(float val)
{
    __shared__ float smem[details::kWarpSize];
    int lane_id = threadIdx.x % details::kWarpSize, warp_id = threadIdx.x / details::kWarpSize,
        warp_num = blockDim.x / details::kWarpSize;
    val = warp_reduce_sum(val);
    if (lane_id == 0)
    {
        smem[warp_id] = val;
    }
    __syncthreads();
    val = lane_id < warp_num ? smem[lane_id] : 0.f;
    val = warp_reduce_sum(val);
    return val;
}

template <typename T, typename PackedStruct>
inline __device__ float accumulate(float acc, PackedStruct& vec)
{
    static constexpr int kLoopNum = sizeof(PackedStruct) / sizeof(T);
#pragma unroll
    for (int i = 0; i < kLoopNum; ++i)
    {
        float v = static_cast<float>(reinterpret_cast<T*>(vec.unpacked)[i]);
        acc += v * v;
    }
    return acc;
}

template <typename T, bool Affine, typename PackedStruct>
inline __device__ int4 rms_norm(float denom, PackedStruct& vec, PackedStruct& weight)
{
    static constexpr int kLoopNum = sizeof(PackedStruct) / sizeof(T);
    PackedStruct ret;
#pragma unroll
    for (int i = 0; i < kLoopNum; ++i)
    {
        float v1 = static_cast<float>(reinterpret_cast<T*>(vec.unpacked)[i]);
        if constexpr (Affine)
        {
            float v2 = static_cast<float>(reinterpret_cast<T*>(weight.unpacked)[i]);
            reinterpret_cast<T*>(ret.unpacked)[i] = static_cast<T>(__fdividef(v1, denom) * v2);
        }
        else
        {
            reinterpret_cast<T*>(ret.unpacked)[i] = static_cast<T>(__fdividef(v1, denom));
        }
    }
    return ret.packed;
}

template <typename T, bool Bias = false, bool Residual = false, bool Affine = false, bool UseSmem = false>
__global__ void rms_norm_kernel(AllReduceParams params)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;

    extern __shared__ uint8_t smem_ptr[];
    T* smem = reinterpret_cast<T*>(smem_ptr);

    int bid = blockIdx.x, tid = threadIdx.x;

    T const* bias_buffer = reinterpret_cast<T const*>(params.fusion_params.bias_buffer);
    T const* residual_buffer = reinterpret_cast<T const*>(params.fusion_params.residual_buffer);
    T const* weight_buffer = reinterpret_cast<T const*>(params.fusion_params.weight_buffer);
    T* local_final_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);
    T* intermediate_buffer = reinterpret_cast<T*>(params.fusion_params.intermediate_buffer);

    int block_offset = bid * params.fusion_params.hidden_size;
    int thread_offset = tid * kPackedSize;

    if constexpr (Residual)
    {
        residual_buffer += block_offset;
    }
    local_final_output_buffer += block_offset;
    intermediate_buffer += block_offset;

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif

    PackedStruct inter_vec, weight_vec;
    float acc = 0.f;
    for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
    {
        inter_vec.packed = *reinterpret_cast<int4 const*>(intermediate_buffer + offset);
        if constexpr (Bias)
        {
            PackedStruct bias_vec;
            bias_vec.packed = *reinterpret_cast<int4 const*>(bias_buffer + offset);
            inter_vec.packed = add128b(inter_vec, bias_vec);
        }
        if constexpr (Residual)
        {
            PackedStruct residual_vec;
            residual_vec.packed = *reinterpret_cast<int4 const*>(residual_buffer + offset);
            inter_vec.packed = add128b(inter_vec, residual_vec);
            *reinterpret_cast<int4*>(intermediate_buffer + offset) = inter_vec.packed;
        }
        acc = accumulate<T>(acc, inter_vec);
        if constexpr (UseSmem)
        {
            *reinterpret_cast<int4*>(&smem[offset]) = inter_vec.packed;
        }
    }
    acc = block_reduce_sum(acc);
    float denom = __fsqrt_rn(__fdividef(acc, params.fusion_params.hidden_size) + params.fusion_params.eps);
    for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
    {
        if constexpr (UseSmem)
        {
            inter_vec.packed = *reinterpret_cast<int4 const*>(&smem[offset]);
        }
        if constexpr (Affine)
        {
            weight_vec.packed = *reinterpret_cast<int4 const*>(weight_buffer + offset);
        }
        inter_vec.packed = rms_norm<T, Affine>(denom, inter_vec, weight_vec);
        *reinterpret_cast<int4*>(&local_final_output_buffer[offset]) = inter_vec.packed;
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

template <typename T, bool Bias = false, bool Residual = false, bool Affine = false>
void rms_norm_kernel_launcher(AllReduceParams params, hipStream_t stream)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    TLLM_CHECK(params.fusion_params.hidden_size % kPackedSize == 0);
    int need_threads = params.fusion_params.hidden_size / kPackedSize;
    int cta_size;
    if (need_threads <= details::kMaxCtaSize)
    {
        cta_size = (need_threads + details::kWarpSize - 1) / details::kWarpSize * details::kWarpSize;
    }
    else
    {
        cta_size = details::kMaxCtaSize;
    }
    int cta_num = params.elts_total / params.fusion_params.hidden_size;
    int smem_size = 0;
    if (cta_size * details::kBytesPerAccess / sizeof(T) < params.fusion_params.hidden_size)
    {
        smem_size = params.fusion_params.hidden_size * sizeof(T);
        if (tensorrt_llm::common::getEnvEnablePDL())
        {
            TLLM_LOG_DEBUG("Enable PDL in rms_norm_kernel");
            hipLaunchConfig_t kernelConfig = {0};
            kernelConfig.gridDim = cta_num;
            kernelConfig.blockDim = cta_size;
            kernelConfig.dynamicSmemBytes = smem_size;
            kernelConfig.stream = stream;

            hipLaunchAttribute attribute[1];
            attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
            attribute[0].val.programmaticStreamSerializationAllowed = 1;
            kernelConfig.attrs = attribute;
            kernelConfig.numAttrs = 1;

            TLLM_CUDA_CHECK(
                cudaLaunchKernelEx(&kernelConfig, rms_norm_kernel<T, Bias, Residual, Affine, true>, params));
        }
        else
        {
            rms_norm_kernel<T, Bias, Residual, Affine, true><<<cta_num, cta_size, smem_size, stream>>>(params);
        }
    }
    else
    {
        if (tensorrt_llm::common::getEnvEnablePDL())
        {
            TLLM_LOG_DEBUG("Enable PDL in rms_norm_kernel");
            hipLaunchConfig_t kernelConfig = {0};
            kernelConfig.gridDim = cta_num;
            kernelConfig.blockDim = cta_size;
            kernelConfig.dynamicSmemBytes = smem_size;
            kernelConfig.stream = stream;

            hipLaunchAttribute attribute[1];
            attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
            attribute[0].val.programmaticStreamSerializationAllowed = 1;
            kernelConfig.attrs = attribute;
            kernelConfig.numAttrs = 1;

            TLLM_CUDA_CHECK(
                cudaLaunchKernelEx(&kernelConfig, rms_norm_kernel<T, Bias, Residual, Affine, false>, params));
        }
        else
        {
            rms_norm_kernel<T, Bias, Residual, Affine, false><<<cta_num, cta_size, smem_size, stream>>>(params);
        }
    }
}

template <typename T, int RanksPerNode, bool Bias = false, bool Affine = false, bool UseSmem = false>
static __global__ void __launch_bounds__(1024, 1) one_shot_all_reduce_norm_kernel(AllReduceParams params)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;

    extern __shared__ uint8_t smem_ptr[];
    T* smem = reinterpret_cast<T*>(smem_ptr);

    int bid = blockIdx.x, tid = threadIdx.x;
    int norm_num = params.elts_total / params.fusion_params.hidden_size;
    int norm_per_block = (norm_num + gridDim.x - 1) / gridDim.x;
    int norm_this_block = std::min(norm_per_block, norm_num - bid * norm_per_block);

    T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
    T const* bias_buffer = reinterpret_cast<T const*>(params.fusion_params.bias_buffer);
    T const* residual_buffer = reinterpret_cast<T const*>(params.fusion_params.residual_buffer);
    T const* weight_buffer = reinterpret_cast<T const*>(params.fusion_params.weight_buffer);
    T* local_shared_buffer = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[params.local_rank]);
    T* local_final_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);
    T* intermediate_buffer = reinterpret_cast<T*>(params.fusion_params.intermediate_buffer);

    int block_offset = bid * norm_per_block * params.fusion_params.hidden_size;
    int thread_offset = tid * kPackedSize;

    local_input_buffer += block_offset;
    residual_buffer += block_offset;
    local_shared_buffer += block_offset;
    local_final_output_buffer += block_offset;
    intermediate_buffer += block_offset;

    T* buffers[RanksPerNode];
#pragma unroll
    for (int ii = 0; ii < RanksPerNode; ++ii)
    {
        int rank = (params.local_rank + ii) % RanksPerNode;
        buffers[ii] = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[rank]);
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif

    for (int offset = thread_offset; offset < norm_this_block * params.fusion_params.hidden_size;
         offset += blockDim.x * kPackedSize)
    {
        *reinterpret_cast<int4*>(&local_shared_buffer[offset])
            = *reinterpret_cast<int4 const*>(&local_input_buffer[offset]);
    }
    block_barrier(
        params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RanksPerNode, tid, bid, gridDim.x);
    for (int norm_idx = 0; norm_idx < norm_this_block; ++norm_idx)
    {
        int norm_offset = norm_idx * params.fusion_params.hidden_size;
        float acc = 0.f;
        PackedStruct sum_vec, weight_vec, bias_vec, residual_vec;
        for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
        {
            PackedStruct vals[RanksPerNode];
            sum_vec.packed = {0, 0, 0, 0};
            if constexpr (Bias)
            {
                bias_vec.packed = *reinterpret_cast<int4 const*>(&bias_buffer[offset]);
            }
            residual_vec.packed = *reinterpret_cast<int4 const*>(&residual_buffer[norm_offset + offset]);
#pragma unroll
            for (int ii = 0; ii < RanksPerNode; ++ii)
            {
                vals[ii].packed = *reinterpret_cast<int4 const*>(&buffers[ii][block_offset + norm_offset + offset]);
            }
#pragma unroll
            for (int ii = 0; ii < RanksPerNode; ++ii)
            {
                sum_vec.packed = add128b(sum_vec, vals[ii]);
            }
            if constexpr (Bias)
            {
                sum_vec.packed = add128b(sum_vec, bias_vec);
            }
            sum_vec.packed = add128b(sum_vec, residual_vec);
            *reinterpret_cast<int4*>(&intermediate_buffer[norm_offset + offset]) = sum_vec.packed;
            acc = accumulate<T>(acc, sum_vec);
            if constexpr (UseSmem)
            {
                *reinterpret_cast<int4*>(&smem[offset]) = sum_vec.packed;
            }
        }
        acc = block_reduce_sum(acc);
        float denom = __fsqrt_rn(__fdividef(acc, params.fusion_params.hidden_size) + params.fusion_params.eps);
        for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
        {
            if constexpr (UseSmem)
            {
                sum_vec.packed = *reinterpret_cast<int4 const*>(&smem[offset]);
            }
            if constexpr (Affine)
            {
                weight_vec.packed = *reinterpret_cast<int4 const*>(weight_buffer + offset);
            }
            sum_vec.packed = rms_norm<T, Affine>(denom, sum_vec, weight_vec);
            *reinterpret_cast<int4*>(&local_final_output_buffer[norm_offset + offset]) = sum_vec.packed;
        }
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

template <typename T, int RanksPerNode, bool Bias, bool Affine>
void one_shot_all_reduce_norm_kernel_launcher(AllReduceParams params, hipStream_t stream)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    TLLM_CHECK(params.fusion_params.hidden_size % kPackedSize == 0);
    int need_threads = params.fusion_params.hidden_size / kPackedSize;
    int cta_size;
    if (need_threads <= details::kMaxCtaSize)
    {
        cta_size = (need_threads + details::kWarpSize - 1) / details::kWarpSize * details::kWarpSize;
    }
    else
    {
        cta_size = details::kMaxCtaSize;
    }
    int norm_num = params.elts_total / params.fusion_params.hidden_size;
    int cta_num = std::min(norm_num, static_cast<int>(MAX_ALL_REDUCE_BLOCKS));
    int smem_size = 0;

    if (cta_size * kPackedSize < params.fusion_params.hidden_size)
    {
        smem_size = params.fusion_params.hidden_size * sizeof(T);
        if (tensorrt_llm::common::getEnvEnablePDL())
        {
            TLLM_LOG_DEBUG("Enable PDL in one_shot_all_reduce_norm_kernel");

            hipLaunchConfig_t kernelConfig = {0};
            kernelConfig.gridDim = cta_num;
            kernelConfig.blockDim = cta_size;
            kernelConfig.dynamicSmemBytes = smem_size;
            kernelConfig.stream = stream;

            hipLaunchAttribute attribute[1];
            attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
            attribute[0].val.programmaticStreamSerializationAllowed = 1;
            kernelConfig.attrs = attribute;
            kernelConfig.numAttrs = 1;

            TLLM_CUDA_CHECK(cudaLaunchKernelEx(
                &kernelConfig, one_shot_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine, true>, params));
        }
        else
        {
            one_shot_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine, true>
                <<<cta_num, cta_size, smem_size, stream>>>(params);
        }
    }
    else
    {
        if (tensorrt_llm::common::getEnvEnablePDL())
        {
            hipLaunchConfig_t kernelConfig = {0};
            kernelConfig.gridDim = cta_num;
            kernelConfig.blockDim = cta_size;
            kernelConfig.dynamicSmemBytes = smem_size;
            kernelConfig.stream = stream;

            hipLaunchAttribute attribute[1];
            attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
            attribute[0].val.programmaticStreamSerializationAllowed = 1;
            kernelConfig.attrs = attribute;
            kernelConfig.numAttrs = 1;

            TLLM_LOG_DEBUG("Enable PDL in one_shot_all_reduce_norm_kernel");
            TLLM_CUDA_CHECK(cudaLaunchKernelEx(
                &kernelConfig, one_shot_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine, false>, params));
        }
        else
        {
            one_shot_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine, false>
                <<<cta_num, cta_size, smem_size, stream>>>(params);
        }
    }
}
}; // namespace reduce_fusion

template <typename T, int RANKS_PER_NODE, bool COPY_INPUT = true, bool PUSH_MODE = false>
static __global__ void oneShotAllReduceKernel(AllReduceParams params)
{
    // Suppose that two GPUs participate in the AR exchange, and we start four blocks.
    // The message is partitioned into chunks as detailed below:
    //               message
    //       |-------------------|
    // GPU 0 | B0 | B1 | B2 | B3 |
    // GPU 1 | B0 | B1 | B2 | B3 |
    //
    // Here the step-by-step behavior of one block:
    // 1. B0 copies the chunk it  is responsible for, from local_input to shareable buffer
    // 2. B0 on GPU 0 and B0 on GPU 1 wait for each other (block_barrier)
    // 3. B0 on GPU 0 pull and sum the chunk from GPU 1, writes the result to local_output
    //
    // With COPY_INPUT == false, skip step 1. and use gpu_barrier instead of block barrier during step 2.
    // We only to know if the other GPU as arrived at the AR kernel, that would mean that data is ready
    //
    // With PUSH_MODE, we consider that the shared buffer is of size:
    // params.peer_comm_buffer_ptrs: [world_size, world_size, message_size]
    //
    // Here the step-by-step behavior of one block:
    // 1. B0 push the chunk is it responsible for into all other GPUs:
    //    params.peer_comm_buffer_ptrs[:, local_gpu, B0 slice]
    // 2. block sync so the block is shared by other GPUs
    // 3. Reduce along second dimension params.peer_comm_buffer_ptrs[local_gpu, :, B0 slice]

    int const bidx = blockIdx.x;
    int const tidx = threadIdx.x;
    int const grid_size = gridDim.x;

    // The number of elements packed into one for comms
    static constexpr int PACKED_ELTS = 16 / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;

    T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
    T* local_shared_buffer = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[params.local_rank]);
    T* local_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);

    // Start and end offsets of the thread
    size_t const chunk_start = bidx * params.elts_per_block + tidx * PACKED_ELTS;
    size_t const chunk_end = std::min((bidx + 1) * params.elts_per_block, params.elts_total);

    T* buffers[RANKS_PER_NODE];
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
    {
        // buffers[0] is always the local buffers. Helps load balancing reads.
        int rank = (params.local_rank + ii) % RANKS_PER_NODE;
        buffers[ii] = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[rank]);
    }

    if constexpr (PUSH_MODE || COPY_INPUT)
    {
        // Copy from local buffer to shareable buffer
        for (size_t iter_offset = chunk_start; iter_offset < chunk_end; iter_offset += blockDim.x * PACKED_ELTS)
        {
            if constexpr (PUSH_MODE)
            {
#pragma unroll
                for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
                {
                    *reinterpret_cast<int4*>(&buffers[ii][params.local_rank * params.elts_total + iter_offset])
                        = *reinterpret_cast<int4 const*>(&local_input_buffer[iter_offset]);
                }
            }
            else
            {
                *reinterpret_cast<int4*>(&local_shared_buffer[iter_offset])
                    = *reinterpret_cast<int4 const*>(&local_input_buffer[iter_offset]);
            }
        }

        // wait for equivalent blocks of other GPUs to have copied data to their shareable buffer
        block_barrier(
            params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx, grid_size);
    }
    else
    {
        // In the non-copy case, we assume that once the kernel has been started, data is ready to be consumed
        multi_gpu_barrier(
            params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx);
    }

    // Each block accumulates the values from the different GPUs on the same node.
    for (size_t iter_offset = chunk_start; iter_offset < chunk_end; iter_offset += blockDim.x * PACKED_ELTS)
    {
        // Iterate over the different ranks/devices on the node to load the values.
        PackedStruct vals[RANKS_PER_NODE];
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            if constexpr (PUSH_MODE)
            {
                vals[ii].packed
                    = *reinterpret_cast<int4 const*>(&buffers[params.local_rank][ii * params.elts_total + iter_offset]);
            }
            else
            {
                vals[ii].packed = *reinterpret_cast<int4 const*>(&buffers[ii][iter_offset]);
            }
        }

        // Sum the values from the different ranks.
        PackedStruct sums;
        sums.packed = {0, 0, 0, 0};
#pragma unroll
        for (int rank = 0; rank < RANKS_PER_NODE; ++rank)
        {
            // Always reduce from rank 0 to ensure stable reduce order.
            int ii = (rank + RANKS_PER_NODE - params.local_rank) % RANKS_PER_NODE;
            sums.packed = add128b(sums, vals[ii]);
        }
        // Store to the destination buffer.
        *reinterpret_cast<int4*>(&local_output_buffer[iter_offset]) = sums.packed;
    }
}

template <typename T, int RANKS_PER_NODE, bool COPY_INPUT = true, bool PUSH_MODE = false, bool Bias = false,
    bool Residual = false>
static __global__ void __launch_bounds__(512, 1) twoShotAllReduceKernel(AllReduceParams params)
{
    // Suppose that two GPUs participate in the AR exchange, and we start two blocks.
    // The message is partitioned into chunks as detailed below:
    //               message
    //       |-------------------|
    //       |--GPU 0--|--GPU 1--| (GPU responsibility parts)
    // GPU 0 | B0 | B1 | B0 | B1 |
    // GPU 1 | B0 | B1 | B0 | B1 |
    //
    // Here the step-by-step behavior of one block:
    // 1. B0 copies all chunks is it responsible for, from local_input to shareable buffer
    // 2. B0 on GPU 0 and B0 on GPU 1 wait for each other (block_barrier #0)
    // 3. B0 on GPU 0 gather and sum the B0 chunks from GPU 1, that are in the GPU 0 responsibility
    //    part (the first half of the message, see GPU responsibility row above)
    // 3bis. Likewise, B0 on GPU 1 copies and sum the chunks for GPU 0,
    //       where GPU 1 is responsible: the second half of the message.
    // 4. B0 on GPU 0 and B0 on GPU 1 wait for each other (block_barrier #1)
    // 5. B0 writes result to local_output. It gathers each chunk from its responsible GPU.
    //    For example, here it reads the first chunk from GPU 0 and second chunk from GPU 1.
    //
    // With COPY_INPUT == false, skip step 1. and use gpu_barrier instead of block barrier during step 2.
    // We only to know if the other GPU as arrived at the AR kernel, that would mean that data is ready
    // to be read.
    //
    // Note that compared to one-shot, one block (CTA) writes multiple input chunks and write multiple output chunks.
    // However, it's only responsible for the summation of a single chunk.
    //
    // With PUSH_MODE, we consider that the shared buffer is of size:
    // params.peer_comm_buffer_ptrs: [world_size, world_size, message_size / world_size]
    //
    // Here the step-by-step behavior of one block:
    // 1. B0 push the chunks is it responsible for into the corresponding GPUs:
    //    params.peer_comm_buffer_ptrs[target_gpu, local_gpu, current B0 slice]
    // 2. block sync so the blocks have been shared by other GPUs
    // 3. Reduce along second dimension params.peer_comm_buffer_ptrs[local_gpu, :, B0 slice]
    // 4. block barrier (corresponding blocks have finished reduction)
    // 5. pull and write on local buffer, by reading params.peer_comm_buffer_ptrs[:, 0, B0 slice] (reduction result is
    //    written at index 0 of 2nd dim)

    int const bidx = blockIdx.x;
    int const tidx = threadIdx.x;
    int const grid_size = gridDim.x;

    // The number of elements packed into one for comms
    static constexpr int PACKED_ELTS = 16 / sizeof(T);
    using PackedType = typename PackedOn16Bytes<T>::Type;

    T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
    T* local_shared_buffer = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[params.local_rank]);
    T* local_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);

    size_t const chunk_start = bidx * params.elts_per_block + tidx * PACKED_ELTS;
    size_t const chunk_end = min(chunk_start + params.elts_per_block, params.elts_per_rank);

    T* buffers[RANKS_PER_NODE];
    int ranks[RANKS_PER_NODE];
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
    {
        // A mapping of the ranks to scatter reads as much as possible
        int rank = (params.local_rank + ii) % RANKS_PER_NODE;
        ranks[ii] = rank;
        buffers[ii] = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[rank]);
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif

    if constexpr (PUSH_MODE || COPY_INPUT)
    {
        // Copy all blocks from local buffer to shareable buffer
        for (size_t local_offset = chunk_start; local_offset < chunk_end; local_offset += blockDim.x * PACKED_ELTS)
        {
#pragma unroll
            for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
            {
                size_t offset_rank = ranks[ii] * params.elts_per_rank + local_offset;
                if (offset_rank >= params.elts_total)
                {
                    continue;
                }

                if constexpr (PUSH_MODE)
                {
                    *reinterpret_cast<int4*>(&buffers[ii][params.local_rank * params.elts_per_rank + local_offset])
                        = *reinterpret_cast<int4 const*>(&local_input_buffer[offset_rank]);
                }
                else
                {
                    *reinterpret_cast<int4*>(&local_shared_buffer[offset_rank])
                        = *reinterpret_cast<int4 const*>(&local_input_buffer[offset_rank]);
                }
            }
        }
        block_barrier(
            params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx, grid_size);
    }
    else
    {
        // In the non-copy case, we assume that once the kernel has been started, data is ready to be consumed
        multi_gpu_barrier(
            params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx);
    }

    // Each block accumulates the values from the different GPUs on the same node.
    for (size_t local_offset = chunk_start; local_offset < chunk_end; local_offset += blockDim.x * PACKED_ELTS)
    {
        size_t const responsible_block_offset = local_offset + params.rank_offset;

        // Iterate over the different ranks/devices on the node to load the values.
        PackedType vals[RANKS_PER_NODE];
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            if constexpr (PUSH_MODE)
            {
                vals[ii].packed
                    = *reinterpret_cast<int4 const*>(&local_shared_buffer[ii * params.elts_per_rank + local_offset]);
            }
            else
            {
                vals[ii].packed = *reinterpret_cast<int4 const*>(&buffers[ii][responsible_block_offset]);
            }
        }

        // Sum the values from the different ranks.
        PackedType sums;
        sums.packed = {0, 0, 0, 0};
#pragma unroll
        for (int rank = 0; rank < RANKS_PER_NODE; ++rank)
        {
            // Always reduce from rank 0 to ensure stable reduce order.
            int ii = (rank + RANKS_PER_NODE - params.local_rank) % RANKS_PER_NODE;
            sums.packed = add128b(sums, vals[ii]);
        }

        // Store to the local buffer.
        if constexpr (PUSH_MODE)
        {
            *reinterpret_cast<int4*>(&local_shared_buffer[local_offset]) = sums.packed;
        }
        else
        {
            *reinterpret_cast<int4*>(&local_shared_buffer[responsible_block_offset]) = sums.packed;
        }
    }

    block_barrier(
        params.peer_barrier_ptrs_out, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx, grid_size);

    // Gather all needed elts from other intra-node ranks
    for (size_t local_offset = chunk_start; local_offset < chunk_end; local_offset += blockDim.x * PACKED_ELTS)
    {
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            // use round-robin gathering from other ranks
            size_t offset_rank = ranks[ii] * params.elts_per_rank + local_offset;
            if (offset_rank >= params.elts_total)
            {
                continue;
            }
            PackedType sums, residual_vec, bias_vec;
            if constexpr (Bias)
            {
                bias_vec.packed
                    = *reinterpret_cast<int4 const*>(reinterpret_cast<T const*>(params.fusion_params.bias_buffer)
                        + offset_rank % params.fusion_params.hidden_size);
            }
            if constexpr (Residual)
            {
                residual_vec.packed = *reinterpret_cast<int4 const*>(
                    reinterpret_cast<T const*>(params.fusion_params.residual_buffer) + offset_rank);
            }
            if constexpr (PUSH_MODE)
            {
                *reinterpret_cast<int4*>(&local_output_buffer[offset_rank])
                    = *reinterpret_cast<int4*>(&buffers[ii][local_offset]);
                sums.packed = *reinterpret_cast<int4*>(&buffers[ii][local_offset]);
            }
            else
            {
                *reinterpret_cast<int4*>(&local_output_buffer[offset_rank])
                    = *reinterpret_cast<int4*>(&buffers[ii][offset_rank]);
                sums.packed = *reinterpret_cast<int4*>(&buffers[ii][offset_rank]);
            }
            if constexpr (Bias)
            {
                sums.packed = add128b(sums, bias_vec);
            }
            if constexpr (Residual)
            {
                sums.packed = add128b(sums, residual_vec);
            }
            *reinterpret_cast<int4*>(&local_output_buffer[offset_rank]) = sums.packed;
        }
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

bool configurationSupported(AllReduceStrategyType algo, size_t msg_size, size_t n_ranks, nvinfer1::DataType type)
{
    size_t elts_per_thread = 16 / common::getDTypeSize(type);
    int const msg_align = (algo == AllReduceStrategyType::TWOSHOT) ? n_ranks * elts_per_thread : elts_per_thread;
    bool supported_algo = (algo == AllReduceStrategyType::ONESHOT || algo == AllReduceStrategyType::TWOSHOT);
    return supported_algo && (msg_size % msg_align == 0);
}

std::tuple<int, int> kernelLaunchConfig(AllReduceStrategyType algo, AllReduceParams& params, size_t elts_per_thread)
{
    int blocks_per_grid = 1, threads_per_block = DEFAULT_BLOCK_SIZE;

    switch (algo)
    {
    case AllReduceStrategyType::ONESHOT:
    {
        TLLM_CHECK(params.elts_total % elts_per_thread == 0);
        size_t const total_threads = roundUp(params.elts_total / elts_per_thread, WARP_SIZE);
        threads_per_block = std::min(DEFAULT_BLOCK_SIZE, total_threads);
        blocks_per_grid = std::min(static_cast<size_t>(MAX_ALL_REDUCE_BLOCKS), divUp(total_threads, threads_per_block));
        params.elts_per_block = roundUp(divUp(params.elts_total, blocks_per_grid), elts_per_thread);
        break;
    }
    case AllReduceStrategyType::TWOSHOT:
    {
        TLLM_CHECK(params.elts_total % (elts_per_thread * params.ranks_per_node) == 0);
        size_t const total_threads = roundUp(params.elts_total / (elts_per_thread * params.ranks_per_node), WARP_SIZE);

        /*
        threads_per_block = std::min(DEFAULT_BLOCK_SIZE, total_threads);
        blocks_per_grid = std::min(static_cast<size_t>(MAX_ALL_REDUCE_BLOCKS), divUp(total_threads, threads_per_block));
        */
        while (total_threads % blocks_per_grid != 0 || total_threads / blocks_per_grid > DEFAULT_BLOCK_SIZE)
        {
            blocks_per_grid += 1;
        }

        threads_per_block = total_threads / blocks_per_grid;

        // NOTE: need to adjust here
        if (blocks_per_grid > MAX_ALL_REDUCE_BLOCKS)
        {
            size_t iter_factor = 1;
            while (blocks_per_grid / iter_factor > MAX_ALL_REDUCE_BLOCKS || blocks_per_grid % iter_factor)
            {
                iter_factor += 1;
            }
            blocks_per_grid /= iter_factor;
        }
        params.elts_per_rank = params.elts_total / params.ranks_per_node;
        params.rank_offset = params.local_rank * params.elts_per_rank;
        params.elts_per_block = roundUp(divUp(params.elts_per_rank, blocks_per_grid), elts_per_thread);
        break;
    }
    default: TLLM_THROW("Algorithm not supported here.");
    }

    return std::make_tuple(blocks_per_grid, threads_per_block);
}

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false, bool USE_MEMCPY = false, bool Bias = false,
    bool Affine = false>
void AllReduceNormKernelLaunch(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream)
{
    TLLM_CHECK_WITH_INFO(fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM, "Unsupported AllReduceFusionOp: %d",
        static_cast<int>(fusionOp));
    if (algo == AllReduceStrategyType::ONESHOT)
    {
        reduce_fusion::one_shot_all_reduce_norm_kernel_launcher<T, RANKS_PER_NODE, Bias, Affine>(params, stream);
    }
    else
    {
        TLLM_CHECK_WITH_INFO(!(USE_MEMCPY && PUSH_MODE), "Memcpy cannot be used with PUSH_MODE.");
        size_t elts_per_thread = 16 / sizeof(T);
        auto [blocks_per_grid, threads_per_block] = kernelLaunchConfig(algo, params, elts_per_thread);
        if (USE_MEMCPY)
        {
            hipMemcpyAsync(params.peer_comm_buffer_ptrs[params.local_rank], params.local_input_buffer_ptr,
                params.elts_total * sizeof(T), hipMemcpyDeviceToDevice, stream);
        }
        auto output_ptr = params.local_output_buffer_ptr;
        params.local_output_buffer_ptr = params.fusion_params.intermediate_buffer;

        if (tensorrt_llm::common::getEnvEnablePDL())
        {
            TLLM_LOG_DEBUG("Enable PDL in twoShotAllReduceKernel");
            hipLaunchConfig_t kernelConfig = {0};
            kernelConfig.gridDim = blocks_per_grid;
            kernelConfig.blockDim = threads_per_block;
            kernelConfig.dynamicSmemBytes = 0;
            kernelConfig.stream = stream;

            hipLaunchAttribute attribute[1];
            attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
            attribute[0].val.programmaticStreamSerializationAllowed = 1;
            kernelConfig.attrs = attribute;
            kernelConfig.numAttrs = 1;

            TLLM_CUDA_CHECK(cudaLaunchKernelEx(
                &kernelConfig, twoShotAllReduceKernel<T, RANKS_PER_NODE, !USE_MEMCPY, PUSH_MODE, Bias, true>, params));
        }
        else
        {
            twoShotAllReduceKernel<T, RANKS_PER_NODE, !USE_MEMCPY, PUSH_MODE, Bias, true>
                <<<blocks_per_grid, threads_per_block, 0, stream>>>(params);
        }
        params.local_output_buffer_ptr = output_ptr;
        reduce_fusion::rms_norm_kernel_launcher<T, false, false, Affine>(params, stream);
    }
}

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false, bool USE_MEMCPY = false>
void AllReduceNormDispatch(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream)
{
    if (params.fusion_params.bias_buffer && params.fusion_params.weight_buffer)
    {
        AllReduceNormKernelLaunch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY, true, true>(
            algo, config, fusionOp, params, stream);
    }
    else if (params.fusion_params.bias_buffer && !params.fusion_params.weight_buffer)
    {
        AllReduceNormKernelLaunch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY, true, false>(
            algo, config, fusionOp, params, stream);
    }
    else if (!params.fusion_params.bias_buffer && params.fusion_params.weight_buffer)
    {
        AllReduceNormKernelLaunch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY, false, true>(
            algo, config, fusionOp, params, stream);
    }
    else
    {
        AllReduceNormKernelLaunch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY, false, false>(
            algo, config, fusionOp, params, stream);
    }
}

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false, bool USE_MEMCPY = false>
void AllReduceDispatch(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream)
{
    TLLM_CHECK(fusionOp == AllReduceFusionOp::NONE);
    TLLM_CHECK_WITH_INFO(!(USE_MEMCPY && PUSH_MODE), "Memcpy cannot be used with PUSH_MODE.");
    size_t elts_per_thread = 16 / sizeof(T);
    auto [blocks_per_grid, threads_per_block] = kernelLaunchConfig(algo, params, elts_per_thread);
    if (USE_MEMCPY)
    {
        hipMemcpyAsync(params.peer_comm_buffer_ptrs[params.local_rank], params.local_input_buffer_ptr,
            params.elts_total * sizeof(T), hipMemcpyDeviceToDevice, stream);
    }
    if (algo == AllReduceStrategyType::ONESHOT)
    {
        oneShotAllReduceKernel<T, RANKS_PER_NODE, !USE_MEMCPY, PUSH_MODE>
            <<<blocks_per_grid, threads_per_block, 0, stream>>>(params);
    }
    else
    {
        twoShotAllReduceKernel<T, RANKS_PER_NODE, !USE_MEMCPY, PUSH_MODE>
            <<<blocks_per_grid, threads_per_block, 0, stream>>>(params);
    }
}

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false, bool USE_MEMCPY = false>
void AllReduceDispatchMemcpy(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream)
{
    if (fusionOp == AllReduceFusionOp::NONE)
    {
        AllReduceDispatch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY>(algo, config, fusionOp, params, stream);
    }
    else
    {
        AllReduceNormDispatch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY>(algo, config, fusionOp, params, stream);
    }
}

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false>
void AllReduceDispatchPushMode(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream)
{
    if (static_cast<std::underlying_type_t<AllReduceStrategyConfig>>(config)
        & static_cast<std::underlying_type_t<AllReduceStrategyConfig>>(AllReduceStrategyConfig::USE_MEMCPY))
    {
        AllReduceDispatchMemcpy<T, RANKS_PER_NODE, PUSH_MODE, true>(algo, config, fusionOp, params, stream);
    }
    else
    {
        AllReduceDispatchMemcpy<T, RANKS_PER_NODE, PUSH_MODE, false>(algo, config, fusionOp, params, stream);
    }
}

template <typename T, int RANKS_PER_NODE> //, bool USE_MEMCPY = false, bool PUSH_MODE = false>
void AllReduceDispatchRanksPerNode(AllReduceStrategyType algo, AllReduceStrategyConfig config,
    AllReduceFusionOp fusionOp, AllReduceParams& params, hipStream_t stream)
{
    if (static_cast<std::underlying_type_t<AllReduceStrategyConfig>>(config)
        & static_cast<std::underlying_type_t<AllReduceStrategyConfig>>(AllReduceStrategyConfig::PUSH_MODE))
    {
        AllReduceDispatchPushMode<T, RANKS_PER_NODE, true>(algo, config, fusionOp, params, stream);
    }
    else
    {
        AllReduceDispatchPushMode<T, RANKS_PER_NODE, false>(algo, config, fusionOp, params, stream);
    }
}

template <typename T>
void AllReduceDispatchType(AllReduceParams& params, AllReduceStrategyType strat, AllReduceStrategyConfig config,
    AllReduceFusionOp fusionOp, hipStream_t stream)
{
    switch (params.ranks_per_node)
    {
    case 2: AllReduceDispatchRanksPerNode<T, 2>(strat, config, fusionOp, params, stream); break;
    case 4: AllReduceDispatchRanksPerNode<T, 4>(strat, config, fusionOp, params, stream); break;
    case 6: AllReduceDispatchRanksPerNode<T, 6>(strat, config, fusionOp, params, stream); break;
    case 8: AllReduceDispatchRanksPerNode<T, 8>(strat, config, fusionOp, params, stream); break;
    default: TLLM_THROW("Custom all reduce only supported on {2, 4, 6, 8} GPUs per node.");
    }
}

AllReduceParams AllReduceParams::deserialize(int64_t* buffer, size_t tpSize, size_t tpRank)
{
    void* const* buffer_ptrs = reinterpret_cast<void* const*>(buffer);
    auto const flag_ptr = &buffer[4 * tpSize];
    // cannot use 0 since 0 represents released state for barrier
    *flag_ptr += 1;
    TLLM_LOG_TRACE("AllReduceParams's flag value is %d", *flag_ptr);
    uint32_t flag_value = *flag_ptr;
    AllReduceParams params;
    // Even plugins use ping buffers, odd plugins use pong.
    // That way, we don't need to wait for other GPUs to be done
    // before copying input tensor to workspace.
    auto const buffer_offset = (flag_value % 2 == 0) ? 0 : tpSize;

    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_comm_buffer_ptrs[i] = buffer_ptrs[buffer_offset + i];
    }
    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_barrier_ptrs_in[i] = reinterpret_cast<uint32_t*>(buffer_ptrs[2 * tpSize + i]);
    }
    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_barrier_ptrs_out[i] = reinterpret_cast<uint32_t*>(buffer_ptrs[3 * tpSize + i]);
    }
    params.barrier_flag = flag_value;
    params.ranks_per_node = tpSize;
    params.local_rank = tpRank;

    return params;
}

void customAllReduce(kernels::AllReduceParams& params, nvinfer1::DataType dataType, AllReduceStrategyType strat,
    AllReduceStrategyConfig config, AllReduceFusionOp fusionOp, hipStream_t stream)
{
    TLLM_CHECK_WITH_INFO(configurationSupported(strat, params.elts_total, params.ranks_per_node, dataType),
        "Custom all-reduce configuration unsupported");

    sync_check_cuda_error();

    switch (dataType)
    {
    case nvinfer1::DataType::kFLOAT: AllReduceDispatchType<float>(params, strat, config, fusionOp, stream); break;
    case nvinfer1::DataType::kHALF: AllReduceDispatchType<half>(params, strat, config, fusionOp, stream); break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16:
        AllReduceDispatchType<__hip_bfloat16>(params, strat, config, fusionOp, stream);
        break;
#endif
    default: TLLM_THROW("Unsupported dataType for customAllReduce");
    }
    sync_check_cuda_error();
}

template <typename T>
void launchResidualRmsNormKernel(kernels::AllReduceParams& params, hipStream_t stream)
{
    if (params.fusion_params.bias_buffer && params.fusion_params.weight_buffer)
    {
        reduce_fusion::rms_norm_kernel_launcher<T, true, true, true>(params, stream);
    }
    else if (params.fusion_params.bias_buffer && !params.fusion_params.weight_buffer)
    {
        reduce_fusion::rms_norm_kernel_launcher<T, true, true, false>(params, stream);
    }
    else if (!params.fusion_params.bias_buffer && params.fusion_params.weight_buffer)
    {
        reduce_fusion::rms_norm_kernel_launcher<T, false, true, true>(params, stream);
    }
    else
    {
        reduce_fusion::rms_norm_kernel_launcher<T, false, true, false>(params, stream);
    }
}

void residualRmsNorm(kernels::AllReduceParams& params, nvinfer1::DataType dataType, hipStream_t stream)
{
    sync_check_cuda_error();
    switch (dataType)
    {
    case nvinfer1::DataType::kFLOAT: launchResidualRmsNormKernel<float>(params, stream); break;
    case nvinfer1::DataType::kHALF: launchResidualRmsNormKernel<half>(params, stream); break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16: launchResidualRmsNormKernel<__hip_bfloat16>(params, stream); break;
#endif
    default: TLLM_THROW("Unsupported dataType for customAllReduce");
    }
    sync_check_cuda_error();
}

} // namespace tensorrt_llm::kernels
