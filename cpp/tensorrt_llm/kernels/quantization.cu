#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/quantTypeUtils.cuh"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/quantization.h"
#include <float.h>

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

__global__ void quantizedKernel(char4* dst, float4 const* src, const int64_t sizeDiv4, float const* scalePtr)
{
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x)
    {
        float const scale = __ldg(scalePtr);
        char4 tmp;
        const float4 floatTmp = __ldg(src + idx);
        tmp.x = cuda_cast<int8_t>(floatTmp.x * scale);
        tmp.y = cuda_cast<int8_t>(floatTmp.y * scale);
        tmp.z = cuda_cast<int8_t>(floatTmp.z * scale);
        tmp.w = cuda_cast<int8_t>(floatTmp.w * scale);
        dst[idx] = tmp;
    }
}

__global__ void quantizedKernel(char4* dst, half2 const* src, const int64_t sizeDiv4, float const* scalePtr)
{
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x)
    {
        float const scale = __ldg(scalePtr);
        char4 tmp;
        int srcId = idx << 1;

        const uint2 h2 = __ldg(reinterpret_cast<uint2 const*>(src + srcId));

        const half2 half2Tmp = reinterpret_cast<half2 const&>(h2.x);
        const half2 half2Tmp2 = reinterpret_cast<half2 const&>(h2.y);

        tmp.x = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.x) * scale);
        tmp.y = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.y) * scale);
        tmp.z = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.x) * scale);
        tmp.w = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.y) * scale);
        dst[idx] = tmp;
    }
}

#ifdef ENABLE_BF16
__global__ void quantizedKernel(char4* dst, __hip_bfloat162 const* src, const int64_t sizeDiv4, float const* scalePtr)
{
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x)
    {
        float const scale = __ldg(scalePtr);
        char4 tmp;
        int srcId = idx << 1;

        const uint2 h2 = __ldg(reinterpret_cast<uint2 const*>(src + srcId));

        const __hip_bfloat162 bfloat162Tmp = reinterpret_cast<__hip_bfloat162 const&>(h2.x);
        const __hip_bfloat162 bfloat162Tmp2 = reinterpret_cast<__hip_bfloat162 const&>(h2.y);

        tmp.x = cuda_cast<int8_t>(cuda_cast<float>(bfloat162Tmp.x) * scale);
        tmp.y = cuda_cast<int8_t>(cuda_cast<float>(bfloat162Tmp.y) * scale);
        tmp.z = cuda_cast<int8_t>(cuda_cast<float>(bfloat162Tmp2.x) * scale);
        tmp.w = cuda_cast<int8_t>(cuda_cast<float>(bfloat162Tmp2.y) * scale);

        dst[idx] = tmp;
    }
}
#endif

template <typename T>
void invokeQuantization(
    int8_t* dst, T const* src, const int64_t size, float const* scalePtr, hipStream_t stream, int maxGridSize)
{
    TLLM_CHECK_WITH_INFO(size % 4 == 0, "[ERROR][invokeQuantization] size should be a multiple of 4.\n");

    int numBlocks{static_cast<int>((size + 255) / 256)};
    dim3 grid(std::min(numBlocks, maxGridSize));
    TLLM_CHECK_WITH_INFO(grid.x <= maxGridSize, "[ERROR][invokeQuantization] grid max size is exceeded\n");
    dim3 block(64);
    if (std::is_same_v<T, float>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (float4 const*) src, size / 4, scalePtr);
    }
    else if (std::is_same_v<T, half>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (half2 const*) src, size / 4, scalePtr);
    }
#ifdef ENABLE_BF16
    else if (std::is_same_v<T, __hip_bfloat16>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (__hip_bfloat162 const*) src, size / 4, scalePtr);
    }
#endif
}

template void invokeQuantization<float>(
    int8_t* dst, float const* src, const int64_t size, float const* scalePtr, hipStream_t stream, int maxGridSize);

template void invokeQuantization<half>(
    int8_t* dst, half const* src, const int64_t size, float const* scalePtr, hipStream_t stream, int maxGridSize);

#ifdef ENABLE_BF16
template void invokeQuantization<__hip_bfloat16>(int8_t* dst, __hip_bfloat16 const* src, const int64_t size,
    float const* scalePtr, hipStream_t stream, int maxGridSize);
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T, int NUM_ELTS>
struct DstVec
{
    static_assert("not implemented.");
};

template <>
struct DstVec<float2, 2>
{
    using Type = uint32_t;
};

template <>
struct DstVec<half2, 4>
{
    using Type = uint2;
};

#ifdef ENABLE_BF16

template <>
struct DstVec<__hip_bfloat162, 4>
{
    using Type = uint2;
};

#endif // ENABLE_BF16

template <typename T>
struct DstVec<T, 4>
{
    static_assert(sizeof(T) == 4, "not implemented.");
    using Type = uint32_t;
};

template <typename T>
struct DstVec<T, 8>
{
    static_assert(sizeof(T) == 2, "not implemented.");
    using Type = uint2;
};

////////////////////////////////////////////////////////////////////////////////////////////////////

// Helper function of getting the absMax of all elements in the vector after clamping.
// Pack two elements in order to use possible hmax2 instructions.
template <typename T>
inline __device__ void clampAndAbsMax(T& localMax, uint4& vec, T const clampMin, T const clampMax)
{
    static constexpr int NUM_ELTS = sizeof(uint4) / sizeof(T);

#pragma unroll
    for (int i = 0; i < NUM_ELTS; ++i)
    {
        T& val = reinterpret_cast<T*>(&vec)[i];
        val = cuda_clamp(val, clampMin, clampMax);
        localMax = cuda_max(localMax, cuda_abs(val));
    }
}

// Helper function of quantizing the vector and storing it to global memory.
// Pack two elements in order to use fast convert instructions.
template <typename T, typename QuantT, bool USE_SMEM>
inline __device__ void quantizeAndStore(
    QuantT* dstPtr, uint4 vec, T const clampMin, T const clampMax, float const scaleOrigQuant)
{
    static constexpr int NUM_ELTS = sizeof(uint4) / sizeof(T);

    using DstVecType = typename DstVec<T, NUM_ELTS>::Type;
    DstVecType dstVec;
#pragma unroll
    for (int i = 0; i < NUM_ELTS; ++i)
    {
        T val = reinterpret_cast<T*>(&vec)[i];
        // Values loaded from smem has already been clamped.
        if constexpr (!USE_SMEM)
        {
            val = cuda_clamp(val, clampMin, clampMax);
        }
        float2 val2 = cuda_cast<float2>(val);
        val2.x *= scaleOrigQuant;
        val2.y *= scaleOrigQuant;
        QuantT quantVal = cuda_cast<QuantT>(val2);
        reinterpret_cast<QuantT*>(&dstVec)[i] = quantVal;
    }
    // Store to destination buffer.
    *reinterpret_cast<DstVecType*>(dstPtr) = dstVec;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T, typename QuantT, bool USE_SMEM>
__global__ void perTokenQuantization(QuantT* dst, T const* src, const int64_t numRows, const int64_t numCols,
    float const* clampPtr, float* scalePtr, bool hasFp8MinScaling)
{
    // Smem buffer.
    extern __shared__ uint4 smemBuffer[];

    // The clamping minimum / maximum values.
    T const clampMin = cuda_cast<T>(clampPtr ? clampPtr[0] : -FLT_MAX);
    T const clampMax = cuda_cast<T>(clampPtr ? clampPtr[1] : FLT_MAX);

    // Pack two elements in order to use higher through instructions.
    using T2 = typename packed_as<T, 2>::type;
    using QuantT2 = typename packed_as<QuantT, 2>::type;
    T2 const clampMin2 = cuda_cast<T2, T>(clampMin);
    T2 const clampMax2 = cuda_cast<T2, T>(clampMax);

    // The quantized data type's maximum value (upper-bound).
    static constexpr float MAX_QUANT_VAL = QuantTypeStaticVals<QuantT>::MAX_VAL;
    // The minimum scaling factor (lower-bound).
    static constexpr float MIN_SCALING_FACTOR = QuantTypeStaticVals<QuantT>::MIN_SCALING_FACTOR;
    static constexpr float MIN_SCALING_FACTOR_RCP = QuantTypeStaticVals<QuantT>::MIN_SCALING_FACTOR_RCP;

    // The number of elements in the packed uint4 vec.
    static constexpr int NUM_ELTS_PER_VEC = sizeof(uint4) / sizeof(T);
    // The number of vectors in the column.
    int const numColVecs = numCols / NUM_ELTS_PER_VEC;
    // The vector pointers for src.
    uint4 const* srcVec = reinterpret_cast<uint4 const*>(src) + blockIdx.x * numColVecs;
    // The pointer for dst.
    QuantT* dstRow = dst + blockIdx.x * numCols;
    // T const* srcRow = src + blockIdx.x * numCols;

    T2 localMax2 = cuda_cast<T2, T>(T(1e-6f));
    for (int i = threadIdx.x; i < numColVecs; i += blockDim.x)
    {
        uint4 vec = srcVec[i];
        clampAndAbsMax(localMax2, vec, clampMin2, clampMax2);
        // Avoid reloading from global memory.
        if constexpr (USE_SMEM)
        {
            smemBuffer[i] = vec;
        }
    }
    float const rowMax = blockAllReduceMax(cuda_cast<float>(cuda_max<T, T2>(localMax2)));

    if (threadIdx.x == 0)
    {
        scalePtr[blockIdx.x]
            = hasFp8MinScaling ? cuda_max(rowMax / MAX_QUANT_VAL, MIN_SCALING_FACTOR) : (rowMax / MAX_QUANT_VAL);
    }

    float const scaleOrigQuant
        = hasFp8MinScaling ? fminf(MAX_QUANT_VAL / rowMax, MIN_SCALING_FACTOR_RCP) : MAX_QUANT_VAL / rowMax;
    for (int i = threadIdx.x; i < numColVecs; i += blockDim.x)
    {
        uint4 vec = USE_SMEM ? smemBuffer[i] : srcVec[i];
        QuantT2* dstPtr = reinterpret_cast<QuantT2*>(dstRow + i * NUM_ELTS_PER_VEC);
        quantizeAndStore<T2, QuantT2, USE_SMEM>(dstPtr, vec, clampMin2, clampMax2, scaleOrigQuant);
    }
}

// Do per-token (row) quantization from fp16/bf16/fp32 to int8/fp8_e4m3.
template <typename T, typename QuantT>
void invokePerTokenQuantization(QuantT* dst, T const* src, const int64_t numRows, const int64_t numCols,
    float const* clampPtr, float* scalePtr, QuantMode quantMode, hipStream_t stream)
{
    // each block is responsible for a single row
    const dim3 block(512);
    const dim3 grid(numRows);

    // The number of elements in the packed uint4 vec.
    static constexpr int NUM_ELTS_PER_VEC = sizeof(uint4) / sizeof(T);
    TLLM_CHECK_WITH_INFO(numCols % NUM_ELTS_PER_VEC == 0, "Not supported.");

    // Cache vectors to smem to avoid reloading.
    size_t const dynamicSmemSz = numCols * sizeof(T);
    // Need to check if smem capacity is enough.
    bool useSmem = true;
    if (dynamicSmemSz >= 48 * 1024)
    {
        hipError_t res = hipFuncSetAttribute(reinterpret_cast<const void*>(
            perTokenQuantization<T), QuantT, true>, hipFuncAttributeMaxDynamicSharedMemorySize, dynamicSmemSz);
        // Fall back to reloading-reversion if smem is not enough.
        useSmem = (res == hipSuccess);
    }

    // Enable min_scaling_factor if it is fp8 rowwise per-token quantization.
    bool hasFp8MinScaling = quantMode.hasFp8RowWise();
    // Do we use smem ?
    if (useSmem)
    {
        perTokenQuantization<T, QuantT, true>
            <<<grid, block, dynamicSmemSz, stream>>>(dst, src, numRows, numCols, clampPtr, scalePtr, hasFp8MinScaling);
    }
    else
    {
        perTokenQuantization<T, QuantT, false>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, clampPtr, scalePtr, hasFp8MinScaling);
    }
}

#define INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(T, QuantT)                                                           \
    template void invokePerTokenQuantization(QuantT* dst, const T* src, const int64_t numRows, const int64_t numCols,  \
        float const* clampPtr, float* scalePtr, QuantMode quantMode, hipStream_t stream)

INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(float, int8_t);
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(half, int8_t);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(__hip_bfloat16, int8_t);
#endif

#ifdef ENABLE_FP8
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(float, __hip_fp8_e4m3_fnuz);
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(half, __hip_fp8_e4m3_fnuz);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(__hip_bfloat16, __hip_fp8_e4m3_fnuz);
#endif
#endif

} // namespace kernels
} // namespace tensorrt_llm
