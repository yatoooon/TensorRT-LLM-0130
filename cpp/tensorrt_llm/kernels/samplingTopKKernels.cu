#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

#include "tensorrt_llm/common/logger.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/samplingTopKKernels.h"

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm::kernels
{

template <typename T, int32_t BLOCK_SIZE_, int32_t BLOCKS_PER_BEAM_>
__global__ void topKStage1(T const* __restrict logProbs, T const* const* __restrict logProbsPtrs, T* tmpLogProbs,
    SizeType32* topKTmpIdBuf, T* topKTmpValBuf, FinishedState const* finished, SizeType32 maxTopK,
    SizeType32 const* topKs, SizeType32 vocabSize, TokenIdType const* endIds, bool const* skipDecode,
    SizeType32 const* batchSlots, SizeType32 const* tokensPerStep, SizeType32 maxTokensPerStep)
{
    typedef hipcub::BlockReduce<TopK_2<T>, BLOCK_SIZE_> BlockReduce;
    __shared__ typename BlockReduce::TempStorage tempStorage;

    auto const tid = static_cast<SizeType32>(threadIdx.x);
    auto const bid = static_cast<SizeType32>(blockIdx.x);
    auto const tokenIdx = static_cast<SizeType32>(blockIdx.y);

    auto const batchId = bid / BLOCKS_PER_BEAM_; // row id for logProbs
    auto const batchSlot = batchSlots[batchId];
    if (tokensPerStep != nullptr && tokenIdx >= tokensPerStep[batchSlot])
    {
        return;
    }

    FinishedState const finishState = finished != nullptr ? finished[batchSlot] : FinishedState::empty();
    if ((skipDecode != nullptr && skipDecode[batchSlot]) || (finishState.isSkipDecoding()))
    {
        return;
    }

    auto const logBufIndex = batchId * maxTokensPerStep * vocabSize + tokenIdx * vocabSize;
    auto logProbsSlot
        = logProbsPtrs == nullptr ? logProbs + logBufIndex : logProbsPtrs[batchId * maxTokensPerStep + tokenIdx];

    auto const blockLane = bid % BLOCKS_PER_BEAM_;                  // block id for a beam
    auto const k = (topKs != nullptr) ? topKs[batchSlot] : maxTopK; // batchId = batch index

    auto const tmpLogBufIndex = batchId * maxTokensPerStep * vocabSize + tokenIdx * vocabSize;
    auto const tmpTopKBufIndex = batchId * maxTokensPerStep * BLOCKS_PER_BEAM_ * maxTopK
        + tokenIdx * BLOCKS_PER_BEAM_ * maxTopK + blockLane * k;

    TopK_2<T> partial;
    bool const IS_FP16 = std::is_same<T, half>::value;
    T const MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    if (finished != nullptr && finishState.isFinished())
    {
        if (tid < k)
        {
            auto const index = tmpTopKBufIndex + tid;
            if (blockLane == 0 && tid == 0)
            {
                auto const endId = endIds[batchSlot];
                topKTmpIdBuf[index] = tmpLogBufIndex + endId;
                topKTmpValBuf[index] = logProbsSlot[endId];
            }
            else
            {
                topKTmpIdBuf[index] = -1;
                topKTmpValBuf[index] = -MAX_T_VAL;
            }
        }
        return;
    }

    for (auto elemId = tid + blockLane * BLOCK_SIZE_; elemId < vocabSize; elemId += BLOCK_SIZE_ * BLOCKS_PER_BEAM_)
    {
        auto localIndex = elemId + tmpLogBufIndex;
        tmpLogProbs[localIndex] = logProbsSlot[elemId];
    }

    for (SizeType32 ite = 0; ite < k; ite++)
    {
        partial.init();
#pragma unroll
        for (auto elemId = tid + blockLane * BLOCK_SIZE_; elemId < vocabSize; elemId += BLOCK_SIZE_ * BLOCKS_PER_BEAM_)
        {
            auto index = elemId + tmpLogBufIndex;
            partial.insert(tmpLogProbs[index], index);
        }

        TopK_2<T> total = BlockReduce(tempStorage).Reduce(partial, reduce_topk_op_2<T>);

        if (tid == 0)
        {
            auto const index = tmpTopKBufIndex + ite;
            topKTmpIdBuf[index] = total.p;
            topKTmpValBuf[index] = total.u;
            if (total.p >= 0)
            {
                tmpLogProbs[total.p] = -MAX_T_VAL;
            }
        }
        __syncthreads();
    }
}

template <typename T, int BLOCK_SIZE_, int BLOCKS_PER_BEAM_>
__global__ void topKStage2Sampling(SizeType32 const* __restrict topKTmpIdBuf, T* topKTmpValBuf, TokenIdType** idsPtrs,
    TokenIdType* ids, SizeType32* sequenceLengths, FinishedState const* finishedInput, FinishedState* finishedOutput,
    float* cumLogProbs, float* outputLogProbs, SizeType32 maxTopK, SizeType32 const* topKs, float topP,
    float const* topPs, hiprandState_t* hiprandState, TokenIdType const* endIds, SizeType32 vocabSize,
    bool const* skipDecode, SizeType32 const* batchSlots, SizeType32 maxBatchSize, bool normalizeLogProbs,
    bool logitHasProbs, SizeType32 const* tokensPerStep, SizeType32 maxTokensPerStep, SizeType32 maxSeqLen,
    bool returnAllTopK)
{
    bool const IS_FP16 = std::is_same<T, half>::value;
    T const MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

    auto const tid = static_cast<SizeType32>(threadIdx.x);
    auto const batchIdx = static_cast<SizeType32>(blockIdx.x);
    auto const tokenIdx = static_cast<SizeType32>(blockIdx.y);
    auto const batchSlot = batchSlots[batchIdx];
    FinishedState const finishState = finishedInput != nullptr ? finishedInput[batchSlot] : FinishedState::empty();
    if ((skipDecode != nullptr && skipDecode[batchSlot]) || (finishState.isSkipDecoding()))
    {
        return;
    }
    if (tokensPerStep != nullptr && tokenIdx >= tokensPerStep[batchSlot])
    {
        return;
    }

    auto const k = (topKs != nullptr) ? topKs[batchSlot] : maxTopK;
    auto const probThreshold = (topPs != nullptr) ? topPs[batchSlot] : topP;
    auto const size = k * BLOCKS_PER_BEAM_;
    auto const stride = maxTopK * BLOCKS_PER_BEAM_;

    typedef hipcub::BlockReduce<TopK_2<float>, BLOCK_SIZE_> BlockReduce;
    __shared__ typename BlockReduce::TempStorage tempStorage;
    extern __shared__ char array[];
    __shared__ float sSum;
    T* sVal = topKTmpValBuf + (batchIdx * maxTokensPerStep + tokenIdx) * stride;
    auto* sId = reinterpret_cast<SizeType32*>(array);
    if (tid == 0)
    {
        sSum = 0.0f;
    }
    TopK_2<float> partial;

    if (finishState.isFinished())
    {
        if (finishedOutput != nullptr)
        {
            finishedOutput[batchSlot] = finishState;
        }
        return;
    }

    auto sVal2 = reinterpret_cast<float*>(sId + k);
    float maxLogit;
    for (SizeType32 ite = 0; ite < k; ite++)
    {
        partial.init();
#pragma unroll
        for (SizeType32 i = tid; i < size; i += BLOCK_SIZE_)
        {
            partial.insert((float) sVal[i], i);
        }

        TopK_2<float> total = BlockReduce(tempStorage).Reduce(partial, reduce_topk_op_2<float>);

        if (tid == 0)
        {
            if (ite == 0)
            {
                maxLogit = total.u;
            }
            sId[ite] = total.p;
            sVal[total.p] = -MAX_T_VAL;

            // when cumLogProbs are computed, topKTmpValBuf (logits_buf_) are
            // already pre-processed by softmax_kernel
            if (!logitHasProbs)
            {
                total.u = __expf(total.u - maxLogit);
            }
            sVal2[ite] = total.u;
            sSum += total.u;
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        auto randNum = static_cast<float>(hiprand_uniform(hiprandState + batchSlot) * probThreshold * sSum);
        auto* outputIdsRequestPtr = idsPtrs == nullptr ? ids + batchSlot * maxSeqLen : idsPtrs[batchSlot];
        for (SizeType32 ki = 0; ki < k; ki++)
        {
            auto expLogit = sVal2[ki];
            randNum = randNum - expLogit;
            if (randNum <= 0.0f || ki == k - 1 || returnAllTopK)
            {
                auto idx = sId[ki];
                // If sId is -1 here we force output token to the last from vocabulary to get vivid indicator of smth
                // going wrong for the debug
                auto outputId = idx != -1
                    ? topKTmpIdBuf[(batchIdx * maxTokensPerStep + tokenIdx) * stride + idx] % vocabSize
                    : vocabSize - 1;
                auto const curSeqLen = sequenceLengths == nullptr ? 0 : sequenceLengths[batchSlot];
                auto const outIdx = returnAllTopK ? tokenIdx * maxTopK + ki : curSeqLen + tokenIdx;
                outputIdsRequestPtr[outIdx] = outputId;
                // cum log prob is not supported with returnAllTopK
                if (!returnAllTopK)
                {
                    if (cumLogProbs != nullptr || outputLogProbs != nullptr)
                    {
                        auto logProb = logf(expLogit);
                        if (cumLogProbs != nullptr)
                        {
                            cumLogProbs[batchSlot] += logProb;
                        }
                        if (outputLogProbs != nullptr)
                        {
                            // 'outputLogProbs' is the probability induced by the top-k sampling:
                            // NOT normalized (same way as OpenAI does):
                            // log_prob = log P(i | i is in vocab) = log(expLogit)
                            // normalized:
                            // log_prob = log P(i | i is in top-k) = log(expLogit / sum)
                            outputLogProbs[curSeqLen * maxBatchSize + batchSlot]
                                = normalizeLogProbs ? logProb - logf(sSum) : logProb;
                        }
                    }
                    break;
                }
            }
        }
        if (maxTokensPerStep == 1 && !returnAllTopK && sequenceLengths != nullptr && finishedOutput != nullptr
            && endIds != nullptr)
        {
            auto const seqLen = sequenceLengths[batchSlot];
            if (outputIdsRequestPtr[seqLen] == endIds[batchSlot])
            {
                finishedOutput[batchSlot].setFinishedEOS();
                // Do not increase seq len when EOS is generated. Seq len should always contain only tokens to be
                // outputted
            }
            else
            {
                // We don't need to set output finished state as it is assumed to be in non finished state
                sequenceLengths[batchSlot] += 1;
            }
        }
    }
}

#define CASE_K(K_MAX, BLOCK_SIZE_1_, BLOCK_SIZE_2_, BLOCKS_PER_BEAM_)                                                  \
    do                                                                                                                 \
    {                                                                                                                  \
        {                                                                                                              \
            dim3 grid(params.batchSize* BLOCKS_PER_BEAM_, params.maxTokensPerStep);                                    \
            dim3 block(BLOCK_SIZE_1_);                                                                                 \
            topKStage1<T, BLOCK_SIZE_1_, BLOCKS_PER_BEAM_><<<grid, block, 0, stream>>>(params.logProbs,                \
                params.logProbsPtrs, tempLogProbs, topKTmpIdBuf, topKTmpValBuf, params.finishedInput, params.maxTopK,  \
                params.topKs, params.vocabSizePadded, params.endIds, params.skipDecode, params.batchSlots,             \
                params.tokensPerStep, params.maxTokensPerStep);                                                        \
        }                                                                                                              \
        {                                                                                                              \
            dim3 grid(params.batchSize, params.maxTokensPerStep);                                                      \
            dim3 block(BLOCK_SIZE_2_);                                                                                 \
            topKStage2Sampling<T, BLOCK_SIZE_2_, BLOCKS_PER_BEAM_>                                                     \
                <<<grid, block, K_MAX * sizeof(SizeType32) + K_MAX * sizeof(float), stream>>>(topKTmpIdBuf,            \
                    topKTmpValBuf, params.outputIdsPtrs, params.outputIds, params.sequenceLengths,                     \
                    params.finishedInput, params.finishedOutput, params.cumLogProbs, params.outputLogProbs,            \
                    params.maxTopK, params.topKs, params.maxTopP, params.topPs, params.hiprandState, params.endIds,     \
                    params.vocabSizePadded, params.skipDecode, params.batchSlots, params.maxBatchSize,                 \
                    params.normalizeLogProbs, params.logitsHasProbs, params.tokensPerStep, params.maxTokensPerStep,    \
                    params.maxSeqLen, params.returnAllTopK);                                                           \
        }                                                                                                              \
    } while (0)

template <typename T>
void invokeBatchTopKSampling(TopKSamplingKernelParams<T> const& params, hipStream_t stream)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    params.checkParams();

    // Not allow an ambiguous inputs topP and topPs.
    auto const workspaceSizes
        = getTopKWorkspaceSizes<T>(params.batchSize, params.maxTokensPerStep, params.maxTopK, params.vocabSizePadded);

    if (params.maxTopK == 0)
    {
        return;
    }

    std::vector<void*> alignedPointers;
    calcAlignedPointers(alignedPointers, params.workspace, workspaceSizes);

    auto tempLogProbs = static_cast<T*>(alignedPointers[0]);
    auto topKTmpIdBuf = static_cast<SizeType32*>(alignedPointers[1]);
    auto topKTmpValBuf = static_cast<T*>(alignedPointers[2]);

    SizeType32 logMaxTopK{0};
    SizeType32 recursor{params.maxTopK - 1};
    while (recursor >>= 1)
    {
        ++logMaxTopK;
    }

    switch (logMaxTopK)
    {
    case 0:
    case 1:
    case 2:
    case 3: // 0 < maxTopK <= 16
        CASE_K(16, 128, 128, 8);
        break;
    case 4: // 16 < maxTopK <= 32
        CASE_K(32, 256, 128, 8);
        break;
    case 5: // 32 < maxTopK <= 64
        CASE_K(64, 256, 256, 8);
        break;
    case 6:
    case 7:
    case 8:
    case 9: // 64 < maxTopK <= 1024
        CASE_K(1024, 256, 256, 8);
        break;
    default: TLLM_CHECK_WITH_INFO(false, "TopK kernel supports 1 <= k <= 1024 but got k=%d", params.maxTopK);
    }

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

#undef CASE_K

template void invokeBatchTopKSampling(TopKSamplingKernelParams<float> const& params, hipStream_t stream);

template void invokeBatchTopKSampling(TopKSamplingKernelParams<half> const& params, hipStream_t stream);

__global__ void setupTopKRuntimeArgs(SizeType32 batchSize, SizeType32 topK, SizeType32* topKs, SizeType32 topKsSize,
    float topP, float* topPs, SizeType32 topPsSize, bool* skipDecode, SizeType32 const* batchSlots)
{
    auto const index = static_cast<SizeType32>(blockIdx.x * blockDim.x + threadIdx.x);
    for (auto bi = index; bi < batchSize; bi += static_cast<SizeType32>(gridDim.x * blockDim.x))
    {
        auto const batchSlot = batchSlots[bi];
        auto k = topKsSize > 1 ? topKs[batchSlot] : topK;
        auto p = topPsSize > 1 ? topPs[batchSlot] : topP;

        if (k == 0 && p == 0.0f)
        {
            // TensorRT-LLM's topp implementation does not support topp = 0.0f, but it
            // equivalent to greedy search. So, we set the topk = 1 as an alternative
            // solution.
            k = 1;
        }
        if (k > 0 && p == 0.0f)
        {
            // This case corresponds to the old topk sampling, which is equivalent to
            // the old topk_topp sampling with topp=1.0f. TopKSamplingLayer and
            // TopKTopPSamplingLayer are now merged by TopKSamplingLayer. Thus, we
            // replace the case topk>0 and topp=0.0f by topk>0 and topp=1.0f for the
            // compatibility.
            p = 1.0f;
        }
        topKs[batchSlot] = k;
        topPs[batchSlot] = p;
        skipDecode[batchSlot] = k == 0;
    }
}

void invokeSetupTopKRuntimeArgs(SizeType32 batchSize, SizeType32 topK, SizeType32* runtimeTopKDevicePtr,
    SizeType32 runtimeTopKSize, float topP, float* runtimeTopPDevicePtr, SizeType32 runtimeTopPSize,
    bool* skipDecodeDevicePtr, SizeType32 const* batchSlotsDevicePtr, hipStream_t stream)
{
    dim3 block(std::min(static_cast<uint32_t>(batchSize), 256u));
    dim3 grid(divUp(static_cast<uint32_t>(batchSize), block.x));
    // support topK up to TOP_K_MAX.
    setupTopKRuntimeArgs<<<grid, block, 0, stream>>>(batchSize, topK, runtimeTopKDevicePtr, runtimeTopKSize, topP,
        runtimeTopPDevicePtr, runtimeTopPSize, skipDecodeDevicePtr, batchSlotsDevicePtr);
}

} // namespace tensorrt_llm::kernels
