#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaBf16Wrapper.h"
#include "tensorrt_llm/common/cudaFp8Utils.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/mathUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/decoderMaskedMultiheadAttentionUtils.h"
#include "tensorrt_llm/kernels/gptKernels.h"
#include <hipcub/hipcub.hpp>

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

// A stateful callback functor that maintains the running sum between consecutive scans.
struct BlockPrefixCallbackOp
{
    // Running prefix
    int mRunningTotal;

    // Constructor
    __device__ BlockPrefixCallbackOp(int runningTotal)
        : mRunningTotal(runningTotal)
    {
    }

    // Thread-0 is responsible for returning a value for seeding the block-wide scan.
    __device__ int operator()(int blockAggregate)
    {
        int oldPrefix = mRunningTotal;
        mRunningTotal += blockAggregate;
        return oldPrefix;
    }
};

// Given an array of sequence lengths, with batchSize elements, that kernel computes the exclusive
// prefix-sums of the sequence lengths. There are (batchSize+1) elements in seqOffsets.
//
// seqOffsets[ 0]        = 0
// seqOffsets[ii]        = seqLengths[0] + .. + seqLengths[ii-1],
// seqOffsets[batchSize] = seqLengths[0] + .. + seqLengths[batchSize-1]
//
// This kernel uses a single thread block of THREADS_PER_BLOCK threads.

// This kernel also computes the padding offsets: Given the index (idx) of a token in a ragged tensor,
// we need the index of the token in the corresponding tensor with padding. We compute an array
// of numTokens elements, called the paddingOffsets, such that the position in the padded tensor
// of the token "idx" in the ragged tensor is given by idx + paddingOffset[idx].
//
// That kernel uses a grid of batchSize blocks.

template <typename T, int THREADS_PER_BLOCK>
__global__ __launch_bounds__(THREADS_PER_BLOCK) void computeSeqAndPaddingOffsets(BuildDecoderInfoParams<T> params)
{
    // Dynamic shared memory for storing seqOffsets.
    extern __shared__ int smem[];
    int* smemSeqQOffsets = (int*) (smem);

    // Fixed Q sequence lengths.
    bool const fixed_q_seqlen = params.seqQLengths == nullptr;

    // Whether to calculate cumulative KV sequence lengths.
    bool const calculate_kv_offsets = params.seqKVOffsets != nullptr;

    // Whether to calculate cumulative packed mask rows.
    bool const calculate_packed_mask_row_offsets = params.packedMaskRowOffsets != nullptr;

    // Compute the padding offsets for Encoder Inputs.
    bool const need_encoder_padding_offsets = (params.encoderPaddingOffsets != nullptr) && calculate_kv_offsets;
    [[maybe_unused]] int* smemEncoderSeqQOffsets;

    // The implementation of the parallel scan in the thread block (see CUB for details).
    using BlockScan = hipcub::BlockScan<int, THREADS_PER_BLOCK>;

    // Allocate storage in shared memory to do the scan.
    __shared__ typename BlockScan::TempStorage tempQStorage;
    [[maybe_unused]] __shared__ typename BlockScan::TempStorage tempMaskStorage;
    [[maybe_unused]] __shared__ typename BlockScan::TempStorage tempKVStorage;

    // This prefixOp operator keeps a running sum for when we need multiple iterations of the loop.
    BlockPrefixCallbackOp prefixQOp(0);
    BlockPrefixCallbackOp prefixMaskOp(0);
    BlockPrefixCallbackOp prefixKVOp(0);

    if (need_encoder_padding_offsets)
    {
        smemEncoderSeqQOffsets = (int*) (&smemSeqQOffsets[params.batchSize + 1]);
    }

    // Iterate over the sequences in the batch.
    //
    // The loop index does not depend on the thread index to make sure all the threads enter the
    // loop as we have __syncthreads in it (and we need all threads to participate to avoid
    // deadlocks).
    // Only the last block computes the full sequence offsets.
    bool const storeSeqOffsets = blockIdx.x == (params.batchSize - 1);
    int const batchSizeBound = blockIdx.x + 1;
    for (int batchOffset = 0; batchOffset <= batchSizeBound; batchOffset += THREADS_PER_BLOCK)
    {
        // The index of the batch.
        int batchIdx = batchOffset + threadIdx.x;

        // Threads that correspond to valid sequences read the length.
        int seqQLength = 0;
        [[maybe_unused]] int packedMaskRows = 0;
        [[maybe_unused]] int seqKVLength = 0;
        if (batchIdx < batchSizeBound)
        {
            seqQLength = fixed_q_seqlen ? params.maxQSeqLength : params.seqQLengths[batchIdx];
            // Need to pad mask rows to multiple of 128 for each sequence in the batch.
            packedMaskRows = calculate_packed_mask_row_offsets
                ? divUp(seqQLength, int(FLASH_ATTEN_PACKED_MASK_M_ALIGNMENT)) * FLASH_ATTEN_PACKED_MASK_M_ALIGNMENT
                : 0;
            seqKVLength = calculate_kv_offsets ? params.seqKVLengths[batchIdx] : 0;
        }

        // Do the prefix-scan (it calls syncthreads internally).
        int seqQOffset;
        [[maybe_unused]] int packedMaskRowOffset;
        [[maybe_unused]] int seqKVOffset;
        BlockScan(tempQStorage).ExclusiveSum(seqQLength, seqQOffset, prefixQOp);
        if (calculate_packed_mask_row_offsets)
        {
            BlockScan(tempMaskStorage).ExclusiveSum(packedMaskRows, packedMaskRowOffset, prefixMaskOp);
        }
        if (calculate_kv_offsets)
        {
            BlockScan(tempKVStorage).ExclusiveSum(seqKVLength, seqKVOffset, prefixKVOp);
        }

        // Store the result to smem.
        if (batchIdx <= batchSizeBound)
        {
            smemSeqQOffsets[batchIdx] = seqQOffset;
            if (need_encoder_padding_offsets)
            {
                smemEncoderSeqQOffsets[batchIdx] = seqKVOffset;
            }
        }

        // Store the result.
        if (batchIdx <= batchSizeBound && storeSeqOffsets)
        {
            params.seqQOffsets[batchIdx] = seqQOffset;
            if (calculate_packed_mask_row_offsets)
            {
                params.packedMaskRowOffsets[batchIdx] = packedMaskRowOffset;
            }
            if (calculate_kv_offsets)
            {
                params.seqKVOffsets[batchIdx] = seqKVOffset;
            }
        }

        // Make sure the shared memory can be reused for the next iteration of the loop.
        __syncthreads();
    }

    int batchIdx = blockIdx.x;

    // Compute the padding offsets.
    auto compute_padding_offset = [&](int* smem_offset, int maxSeqLength, int* paddingOffsets)
    {
        // Block x dimension is the batch dimension, while threads iterate all tokens in the sequence.
        int seqBegin = smem_offset[batchIdx];
        // The offset to the 1st element of the next sequence.
        int seqEnd = smem_offset[batchIdx + 1];
        // The length of the sequence.
        int seqLength = seqEnd - seqBegin;
        // The number of padded tokens in the previous sequences.
        int paddingOffset = batchIdx * maxSeqLength - seqBegin;

        // Iterate over the tokens to update the number of padded elements.
        for (int tokenIdx = threadIdx.x; tokenIdx < seqLength; tokenIdx += blockDim.x)
        {
            paddingOffsets[seqBegin + tokenIdx] = paddingOffset;
        }
    };

    if (params.paddingOffsets != nullptr)
    {
        compute_padding_offset(smemSeqQOffsets, params.maxQSeqLength, params.paddingOffsets);
    }

    if (need_encoder_padding_offsets)
    {
        compute_padding_offset(smemEncoderSeqQOffsets, params.maxEncoderQSeqLength, params.encoderPaddingOffsets);
    }

    // Each block generates the rotary embedding inv_freq tensor for the corresponding sequence.
    int zid = 2 * threadIdx.x;
    int halfRotaryEmbeddingDim = params.rotaryEmbeddingDim / 2;
    if (params.rotaryEmbeddingDim > 0 && zid < params.rotaryEmbeddingDim)
    {
        mmha::update_rotary_base_n_scale(params.rotaryEmbeddingBase, params.rotaryEmbeddingScale,
            params.rotaryScalingType, params.rotaryEmbeddingDim, params.rotaryEmbeddingMaxPositions,
            params.seqKVLengths[batchIdx]);
        // Recompute the rotary scales when it is dynamic scaling.
        if (params.rotaryScalingType == RotaryScalingType::kDYNAMIC || params.rotaryEmbeddingInvFreqCache == nullptr)
        {
            float const invFreq = params.rotaryEmbeddingScale
                / powf(params.rotaryEmbeddingBase, zid / (float) params.rotaryEmbeddingDim);
            params.rotaryEmbeddingInvFreq[batchIdx * halfRotaryEmbeddingDim + threadIdx.x] = invFreq;
        }
        else
        {
            // Otherwise, expand the inv freq cache to batch size.
            float const invFreqCache = params.rotaryEmbeddingInvFreqCache[threadIdx.x];
            params.rotaryEmbeddingInvFreq[batchIdx * halfRotaryEmbeddingDim + threadIdx.x] = invFreqCache;
        }
    }

    // Perpare values for fmha.
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        // Reset fmha tile counter to 0 before launching fmha kernels.
        if (params.fmhaTileCounter)
        {
            params.fmhaTileCounter[0] = 0u;
        }
        // Take the quantization scales into consideration.
        if (params.fmhaBmm1Scale)
        {
            // The scale after fmha bmm1.
            params.fmhaBmm1Scale[0] = params.dequantScaleQkv[0] * params.dequantScaleQkv[0] * params.fmhaHostBmm1Scale;
            // The scale prepared for log2 optimization.
            constexpr float kLog2e = 1.4426950408889634074f;
            params.fmhaBmm1Scale[1] = params.fmhaBmm1Scale[0] * kLog2e;
        }
        if (params.fmhaBmm2Scale)
        {
            // The scale after fmha bmm2.
            params.fmhaBmm2Scale[0] = params.quantScaleO[0] * params.dequantScaleQkv[0];
        }
    }
}

// This kernel computes the attention mask. We must compute this on-the-fly in the future.

template <typename AttentionMaskDataType>
__global__ void computeAttentionMask(AttentionMaskDataType* attentionMask, int const* seqLengths, int maxQSeqLength,
    int attentionWindowSize, AttentionMaskType attentionMaskType, BlockSparseParams blockSparseParams)
{
    // The index of the sequence in the batch.
    int batchIdx = blockIdx.y;

    // The number of items in the mask for each sequence.
    int maskSize = maxQSeqLength * maxQSeqLength;
    // The offset to the 1st element of the mask for that particular sequence.
    int batchOffset = batchIdx * maskSize;

    // The length of the sequence.
    int seqLength = seqLengths[batchIdx];

    // Iterate over the tokens to update the number of padded elements.
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < maskSize; idx += gridDim.x * blockDim.x)
    {
        // The position in the matrix.
        int rowIdx = idx / maxQSeqLength;
        int colIdx = idx % maxQSeqLength;

        // Is it a valid token?
        bool isValid = true;
        switch (attentionMaskType)
        {
        case AttentionMaskType::PADDING:
            isValid = rowIdx < seqLength && colIdx < seqLength;
            // seq_length==4, max_seq_len==5
            // 1 1 1 1 0
            // 1 1 1 1 0
            // 1 1 1 1 0
            // 1 1 1 1 0
            // 0 0 0 0 0
            break;
        case AttentionMaskType::CAUSAL:
            isValid = rowIdx < seqLength && colIdx < seqLength && colIdx <= rowIdx;
            // Sliding_window_causal when there are not enough kv cache.
            isValid = isValid && colIdx >= max(0, rowIdx - attentionWindowSize);
            // seq_length==4, max_seq_len==5
            // 1 0 0 0 0
            // 1 1 0 0 0
            // 1 1 1 0 0
            // 1 1 1 1 0
            // 0 0 0 0 0

            // seq_length==6, max_seq_len==6, max_attention_window_size = 2
            // 1 0 0 0 0 0
            // 1 1 0 0 0 0
            // 1 1 1 0 0 0
            // 0 1 1 1 0 0
            // 0 0 1 1 1 0
            // 0 0 0 1 1 1
            break;
        case AttentionMaskType::BIDIRECTIONAL:
            // clang-format off
              isValid = (rowIdx <  seqLength - 1 && colIdx < seqLength - 1) ||
                        (rowIdx == seqLength - 1 && colIdx < seqLength);
            // clang-format on
            // seq_length==4, max_seq_len==5
            // 1 1 1 0 0
            // 1 1 1 0 0
            // 1 1 1 0 0
            // 1 1 1 1 0
            // 0 0 0 0 0
        case AttentionMaskType::BIDIRECTIONALGLM:
            // clang-format off
              isValid = (colIdx < seqLength - 1) ||
                        (rowIdx == seqLength - 1 && colIdx == seqLength - 1);
            // clang-format on
            // seq_length==4, max_seq_len==5
            // 1 1 1 1 0
            // 1 1 1 1 0
            // 1 1 1 1 0
            // 1 1 1 1 0
            // 1 1 1 1 1
            break;
        case AttentionMaskType::BLOCKSPARSE:
            isValid = blockSparseParams.computeMask(rowIdx, colIdx, seqLength, 1 /*num_heads*/, 0 /*head_id*/);
            break;
        }

        // Store the mask.
        attentionMask[batchOffset + idx] = isValid ? AttentionMaskDataType(1.f) : AttentionMaskDataType(0.f);
    }
}

template <typename T>
void invokeBuildDecoderInfo(BuildDecoderInfoParams<T> const& params, hipStream_t stream)
{
    // Compute the sequence and padding offsets.
    int const THREADS_PER_BLOCK = 256;
    TLLM_CHECK_WITH_INFO(params.rotaryEmbeddingDim / 2 <= 256 && params.rotaryEmbeddingDim % 2 == 0,
        "Rotary embedding dim is assumed to be smaller than 512 and multiple of 2.");
    TLLM_CHECK_WITH_INFO(
        !(params.seqKVLengths == nullptr && params.rotaryEmbeddingDim > 0), "KV sequence lengths buffer is invalid.");
    bool const need_encoder_padding_offsets
        = (params.encoderPaddingOffsets != nullptr) && (params.seqKVOffsets != nullptr);
    const size_t smem_size
        = (need_encoder_padding_offsets ? (params.batchSize + 1) * 2 : (params.batchSize + 1)) * sizeof(int);
    computeSeqAndPaddingOffsets<T, THREADS_PER_BLOCK>
        <<<params.batchSize, THREADS_PER_BLOCK, smem_size, stream>>>(params);

    // Compute the attention mask, if needed.
    if (params.attentionMask != nullptr)
    {
        TLLM_CHECK_WITH_INFO(params.seqQLengths != nullptr, "Q sequence lengths buffer is invalid.");
        int const MIN_BLOCKS = 512;
        int blocksPerSeq = 16;
        while (blocksPerSeq * params.batchSize < MIN_BLOCKS)
        {
            blocksPerSeq *= 2;
        }
        dim3 grid(blocksPerSeq, params.batchSize);
        computeAttentionMask<<<grid, THREADS_PER_BLOCK, 0, stream>>>(params.attentionMask, params.seqQLengths,
            params.maxQSeqLength, params.attentionWindowSize, params.attentionMaskType, params.blockSparseParams);
    }
}

template void invokeBuildDecoderInfo(BuildDecoderInfoParams<float> const&, hipStream_t);
template void invokeBuildDecoderInfo(BuildDecoderInfoParams<half> const&, hipStream_t);
#ifdef ENABLE_BF16
template void invokeBuildDecoderInfo(BuildDecoderInfoParams<__hip_bfloat16> const&, hipStream_t);
#endif
#ifdef ENABLE_FP8
template void invokeBuildDecoderInfo(BuildDecoderInfoParams<__hip_fp8_e4m3_fnuz> const&, hipStream_t);
#endif

__global__ void updatePaddingCountKernel(int* paddingPerSeq, int const* seqLengths, int maxQSeqLength, int batchSize)
{

    for (int ii = threadIdx.x; ii < batchSize; ii += blockDim.x)
    {
        paddingPerSeq[ii] = maxQSeqLength - seqLengths[ii];
    }
}

} // namespace kernels
} // namespace tensorrt_llm
