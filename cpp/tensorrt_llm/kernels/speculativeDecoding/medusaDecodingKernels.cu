#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/speculativeDecoding/medusaDecodingKernels.h"
#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm::kernels::speculative_decoding
{
namespace
{
__device__ __forceinline__ int4 reduceMaxInt4(int4 const& a, int4 const& b)
{
    return a.x >= b.x ? a : b;
}

template <typename T, SizeType32 BLOCK_SIZE>
__global__ void acceptDraftTokensByIdsWithPaths(TokenIdType* outputIds, TokenIdType const* draftIds,
    TokenIdType const* targetIds, SizeType32* sequenceLengths, SizeType32* acceptedLengths,
    FinishedState* finishedFinal, SizeType32 const* batchSlots, SizeType32 const* paths, TokenIdType const* endIds,
    T const** medusaLogits, T const** logitsPtrs, SizeType32* curTokensPerStep, SizeType32 const* targetTokensPerStep,
    SizeType32* bestPathIds, SizeType32 batchSize, SizeType32 vocabSize, SizeType32 maxBatchSize, SizeType32 maxSeqLen,
    SizeType32 maxNumHeads, SizeType32 maxDecodingTokens)
{
    auto const batchIdx = static_cast<SizeType32>(blockIdx.x);
    auto const batchSlot = batchSlots[batchIdx];
    auto const inputLength = sequenceLengths[batchSlot];
    auto const endId = endIds[batchSlot];
    auto const numTokensPerStep = curTokensPerStep[batchSlot];
    auto const maxNumDraftTokens = maxNumHeads + 1;

    int4 partialMax{-1, -1, 0, 0};
    // Go over different paths and construct implicit sequences
    for (auto pathIdx = static_cast<SizeType32>(threadIdx.x); pathIdx < maxDecodingTokens;
         pathIdx += static_cast<SizeType32>(blockDim.x))
    {
        auto acceptedLength = maxNumDraftTokens;
        auto const pathOffset = flat_index3(batchSlot, pathIdx, 0, maxDecodingTokens, maxNumDraftTokens);
        bool hasEnd = false;

        auto const tokenId = paths[pathOffset];
        // Continue if path does not exist
        if (tokenId == -1)
        {
            continue;
        }
        auto const targetTokenIdx = batchSlot * maxDecodingTokens + tokenId;
        auto targetToken = targetIds[targetTokenIdx];
        auto nextIdx = tokenId;

        // Go along the path
        for (SizeType32 ti = 1; ti < maxNumDraftTokens; ++ti)
        {
            auto const tokenId = paths[pathOffset + ti];
            // Break if path terminates
            if (tokenId == -1)
            {
                hasEnd = targetToken == endId; // check if last token is EOS when path terminates.
                acceptedLength = hasEnd ? ti - 1 : ti;
                break;
            }
            auto const targetTokenIdx = batchSlot * maxDecodingTokens + tokenId;
            auto const draftTokenIdx = batchSlot * (maxDecodingTokens - 1) + tokenId - 1;
            // In context phase, no draft tokens are given. Set draft token to -1 to get guaranteed rejection
            auto const draftToken = tokenId >= numTokensPerStep ? -1 : draftIds[draftTokenIdx];
            // Check if draft tokens are the same as target tokens
            bool const accepted = draftToken == targetToken;
            hasEnd = targetToken == endId;
            if (!accepted || hasEnd)
            {
                acceptedLength = hasEnd ? ti - 1 : ti;
                break;
            }
            targetToken = targetIds[targetTokenIdx];
            nextIdx = tokenId;
        }
        // Get longest path of the thread
        if (partialMax.x < acceptedLength)
        {
            partialMax.x = acceptedLength;
            partialMax.y = pathIdx;
            partialMax.z = hasEnd;
            partialMax.w = nextIdx;
        }
    }

    // Get the longest path of the block (request)
    typedef hipcub::BlockReduce<int4, BLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage tempStorage;
    int4 total = BlockReduce(tempStorage).Reduce(partialMax, reduceMaxInt4);

    __shared__ int4 totalShared;
    if (threadIdx.x == 0)
    {
        totalShared = total;
    }

    __syncthreads();

    auto const acceptedLength = totalShared.x;
    auto const bestPathIdx = totalShared.y;
    auto const bestNextIdx = numTokensPerStep == 1 ? 0 : totalShared.w;
    auto const pathOffset = flat_index3(batchSlot, bestPathIdx, 0, maxDecodingTokens, maxNumDraftTokens);
    for (auto ti = static_cast<SizeType32>(threadIdx.x); ti < acceptedLength; ti += static_cast<SizeType32>(blockDim.x))
    {
        auto const tokenId = paths[pathOffset + ti];
        auto const targetSrcTokenIdx = batchSlot * maxDecodingTokens + tokenId;
        auto const outputTokenIdx = batchSlot * maxSeqLen + inputLength + ti;
        auto const targetToken = targetIds[targetSrcTokenIdx];
        // Copy accepted tokens to the sequence with draft tokens (outputIds === outputIds)
        outputIds[outputTokenIdx] = targetToken;
    }

    // Leading thread reconstructs winning path and sets new data
    if (threadIdx.x == 0)
    {
        auto const hasEnd = totalShared.z;
        // Set end condition
        if (hasEnd)
        {
            finishedFinal[batchSlot].setFinishedEOS();
        }
        // Make correction to the sequence length
        sequenceLengths[batchSlot] += acceptedLength;
        acceptedLengths[batchSlot] = acceptedLength;
        // In Medusa decoding step, number of draft tokens is 0 and must be updated for the next steps
        if (numTokensPerStep == 1)
        {
            curTokensPerStep[batchSlot] = targetTokensPerStep[batchSlot];
        }
        bestPathIds[batchSlot] = bestPathIdx;
    }

    // Prepare logits pointers to respective logits from Medusa Heads for the all-top-K sampling kernel
    for (auto hi = static_cast<SizeType32>(threadIdx.x); hi < maxNumHeads; hi += static_cast<SizeType32>(blockDim.x))
    {
        logitsPtrs[batchIdx * maxNumHeads + hi]
            = medusaLogits[batchSlot * maxNumHeads + hi] + flat_index2(bestNextIdx, 0, vocabSize);
    }
}
} // namespace

template <typename T>
void acceptDraftTokensByIdsWithPaths(TokenIdType* outputIds, TokenIdType const* draftIds, TokenIdType const* targetIds,
    SizeType32* sequenceLengths, SizeType32* acceptedLengths, FinishedState* finishedFinal,
    SizeType32 const* batchSlots, SizeType32 const* paths, TokenIdType const* endIds, T const** medusaLogits,
    T const** logitsPtrs, SizeType32* curTokensPerStep, SizeType32 const* targetTokensPerStep, SizeType32* bestPathIds,
    SizeType32 batchSize, SizeType32 vocabSize, SizeType32 maxBatchSize, SizeType32 maxSeqLen, SizeType32 maxNumHeads,
    SizeType32 maxDecodingTokens, hipStream_t stream)
{
    constexpr SizeType32 BLOCK_SIZE = 256;
    dim3 block(BLOCK_SIZE);
    dim3 grid(batchSize);
    acceptDraftTokensByIdsWithPaths<T, BLOCK_SIZE><<<grid, block, 0, stream>>>(outputIds, draftIds, targetIds,
        sequenceLengths, acceptedLengths, finishedFinal, batchSlots, paths, endIds, medusaLogits, logitsPtrs,
        curTokensPerStep, targetTokensPerStep, bestPathIds, batchSize, vocabSize, maxBatchSize, maxSeqLen, maxNumHeads,
        maxDecodingTokens);
}

template void acceptDraftTokensByIdsWithPaths(TokenIdType* outputIds, TokenIdType const* draftIds,
    TokenIdType const* targetIds, SizeType32* sequenceLengths, SizeType32* acceptedLengths,
    FinishedState* finishedFinal, SizeType32 const* batchSlots, SizeType32 const* paths, TokenIdType const* endIds,
    float const** medusaLogits, float const** logitsPtrs, SizeType32* curTokensPerStep,
    SizeType32 const* targetTokensPerStep, SizeType32* bestPathIds, SizeType32 batchSize, SizeType32 vocabSize,
    SizeType32 maxBatchSize, SizeType32 maxSeqLen, SizeType32 maxNumHeads, SizeType32 maxDecodingTokens,
    hipStream_t stream);
template void acceptDraftTokensByIdsWithPaths(TokenIdType* outputIds, TokenIdType const* draftIds,
    TokenIdType const* targetIds, SizeType32* sequenceLengths, SizeType32* acceptedLengths,
    FinishedState* finishedFinal, SizeType32 const* batchSlots, SizeType32 const* paths, TokenIdType const* endIds,
    half const** medusaLogits, half const** logitsPtrs, SizeType32* curTokensPerStep,
    SizeType32 const* targetTokensPerStep, SizeType32* bestPathIds, SizeType32 batchSize, SizeType32 vocabSize,
    SizeType32 maxBatchSize, SizeType32 maxSeqLen, SizeType32 maxNumHeads, SizeType32 maxDecodingTokens,
    hipStream_t stream);

namespace
{
__global__ void scatterMedusaDraftTokens(TokenIdType* treeDraftIds, TokenIdType const* sourceDraftIds,
    SizeType32 const* treeIds, SizeType32 const* tokensPerStepData, SizeType32 const* batchSlots,
    SizeType32 maxDecodingTokens)
{
    auto const batchIdx = static_cast<SizeType32>(blockIdx.x);
    auto const batchSlot = batchSlots[batchIdx];
    auto const tokensPerStep = tokensPerStepData[batchSlot];
    auto const maxDecodingDraftTokens = maxDecodingTokens - 1;
    for (auto index = static_cast<SizeType32>(threadIdx.x); index < tokensPerStep - 1;
         index += static_cast<SizeType32>(blockDim.x))
    {
        auto const indexInTree = treeIds[batchSlot * maxDecodingDraftTokens + index];
        auto const treeDraftIdx = batchSlot * maxDecodingDraftTokens + index;
        auto const sourceDraftIdx = batchSlot * maxDecodingTokens + indexInTree;
        treeDraftIds[treeDraftIdx] = sourceDraftIds[sourceDraftIdx];
    }
}
} // namespace

void scatterMedusaDraftTokens(TokenIdType* treeDraftIds, TokenIdType const* sourceDraftIds, SizeType32 const* treeIds,
    SizeType32 const* tokensPerStep, SizeType32 const* batchSlots, SizeType32 maxDecodingTokens, SizeType32 batchSize,
    hipStream_t stream)
{
    constexpr SizeType32 BLOCK_SIZE = 256;
    scatterMedusaDraftTokens<<<batchSize, BLOCK_SIZE, 0, stream>>>(
        treeDraftIds, sourceDraftIds, treeIds, tokensPerStep, batchSlots, maxDecodingTokens);
}
} // namespace tensorrt_llm::kernels::speculative_decoding
