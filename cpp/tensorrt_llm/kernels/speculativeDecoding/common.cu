#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/speculativeDecoding/common.h"
#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm::kernels::speculative_decoding
{
template <int32_t BLOCK_SIZE>
__global__ void packAcceptedPaths(SizeType32* acceptedLengthsCumSum, SizeType32* pathsOffsets,
    SizeType32 const* acceptedLengths, SizeType32 const* bestPathIds, SizeType32 const* paths,
    SizeType32 const* batchSlots, SizeType32 batchSize, SizeType32 numPaths, SizeType32 maxPathLen,
    bool isPathsLinearBatchIdx)
{
    // Specialize BlockScan for a 1D block of 128 threads of type int
    typedef hipcub::BlockScan<SizeType32, BLOCK_SIZE> BlockScan;

    // Allocate shared memory for BlockScan
    __shared__ typename BlockScan::TempStorage tempStorage;
    auto const batchSizeRounded = ((batchSize + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;
    __shared__ SizeType32 currentCumSum;
    if (threadIdx.x == 0)
    {
        currentCumSum = 0;
    }

    __syncthreads();

    for (auto bi = static_cast<SizeType32>(threadIdx.x); bi < batchSizeRounded;
         bi += static_cast<SizeType32>(blockDim.x))
    {
        auto const valid = bi < batchSize;
        auto const batchSlot = valid ? batchSlots[bi] : 0;
        auto const acceptedLen = valid ? acceptedLengths[batchSlot] - 1 : 0;
        SizeType32 cumSum;
        BlockScan(tempStorage).ExclusiveSum(acceptedLen + currentCumSum, cumSum);
        if (threadIdx.x == blockDim.x - 1)
        {
            currentCumSum = cumSum;
        }
        __syncthreads();

        if (valid)
        {
            acceptedLengthsCumSum[bi] = cumSum;
            auto const pathBatchIdx = isPathsLinearBatchIdx ? bi : batchSlot;
            auto const bestPathIdx = bestPathIds[pathBatchIdx];
            auto const pathIdx = flat_index3(pathBatchIdx, bestPathIdx, 0, numPaths, maxPathLen);
            for (SizeType32 ti = 0; ti < acceptedLen; ++ti)
            {
                pathsOffsets[cumSum + ti] = paths[pathIdx + ti + 1] - 1;
            }
        }
    }
    if (threadIdx.x == 0)
    {
        acceptedLengthsCumSum[batchSize] = currentCumSum;
    }
}

void invokePackAcceptedPaths(SizeType32* acceptedLengthsCumSum, SizeType32* pathsOffsets,
    SizeType32 const* acceptedLengths, SizeType32 const* bestPathIds, SizeType32 const* paths,
    SizeType32 const* batchSlots, SizeType32 batchSize, SizeType32 numPaths, SizeType32 maxPathLen,
    bool isPathsLinearBatchIdx, hipStream_t stream)
{
    constexpr SizeType32 BLOCK_SIZE = 1024;
    packAcceptedPaths<BLOCK_SIZE><<<1, BLOCK_SIZE, 0, stream>>>(acceptedLengthsCumSum, pathsOffsets, acceptedLengths,
        bestPathIds, paths, batchSlots, batchSize, numPaths, maxPathLen, isPathsLinearBatchIdx);
}
} // namespace tensorrt_llm::kernels::speculative_decoding
