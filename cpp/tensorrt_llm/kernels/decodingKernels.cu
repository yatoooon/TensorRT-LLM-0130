#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/decodingKernels.h"
#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm
{

namespace kernels
{

class CopyBeamHypothesesStruct
{
public:
    TokenIdType const* srcOutputIdsCBA; // [BS, BM*2, MSL]
    TokenIdType* dstOutputIdsCBA;       // [BS, BM*2, MSL]
    SizeType32 outputIdsNumElts;

    float const* srcLogProbsCBA; // [BS, BM*2, MSL]
    float* dstLogProbsCBA;       // [BS, BM*2, MSL]
    SizeType32 logProbsNumElts;

    SizeType32 const* srcSequenceLengthsCBA; // [BS, BM*2]
    SizeType32* dstSequenceLengthsCBA;       // [BS, BM*2]
    SizeType32 sequenceLengthsNumElts;

    float const* srcCumLogProbsCBA; // [BS, BM*2]
    float* dstCumLogProbsCBA;       // [BS, BM*2]
    SizeType32 cumLogProbsCBANumElts;

    float const* srcNormedScoresCBA; // [BS, BM*2]
    float* dstNormedScoresCBA;       // [BS, BM*2]
    SizeType32 normedScoresNumElts;

    SizeType32 const* srcNumBeamsCBA; // [BS]
    SizeType32* dstNumBeamsCBA;       // [BS]
    SizeType32 numBeamsNumElts;

    float const* srcMinNormedScoresCBA; // [BS]
    float* dstMinNormedScoresCBA;       // [BS]
    SizeType32 minNormedScoresNumElts;

    bool const* srcBatchDones; // [BS]
    bool* dstBatchDones;       // [BS]
    SizeType32 batchDonesNumElts;

    float const* srcCumLogProbs; // [BS, BM]
    float* dstCumLogProbs;       // [BS, BM]
    SizeType32 cumLogProbsNumElts;
};

__global__ void gatherTree(gatherTreeParam param)
{
    for (int batchbeamIdx = blockIdx.x * blockDim.x + threadIdx.x; batchbeamIdx < param.batchSize * param.beamWidth;
         batchbeamIdx += gridDim.x * blockDim.x)
    {
        int const batch = batchbeamIdx / param.beamWidth;
        int const beam = batchbeamIdx % param.beamWidth;
        int const inputLen = param.inputLengths == nullptr ? 0 : param.inputLengths[batchbeamIdx];

        int const* parentIds = param.parentIds;
        int const* stepIds = param.stepIds;

        // TODO optimize the reduce_max operation for large beamWidth
        int maxLen = -1;
        bool updateResponseInputLength = param.responseInputLengths != nullptr;
        // int selected_beam_index = 0;
        for (int beamIdx = 0; beamIdx < param.beamWidth; beamIdx++)
        {
            int tmpLen
                = param.sequenceLengths[batch * param.beamWidth + beamIdx] + param.maxSequenceLengthFinalStep - 1;
            param.sequenceLengths[batch * param.beamWidth + beamIdx] = tmpLen;
            if (updateResponseInputLength)
            {
                param.responseInputLengths[batch * param.beamWidth + beamIdx] = inputLen;
            }
            if (tmpLen > maxLen)
            {
                maxLen = tmpLen;
            }
        }
        int const maxSeqLenB = min(param.maxSeqLen, maxLen);
        if (maxSeqLenB <= 0)
        {
            continue;
        }

        int const initialTgtIx = batch * param.beamWidth * param.maxSeqLen + beam * param.maxSeqLen + maxSeqLenB - 1;
        int const initialParentIx = batch * param.beamWidth * param.maxSeqLen + beam * param.maxSeqLen + maxSeqLenB - 1;
        param.outputIds[initialTgtIx] = __ldg(stepIds + initialParentIx);
        int parent = parentIds == nullptr ? 0 : __ldg(parentIds + initialParentIx) % param.beamWidth;
        bool foundBad = false;

        for (int level = maxSeqLenB - 2; level >= 0; --level)
        {
            int const levelBeamIx = batch * param.beamWidth * param.maxSeqLen + beam * param.maxSeqLen + level;
            int const levelParentIx = batch * param.beamWidth * param.maxSeqLen + parent * param.maxSeqLen + level;
            if (parent < 0 || parent > param.beamWidth)
            {
                param.outputIds[levelBeamIx] = param.endTokens[batch];
                parent = -1;
                foundBad = true;
            }
            else
            {
                param.outputIds[levelBeamIx] = __ldg(stepIds + levelParentIx);
                parent = parentIds == nullptr ? 0 : __ldg(parentIds + levelParentIx) % param.beamWidth;
            }
        }
        // set the padded part as end_token
        // inputLen
        for (int index = maxLen; index < param.maxSeqLen; ++index)
        {
            param.outputIds[batch * param.beamWidth * param.maxSeqLen + beam * param.maxSeqLen + index]
                = param.endTokens[batch];
        }

        // Not necessary when using a BeamSearchDecoder, but necessary
        // when a user feeds in possibly broken trajectory (i.e., non-eos
        // entries in a beam following eos entries).
        if (!foundBad)
        {
            bool finished = false;
            // skip the step 0 because it is often the start token
            int startStep = 1;
            for (int time = startStep; time < maxSeqLenB; ++time)
            {
                int const levelBeamIx = batch * param.beamWidth * param.maxSeqLen + beam * param.maxSeqLen + time;
                if (finished)
                {
                    param.outputIds[levelBeamIx] = param.endTokens[batch];
                }
                else if (param.outputIds[levelBeamIx] == param.endTokens[batch])
                {
                    finished = true;
                }
            }
        }
    }
}

struct RankNorm
{
    int rank;
    float norm;
};

inline __device__ RankNorm swap(RankNorm const& rankNorm, int mask, int dir)
{
    // Exchange RankNorm data inside the warp
    RankNorm other;
    other.rank = __shfl_xor_sync(unsigned(-1), rankNorm.rank, mask);
    other.norm = __shfl_xor_sync(unsigned(-1), rankNorm.norm, mask);
    // dir == 0 -> return larger one
    // dir == 1 -> return smaller one
    bool doSwap = (rankNorm.norm != other.norm) && ((rankNorm.norm > other.norm) == dir);
    return doSwap ? other : rankNorm;
}

inline __device__ uint32_t bfe(uint32_t a, uint32_t start, uint32_t len = 1)
{
    uint32_t d;
    asm volatile("bfe.u32 %0, %1, %2, %3;" : "=r"(d) : "r"(a), "r"(start), "r"(len));
    return d;
}

__global__ void finalized(gatherTreeParam param)
{
    int const beamIdx = static_cast<int>(threadIdx.x);
    int const beamWidth{param.beamWidth};

    extern __shared__ char array[];
    int* sRank = (int*) (array);
    int* sLength = (int*) (sRank + beamWidth);
    float* sScores = (float*) (sLength + beamWidth);
    float* sNormedScores = (float*) (sScores + beamWidth);
    int* sIds = (int*) (sNormedScores + beamWidth);

    if (beamIdx < beamWidth)
    {
        int const idx = blockIdx.x * param.beamWidth + beamIdx;
        int const numGeneratedToken{param.sequenceLengths[idx] - param.inputLengths[idx]};
        sNormedScores[beamIdx] = applyLengthPenalty(param.cumLogProbs[idx], numGeneratedToken, param.lengthPenalty);
        sLength[beamIdx] = param.sequenceLengths[idx];
        sScores[beamIdx] = param.cumLogProbs[idx];
    }
    for (int idx = beamIdx; idx < beamWidth * param.maxSeqLen; idx += blockDim.x)
    {
        sIds[idx] = param.outputIds[blockIdx.x * param.beamWidth * param.maxSeqLen + idx];
    }
    __syncthreads();

    RankNorm rankNorm;
    rankNorm.rank = beamIdx;
    rankNorm.norm = beamIdx < beamWidth ? sNormedScores[beamIdx] : -FLT_MAX;

    if (beamWidth < 32)
    {
        int warpid = threadIdx.x / 32;
        int laneid = threadIdx.x % 32;

        if (warpid == 0 && beamWidth > 1)
        {
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 1) ^ bfe(laneid, 0)); //  2
        }

        if (warpid == 0 && beamWidth > 2)
        {
            rankNorm = swap(rankNorm, 0x02, bfe(laneid, 2) ^ bfe(laneid, 1)); //  3~4
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 2) ^ bfe(laneid, 0));
        }

        if (warpid == 0 && beamWidth > 4)
        {
            rankNorm = swap(rankNorm, 0x04, bfe(laneid, 3) ^ bfe(laneid, 2)); //  5~8
            rankNorm = swap(rankNorm, 0x02, bfe(laneid, 3) ^ bfe(laneid, 1));
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 3) ^ bfe(laneid, 0));
        }

        if (warpid == 0 && beamWidth > 8)
        {
            rankNorm = swap(rankNorm, 0x08, bfe(laneid, 4) ^ bfe(laneid, 3)); // 9~16
            rankNorm = swap(rankNorm, 0x04, bfe(laneid, 4) ^ bfe(laneid, 2));
            rankNorm = swap(rankNorm, 0x02, bfe(laneid, 4) ^ bfe(laneid, 1));
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 4) ^ bfe(laneid, 0));
        }

        if (warpid == 0 && beamWidth > 16)
        {
            rankNorm = swap(rankNorm, 0x10, bfe(laneid, 5) ^ bfe(laneid, 4)); // 17~32
            rankNorm = swap(rankNorm, 0x08, bfe(laneid, 5) ^ bfe(laneid, 3));
            rankNorm = swap(rankNorm, 0x04, bfe(laneid, 5) ^ bfe(laneid, 2));
            rankNorm = swap(rankNorm, 0x02, bfe(laneid, 5) ^ bfe(laneid, 1));
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 5) ^ bfe(laneid, 0));
        }
    }
    else
    {
        // Not supported! We must have a check before calling that kernel.
    }

    if (beamIdx < beamWidth)
    {
        sRank[beamIdx] = rankNorm.rank;
    }

    __syncthreads();

    if (beamIdx < beamWidth)
    {
        auto srcIdx{rankNorm.rank};
        auto tgtIdx{blockIdx.x * param.beamWidth + beamIdx};
        param.sequenceLengths[tgtIdx] = sLength[srcIdx];
        param.cumLogProbs[tgtIdx] = sScores[srcIdx];
    }

    for (int beamIdx = 0; beamIdx < beamWidth; beamIdx++)
    {
        for (int i = threadIdx.x; i < sLength[sRank[beamIdx]]; i += blockDim.x)
        {
            param.outputIds[blockIdx.x * beamWidth * param.maxSeqLen + beamIdx * param.maxSeqLen + i]
                = sIds[sRank[beamIdx] * param.maxSeqLen + i];
        }
    }
}

void invokeGatherTree(gatherTreeParam param)
{
    int batchbeam = param.batchSize * param.beamWidth;
    dim3 grid(1), block(batchbeam);
    // though decoder do not support > 1024 for now
    if (batchbeam > 1024)
    {
        grid.x = ceil(param.batchSize * param.beamWidth / 1024.);
        block.x = 1024;
    }
    gatherTree<<<grid, block, 0, param.stream>>>(param);
    sync_check_cuda_error();

    if (param.beamWidth > 1)
    {
        TLLM_CHECK_WITH_INFO(param.beamWidth <= 32, "TRT-LLM does not support beam width > 32 now");
        // sort results by normalized cumLogProbs
        dim3 grid(param.batchSize);
        dim3 block(divUp(param.beamWidth, 32) * 32);

        auto shm_size = param.beamWidth * (sizeof(float) * 2 + sizeof(int) * 2 + sizeof(int) * param.maxSeqLen);
        finalized<<<grid, block, shm_size, param.stream>>>(param);
    }
}

__global__ void insertUnfinishedPathKernel(BeamHypotheses bh)
{
    // Move ALL unfinished beams from bh.outputIdsUnfinish to bh.outputIdsCBA
    // So here might be more than `nBM` beams in bh.outputIdsCBA after this kernel
    // Data movement:
    // bh.outputIdsUnfinish -> bh.outputIdsCBA
    // bh.sequenceLengths   -> bh.sequenceLengthsCBA
    // bh.cumLogProbs       -> bh.cumLogProbsCBA
    // bh.logProbsTiled     -> bh.logProbsCBA
    // update bh.normedScoresCBA
    // update bh.numBeamsCBA

    int const bid = blockIdx.x;       // Index of Batch
    int const nBM{bh.nBeamWidth};
    int const nMBS{bh.nMaxBatchSize}; // Only for bh.logProbsTiled
    int const nMSL{bh.nMaxSeqLen};
    bool const bOutputLogProbs{bh.logProbsCBA != nullptr && bh.logProbsTiled != nullptr};
    int const indexDstStart{bh.numBeamsCBA[bid]};

    if (bh.batchDones[bid])
    {
        return;
    }

    for (int i = 0; i < nBM; ++i)
    {
        int const srcBeam = bid * nBM + i;
        int const dstBeam = bid * nBM * 2 + i + indexDstStart;
        int const step = bh.sequenceLengths[srcBeam] - 1;

        // The last token
        int const srcId = srcBeam * nMSL + step;
        int const dstId = dstBeam * nMSL + step;
        bh.outputIdsCBA[dstId] = bh.outputIdsUnfinish[srcId];
        if (bOutputLogProbs)
        {
            bh.logProbsCBA[dstId] = bh.logProbsTiled[step * nMBS * nBM + srcBeam];
        }
        // Previous tokens
        int prevId = bh.parentIdsUnfinish[srcId];
        for (int j = step - 1; j >= 0; --j)
        {
            int const index = bid * nBM * nMSL + prevId * nMSL + j;
            bh.outputIdsCBA[dstBeam * nMSL + j] = bh.outputIdsUnfinish[index];
            prevId = bh.parentIdsUnfinish[index];
        }
        if (bOutputLogProbs)
        {
            prevId = bh.parentIdsUnfinish[srcId];
            for (int j = step - 1; j >= 0; --j)
            {
                int const index = bid * nBM * nMSL + prevId * nMSL + j;
                bh.logProbsCBA[dstBeam * nMSL + j] = bh.logProbsTiled[j * nMBS * nBM + bid * nBM + prevId];
                prevId = bh.parentIdsUnfinish[index];
            }
        }
        // Other parameters
        bh.sequenceLengthsCBA[dstBeam] = bh.sequenceLengths[srcBeam];
        bh.normedScoresCBA[dstBeam]
            = applyLengthPenalty(bh.cumLogProbs[srcBeam], step - bh.inputLengths[srcBeam] + 1, bh.lengthPenalties[bid]);
        bh.cumLogProbsCBA[dstBeam] = bh.cumLogProbs[srcBeam];
        bh.numBeamsCBA[bid]++;
    }
}

void invokeInsertUnfinishedPath(BeamHypotheses& bh, hipStream_t stream)
{
    insertUnfinishedPathKernel<<<bh.nBatchSize, 1, 0, stream>>>(bh);
}

__global__ void finalizeKernel(BeamHypotheses bh)
{
    // Do index sort on bh.normedScoresCBA, then move buffers from CBA to output by the order of index
    // Data movement:
    // bh.outputIdsCBA       -> bh.outputIds
    // bh.sequenceLengthsCBA -> bh.sequenceLengths
    // bh.cumLogProbsCBA     -> bh.cumLogProbs
    // bh.logProbsCBA        -> bh.logProbs

    int const bid = blockIdx.x;          // Index of Batch
    int const tid = threadIdx.x;         // Index of Beam
    int const nBM{bh.nBeamWidth};
    int const nCBA{bh.numBeamsCBA[bid]}; // count of candidate beams in CBA, nBM <= nCBA <= 2*nBM
    int const nMSL{bh.nMaxSeqLen};

    extern __shared__ char smem[];
    int* smemRank = (int*) (smem);                // [nBM]
    float* smemScore = (float*) (smemRank + nBM); // [2*nBM]
    int* smemSL = (int*) (smemScore + nBM * 2);   // [nBM]

    // Sort
    if (tid < nCBA)
    {
        smemScore[tid] = bh.normedScoresCBA[bid * nBM * 2 + tid];
    }
    __syncthreads();

    if (nCBA < 32)
    {
        int const warpid = tid / 32;
        int const laneid = tid % 32;
        RankNorm rankNorm{tid, tid < nCBA ? smemScore[tid] : -FLT_MAX};

        if (warpid == 0 && nCBA > 1)
        {
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 1) ^ bfe(laneid, 0)); //  2
        }

        if (warpid == 0 && nCBA > 2)
        {
            rankNorm = swap(rankNorm, 0x02, bfe(laneid, 2) ^ bfe(laneid, 1)); //  3~4
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 2) ^ bfe(laneid, 0));
        }

        if (warpid == 0 && nCBA > 4)
        {
            rankNorm = swap(rankNorm, 0x04, bfe(laneid, 3) ^ bfe(laneid, 2)); //  5~8
            rankNorm = swap(rankNorm, 0x02, bfe(laneid, 3) ^ bfe(laneid, 1));
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 3) ^ bfe(laneid, 0));
        }

        if (warpid == 0 && nCBA > 8)
        {
            rankNorm = swap(rankNorm, 0x08, bfe(laneid, 4) ^ bfe(laneid, 3)); // 9~16
            rankNorm = swap(rankNorm, 0x04, bfe(laneid, 4) ^ bfe(laneid, 2));
            rankNorm = swap(rankNorm, 0x02, bfe(laneid, 4) ^ bfe(laneid, 1));
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 4) ^ bfe(laneid, 0));
        }

        if (warpid == 0 && nCBA > 16)
        {
            rankNorm = swap(rankNorm, 0x10, bfe(laneid, 5) ^ bfe(laneid, 4)); // 17~32
            rankNorm = swap(rankNorm, 0x08, bfe(laneid, 5) ^ bfe(laneid, 3));
            rankNorm = swap(rankNorm, 0x04, bfe(laneid, 5) ^ bfe(laneid, 2));
            rankNorm = swap(rankNorm, 0x02, bfe(laneid, 5) ^ bfe(laneid, 1));
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 5) ^ bfe(laneid, 0));
        }

        if (tid < nBM)
        {
            smemRank[tid] = rankNorm.rank;
        }
        __syncthreads();
    }
    else
    {
        for (int i = 0; i < nBM; ++i)
        {
            float const score = tid < bh.numBeamsCBA[bid] ? smemScore[tid] : -FLT_MAX;
            float const maxScore = blockReduceMax<float>(score);
            if (tid == 0)
            {
                for (int j = 0; j < nBM * 2; ++j)
                {
                    if (smemScore[j] == maxScore)
                    {
                        smemRank[i] = j;
                        smemScore[j] = -FLT_MAX;
                        break;
                    }
                }
            }
            __syncthreads();
        }
    }

    // Move bh.sequenceLengths, bh.cumLogProbs
    if (tid < nBM)
    {
        smemSL[tid] = bh.sequenceLengthsCBA[bid * nBM * 2 + smemRank[tid]];
        bh.sequenceLengths[bid * nBM + tid] = smemSL[tid];
        if (bh.cumLogProbs != nullptr)
        {
            bh.cumLogProbs[bid * nBM + tid] = bh.cumLogProbsCBA[bid * nBM * 2 + smemRank[tid]];
        }
    }
    __syncthreads();

    // Move bh.outputIds, bh.logProbs
    for (int beamIdx = 0; beamIdx < nBM; beamIdx++)
    {
        for (int i = tid; i < smemSL[beamIdx]; i += blockDim.x)
        {
            int const dst = bid * nBM * nMSL + beamIdx * nMSL + i;
            int const src = bid * nBM * 2 * nMSL + smemRank[beamIdx] * nMSL + i;
            bh.outputIds[dst] = bh.outputIdsCBA[src];
        }
        if (bh.logProbs != nullptr)
        {
            for (int i = tid; i < smemSL[beamIdx]; i += blockDim.x)
            {
                if (int const inputLength = bh.inputLengths[bid * nBM + beamIdx]; i >= inputLength)
                {
                    int const dst = bid * nBM * nMSL + beamIdx * nMSL + i;
                    int const src = bid * nBM * 2 * nMSL + smemRank[beamIdx] * nMSL + i;
                    bh.logProbs[dst - inputLength] = bh.logProbsCBA[src];
                }
            }
        }
    }
}

void invokeFinalize(BeamHypotheses& bh, hipStream_t stream)
{
    TLLM_LOG_TRACE("%s %s start", __FILE__, __PRETTY_FUNCTION__);

    int const nBM = bh.nBeamWidth;
    size_t const smem_size = sizeof(int) * nBM * 2 + sizeof(float) * nBM * 2;
    finalizeKernel<<<bh.nBatchSize, roundUp(nBM * 2, 32), smem_size, stream>>>(bh);
    TLLM_LOG_TRACE("%s %s stop", __FILE__, __PRETTY_FUNCTION__);
}

__global__ void copyBeamHypotheses(CopyBeamHypothesesStruct copyStruct)
{
    auto const idx = static_cast<SizeType32>(threadIdx.x + blockIdx.x * blockDim.x);
    auto const stride = static_cast<SizeType32>(blockDim.x * gridDim.x);

    for (SizeType32 ii = idx; ii < copyStruct.outputIdsNumElts; ii += stride)
    {
        copyStruct.dstOutputIdsCBA[ii] = copyStruct.srcOutputIdsCBA[ii];
    }

    for (SizeType32 ii = idx; ii < copyStruct.logProbsNumElts; ii += stride)
    {
        copyStruct.dstLogProbsCBA[ii] = copyStruct.srcLogProbsCBA[ii];
    }

    for (SizeType32 ii = idx; ii < copyStruct.cumLogProbsNumElts; ii += stride)
    {
        copyStruct.dstCumLogProbs[ii] = copyStruct.srcCumLogProbs[ii];
    }

    for (SizeType32 ii = idx; ii < copyStruct.sequenceLengthsNumElts; ii += stride)
    {
        copyStruct.dstSequenceLengthsCBA[ii] = copyStruct.srcSequenceLengthsCBA[ii];
    }

    for (SizeType32 ii = idx; ii < copyStruct.cumLogProbsCBANumElts; ii += stride)
    {
        copyStruct.dstCumLogProbsCBA[ii] = copyStruct.srcCumLogProbsCBA[ii];
    }

    for (SizeType32 ii = idx; ii < copyStruct.normedScoresNumElts; ii += stride)
    {
        copyStruct.dstNormedScoresCBA[ii] = copyStruct.srcNormedScoresCBA[ii];
    }

    for (SizeType32 ii = idx; ii < copyStruct.numBeamsNumElts; ii += stride)
    {
        copyStruct.dstNumBeamsCBA[ii] = copyStruct.srcNumBeamsCBA[ii];
    }

    for (SizeType32 ii = idx; ii < copyStruct.minNormedScoresNumElts; ii += stride)
    {
        copyStruct.dstMinNormedScoresCBA[ii] = copyStruct.srcMinNormedScoresCBA[ii];
    }

    for (SizeType32 ii = idx; ii < copyStruct.batchDonesNumElts; ii += stride)
    {
        copyStruct.dstBatchDones[ii] = copyStruct.srcBatchDones[ii];
    }
}

void invokeCopyBeamHypotheses(DecodingOutput::BeamHypotheses const& src, DecodingOutput::BeamHypotheses const& dst,
    ITensor& srcCumLogProbs, ITensor& dstCumLogProbs, runtime::CudaStream const& stream, SizeType32 numSMs)
{
    CopyBeamHypothesesStruct copyStruct = {};

    copyStruct.srcOutputIdsCBA = bufferCast<TokenIdType>(*(src.outputIdsCBA));
    copyStruct.dstOutputIdsCBA = bufferCast<TokenIdType>(*(dst.outputIdsCBA));
    copyStruct.outputIdsNumElts = dst.outputIdsCBA->getSize();

    copyStruct.srcLogProbsCBA = bufferCast<float>(*(src.logProbsCBA));
    copyStruct.dstLogProbsCBA = bufferCast<float>(*(dst.logProbsCBA));
    copyStruct.logProbsNumElts = dst.logProbsCBA->getSize();

    copyStruct.srcSequenceLengthsCBA = bufferCast<SizeType32>(*(src.sequenceLengthsCBA));
    copyStruct.dstSequenceLengthsCBA = bufferCast<SizeType32>(*(dst.sequenceLengthsCBA));
    copyStruct.sequenceLengthsNumElts = dst.sequenceLengthsCBA->getSize();

    copyStruct.srcCumLogProbsCBA = bufferCast<float>(*(src.cumLogProbsCBA));
    copyStruct.dstCumLogProbsCBA = bufferCast<float>(*(dst.cumLogProbsCBA));
    copyStruct.cumLogProbsCBANumElts = dst.cumLogProbsCBA->getSize();

    copyStruct.srcNormedScoresCBA = bufferCast<float>(*(src.normedScoresCBA));
    copyStruct.dstNormedScoresCBA = bufferCast<float>(*(dst.normedScoresCBA));
    copyStruct.normedScoresNumElts = dst.normedScoresCBA->getSize();

    copyStruct.srcNumBeamsCBA = bufferCast<SizeType32>(*(src.numBeamsCBA));
    copyStruct.dstNumBeamsCBA = bufferCast<SizeType32>(*(dst.numBeamsCBA));
    copyStruct.numBeamsNumElts = dst.numBeamsCBA->getSize();

    copyStruct.srcMinNormedScoresCBA = bufferCast<float>(*(src.minNormedScoresCBA));
    copyStruct.dstMinNormedScoresCBA = bufferCast<float>(*(dst.minNormedScoresCBA));
    copyStruct.minNormedScoresNumElts = dst.minNormedScoresCBA->getSize();

    copyStruct.srcBatchDones = bufferCast<bool>(*(src.batchDones));
    copyStruct.dstBatchDones = bufferCast<bool>(*(dst.batchDones));
    copyStruct.batchDonesNumElts = dst.batchDones->getSize();

    copyStruct.srcCumLogProbs = bufferCast<float>(srcCumLogProbs);
    copyStruct.dstCumLogProbs = bufferCast<float>(dstCumLogProbs);
    copyStruct.cumLogProbsNumElts = srcCumLogProbs.getSize();

    copyBeamHypotheses<<<numSMs, 256, 0, stream.get()>>>(copyStruct);
}

__global__ void initializeOutput(TokenIdType* finalOutputIds, TokenIdType const* endIds, SizeType32 const nMaxSeqLen)
{
    for (int i = threadIdx.x; i < nMaxSeqLen; i += blockDim.x)
    {
        finalOutputIds[blockIdx.x * nMaxSeqLen + i] = endIds[blockIdx.x];
    }
}

void invokeInitializeOutput(TokenIdType* finalOutputIds, TokenIdType const* endIds, SizeType32 const batchBeam,
    SizeType32 const nMaxSeqLen, hipStream_t stream)
{
    initializeOutput<<<batchBeam, 256, 0, stream>>>(finalOutputIds, endIds, nMaxSeqLen);
}

__global__ void copyNextStepIds(TokenIdType* nextStepIds, TokenIdType const* const* outputIdsPtr,
    SizeType32 const* sequenceLengths, SizeType32 const* numNewTokens, SizeType32 const* batchSlots,
    SizeType32 batchSize, SizeType32 maxBatchSize, SizeType32 beamWidth, SizeType32 maxSeqLen,
    SizeType32 maxTokensPerStep)
{
    for (auto index = static_cast<SizeType32>(blockIdx.x * blockDim.x + threadIdx.x);
         index < batchSize * beamWidth * maxTokensPerStep; index += static_cast<SizeType32>(blockDim.x * gridDim.x))
    {
        // numNewTokens == nullptr when Medusa is disabled
        auto const batchIdx{index / (beamWidth * maxTokensPerStep)};
        auto const batchSlot{batchSlots[batchIdx]};
        auto const remainder{index % (beamWidth * maxTokensPerStep)};
        auto const beamIdx{remainder / maxTokensPerStep};
        auto const tokenIdx{remainder % maxTokensPerStep};
        auto const newTokens{numNewTokens == nullptr ? 1 : numNewTokens[batchSlot]};
        auto const batchBeamIdx = batchSlot * beamWidth + beamIdx;
        auto const tokenBatchBeamIdx = tokenIdx * maxBatchSize * beamWidth + batchSlot * beamWidth + beamIdx;
        auto const indexSrc = sequenceLengths[batchBeamIdx] - newTokens + tokenIdx;
        if (tokenIdx >= newTokens || indexSrc < 0)
        {
            continue;
        }
        nextStepIds[tokenBatchBeamIdx] = outputIdsPtr[batchSlot][beamIdx * maxSeqLen + indexSrc];
    }
}

void invokeCopyNextStepIds(TokenIdType* nextStepIds, TokenIdType const* const* outputIdsPtr,
    SizeType32 const* sequenceLengths, SizeType32 const* numNewTokens, SizeType32 const* batchSlots,
    SizeType32 batchSize, SizeType32 maxBatchSize, SizeType32 beamWidth, SizeType32 maxSeqLen,
    SizeType32 maxTokensPerStep, hipStream_t stream)
{
    int const numElems = batchSize * beamWidth * maxTokensPerStep;
    dim3 block(min(256, numElems));
    dim3 grid(divUp(numElems, block.x));
    copyNextStepIds<<<grid, block, 0, stream>>>(nextStepIds, outputIdsPtr, sequenceLengths, numNewTokens, batchSlots,
        batchSize, maxBatchSize, beamWidth, maxSeqLen, maxTokensPerStep);
}

__global__ void transposeLogProbs(float* outputLogProbs, float* outputLogProbsTiled, SizeType32 const* sequenceLengths,
    SizeType32 const* batchSlots, SizeType32 batchSize, SizeType32 maxBatchSize, SizeType32 beamWidth,
    SizeType32 maxSeqLen)
{
    auto index = static_cast<SizeType32>(blockIdx.x * blockDim.x + threadIdx.x);

    auto const batchIdx = index / (beamWidth * maxSeqLen);
    auto const tmpIdx = index % (beamWidth * maxSeqLen);
    auto const beamIdx = tmpIdx / maxSeqLen;
    auto const pos = tmpIdx % maxSeqLen;
    if (batchIdx >= batchSize)
    {
        return;
    }

    auto const batchSlot = batchSlots[batchIdx];
    if (pos < sequenceLengths[batchSlot])
    {
        auto const batchBeamIdx = batchSlot * beamWidth * maxSeqLen + beamIdx * maxSeqLen + pos;
        outputLogProbs[batchBeamIdx]
            = outputLogProbsTiled[pos * maxBatchSize * beamWidth + batchSlot * beamWidth + beamIdx];
    }
}

void invokeTransposeLogProbs(float* outputLogProbs, float* outputLogProbsTiled, SizeType32 const* sequenceLengths,
    SizeType32 const* batchSlots, SizeType32 batchSize, SizeType32 maxBatchSize, SizeType32 beamWidth,
    SizeType32 maxSeqLen, hipStream_t stream)
{
    dim3 block(256);
    dim3 grid(divUp(batchSize * beamWidth * maxSeqLen, block.x));
    transposeLogProbs<<<grid, block, 0, stream>>>(outputLogProbs, outputLogProbsTiled, sequenceLengths, batchSlots,
        batchSize, maxBatchSize, beamWidth, maxSeqLen);
}

} // namespace kernels
} // namespace tensorrt_llm
