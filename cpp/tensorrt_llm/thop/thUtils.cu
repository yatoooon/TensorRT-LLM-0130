#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/thop/thUtils.h"
#include <NvInferRuntime.h>
#include <array>

namespace torch_ext
{

tensorrt_llm::runtime::ITensor::Shape convert_shape(torch::Tensor tensor)
{
    constexpr auto trtMaxDims = nvinfer1::Dims::MAX_DIMS;
    auto const torchTensorNumDims = tensor.dim();
    TLLM_CHECK_WITH_INFO(torchTensorNumDims <= trtMaxDims,
        "TensorRT supports at most %i tensor dimensions. Found a Torch tensor with %li dimensions.", trtMaxDims,
        torchTensorNumDims);
    auto result = nvinfer1::Dims{};
    result.nbDims = static_cast<int32_t>(torchTensorNumDims);
    for (int i = 0; i < torchTensorNumDims; i++)
    {
        result.d[i] = static_cast<int64_t>(tensor.size(i));
    }
    return result;
}

template <typename T>
tensorrt_llm::runtime::ITensor::UniquePtr convert_tensor(torch::Tensor tensor)
{
    return tensorrt_llm::runtime::ITensor::wrap(
        get_ptr<T>(tensor), tensorrt_llm::runtime::TRTDataType<T>::value, convert_shape(tensor));
}

// Template instantiations
template tensorrt_llm::runtime::ITensor::UniquePtr convert_tensor<int32_t*>(torch::Tensor tensor);
template tensorrt_llm::runtime::ITensor::UniquePtr convert_tensor<int32_t>(torch::Tensor tensor);
template tensorrt_llm::runtime::ITensor::UniquePtr convert_tensor<uint8_t>(torch::Tensor tensor);
template tensorrt_llm::runtime::ITensor::UniquePtr convert_tensor<int8_t>(torch::Tensor tensor);
template tensorrt_llm::runtime::ITensor::UniquePtr convert_tensor<float>(torch::Tensor tensor);
template tensorrt_llm::runtime::ITensor::UniquePtr convert_tensor<half>(torch::Tensor tensor);
#ifdef ENABLE_BF16
template tensorrt_llm::runtime::ITensor::UniquePtr convert_tensor<__hip_bfloat16>(torch::Tensor tensor);
#endif
template tensorrt_llm::runtime::ITensor::UniquePtr convert_tensor<bool>(torch::Tensor tensor);

} // namespace torch_ext
