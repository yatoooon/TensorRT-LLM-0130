#include "hip/hip_runtime.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/cutlass_kernels/cutlass_preprocessors.h"
#include "tensorrt_llm/runtime/cudaStream.h"

#include <algorithm>
#include <gtest/gtest.h>
#include <numeric>

#include "tensorrt_llm/kernels/mixtureOfExperts/moe_kernels.h"
#include "tensorrt_llm/runtime/bufferManager.h"

using namespace tensorrt_llm::kernels;
using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

constexpr static float FP8_MAX = 440; // FP8_E4M3_MAX;

__host__ __device__ constexpr float expertShift(int expert, int num_experts)
{
    return float(expert) / num_experts;
}

template <class T>
__global__ void initWeightsKernel(T* data, int64_t w, int64_t h, float base, float scale)
{
    size_t expert_id = blockIdx.z;
    T* start_offset = data + expert_id * w * h;
    float expert_shift = scale * expertShift(expert_id, gridDim.z);

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < w && y < h)
    {
        start_offset[y * w + x] = (x == y) ? T(base * scale + expert_shift) : T(0.f);
    }
}

template <class T>
__global__ void initWeightsGatedKernel(T* data, int64_t w, int64_t h, float base_1, float base_2, float scale)
{
    size_t expert_id = blockIdx.z;
    T* start_offset = data + expert_id * w * h * 2;

    float expert_shift = scale * expertShift(expert_id, gridDim.z);

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < w && y < h)
    {
        start_offset[y * w + x] = (x == y) ? T(base_1 * scale + expert_shift) : T(0.f);
        start_offset[(y + h) * w + x] = (x == y) ? T(base_2 * scale + expert_shift) : T(0.f);
    }
}

template <class T>
__global__ void initBiasToExpertIdKernel(T* data, int64_t w)
{
    size_t expert_id = blockIdx.y;
    T* start_offset = data + expert_id * w;

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < w)
        start_offset[x] = T(expert_id);
}

template <class T>
__global__ void initBiasToExpertIdGatedKernel(T* data, int64_t w)
{
    size_t expert_id = blockIdx.y;
    T* start_offset = data + expert_id * w * 2;

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < w)
    {
        start_offset[x] = T(expert_id);
        start_offset[x + w] = T(expert_id + 1);
    }
}

#ifdef ENABLE_FP8
using SafeFP8 = __hip_fp8_e4m3_fnuz;
#else
using SafeFP8 = void;
#endif

template <class TypeTuple_>
class MixtureOfExpertsTest : public ::testing::Test
{
protected:
    using DataType = typename TypeTuple_::DataType;
    using WeightType = typename TypeTuple_::WeightType;
    using OutputType = typename TypeTuple_::OutputType;
    constexpr static bool INT4 = std::is_same_v<WeightType, cutlass::uint4b_t>;
    constexpr static bool FP8 = std::is_same_v<DataType, SafeFP8>;
    constexpr static bool INT_QUANT = !std::is_same_v<DataType, WeightType>;
    using WeightStorage = std::conditional_t<INT_QUANT, uint8_t, WeightType>;
    constexpr static int WEIGHT_ELEM_PER_BYTE = INT4 ? 2 : 1;
    constexpr static int64_t HIDDEN_SIZE_MULTIPLIER = 16;
    constexpr static int64_t MINIMUM_ALIGNMENT = 64 / sizeof(WeightType) * WEIGHT_ELEM_PER_BYTE;
    constexpr static int64_t DEFAULT_HIDDEN_SIZE = HIDDEN_SIZE_MULTIPLIER * MINIMUM_ALIGNMENT;

    static BufferManager::CudaStreamPtr mStream;
    static std::unique_ptr<BufferManager> mBufferManager;
    static int mDeviceCount;

    std::vector<BufferManager::IBufferPtr> managed_buffers;
    float* mInputProbabilities{};
    DataType* mInputTensor{};

    int64_t mHiddenSize{};
    int64_t mNumExperts{};
    int64_t mK{};

    float getTolerance(float scale = 1.f)
    {
        // These FP8 tolerances are tuned quite tightly so should pick up any regressions
        // Whether the current results are as tight as they should be requires further investigation
        // They can be much tighter if we use the same value for all experts (so the scaling factors are trivial)
        // But that is hardly representative
        bool loose_fp8 = mIsGated || mNormMode == MOEExpertScaleNormalizationMode::RENORMALIZE;
        float tol = std::is_same_v<DataType, float> ? 0.001
            : std::is_same_v<DataType, half>        ? 0.01
            : std::is_same_v<DataType, SafeFP8>     ? (loose_fp8 ? 0.1 : 0.07)
                                                    : 0.1;

        // Keep the scale in a sane range
        scale = std::clamp(scale, 1.f, 30.f);
        return scale * tol;
    }

    static bool shouldSkip()
    {
#ifndef ENABLE_FP8
        static_assert(!FP8, "FP8 Tests enabled on unsupported CUDA version");
#endif
        bool should_skip_no_device = mDeviceCount <= 0;
        bool should_skip_unsupported_fp8 = getSMVersion() < 89 && FP8;
        return should_skip_no_device || should_skip_unsupported_fp8;
    }

    static void SetUpTestCase()
    {
        mDeviceCount = getDeviceCount();
        if (shouldSkip())
        {
            GTEST_SKIP() << "Skipping due to no/unsupported GPU";
        }

        mStream = std::make_shared<CudaStream>();
        mBufferManager = std::make_unique<BufferManager>(mStream);
    }

    static void TearDownTestCase()
    {
        mBufferManager.reset();
        mStream.reset();
    }

    void SetUp() override
    {
        if (shouldSkip())
        {
            GTEST_SKIP() << "Skipping due to no/unsupported GPU";
        }
        assert(mBufferManager);
    }

    void TearDown() override
    {
        managed_buffers.clear();
    }

    void initWeights(DataType* buffer, int64_t w, int64_t h, float base, float scalar)
    {
        dim3 block(16, 16, 1);
        dim3 grid(divUp(w, block.x), divUp(h, block.y), mNumExperts);
        initWeightsKernel<DataType><<<grid, block, 0, mStream->get()>>>(buffer, w, h, base, scalar);
    }

    void initBias(DataType* buffer, int64_t w)
    {
        dim3 block(256, 1, 1);
        dim3 grid(divUp(w, block.x), mNumExperts);
        initBiasToExpertIdKernel<DataType><<<grid, block, 0, mStream->get()>>>(buffer, w);
    }

    void initWeightsGated(DataType* buffer, int64_t w, int64_t h, float base_1, float base_2, float scalar)
    {
        if (!mIsGated)
            return initWeights(buffer, w, h, base_1, scalar);

        h /= 2;
        dim3 block(16, 16, 1);
        dim3 grid(divUp(w, block.x), divUp(h, block.y), mNumExperts);
        initWeightsGatedKernel<DataType><<<grid, block, 0, mStream->get()>>>(buffer, w, h, base_1, base_2, scalar);
    }

    void initBiasGated(DataType* buffer, int64_t w)
    {
        if (!mIsGated)
            return initBias(buffer, w);

        w /= 2;
        dim3 block(256, 1, 1);
        dim3 grid(divUp(w, block.x), mNumExperts);
        initBiasToExpertIdGatedKernel<DataType><<<grid, block, 0, mStream->get()>>>(buffer, w);
    }

    CutlassMoeFCRunner<DataType, WeightType, OutputType> mMoERunner{};
    char* mWorkspace{};
    float* mScaleProbs{};
    DataType* mRawExpertWeight1{};
    DataType* mRawExpertWeight2{};
    WeightStorage* mExpertWeight1{};
    WeightStorage* mExpertWeight2{};
    DataType* mExpertIntScale1{};
    DataType* mExpertIntScale2{};

    float mFP8WeightScalar1{1.f};
    float mFP8WeightScalar2{1.f};
    float* mExpertFP8Scale1{};
    float* mExpertFP8Scale2{};
    float* mExpertFP8Scale3{};

    DataType* mExpertBias1{};
    DataType* mExpertBias2{};

    void* mTpExpertScratch{}; // Copy the experts here when slicing up inputs
    size_t mTpExpertScratchSize{};

    OutputType* mFinalOutput{};
    int* mSourceToExpandedMap;
    int* mSelectedExpert;
    bool* mFinished{};
    int64_t mInterSize{};
    int64_t mTotalTokens{};
    int64_t mActiveRows{};

    bool mUseBias = true;
    bool mUseLora = false;

    bool mIsGated = false;
    int64_t mGatedMultiplier = 1;

    tensorrt_llm::ActivationType mActType = tensorrt_llm::ActivationType::Relu;
    MOEExpertScaleNormalizationMode mNormMode = MOEExpertScaleNormalizationMode::NONE;

    float mSparseMixerEpsilon = 0.2f;

    // Default this to true. This only matters for K>2, and so by doing this we will test the fused and unfused paths
    bool mUseDeterminsiticHopperReduce = true;

    // If the test sets mOverrideSelectedConfig1 the BasicPermuteTest and *ParallelTests will use that instead of
    // looping over samples for the different architectures we support.
    std::optional<tensorrt_llm::cutlass_extensions::CutlassGemmConfig> mOverrideSelectedConfig1 = std::nullopt;
    std::optional<tensorrt_llm::cutlass_extensions::CutlassGemmConfig> mOverrideSelectedConfig2 = std::nullopt;

    // This is the actual tactic we use internally in runMoePermute
    std::optional<tensorrt_llm::cutlass_extensions::CutlassGemmConfig> mInternalSelectedConfig1 = std::nullopt;
    std::optional<tensorrt_llm::cutlass_extensions::CutlassGemmConfig> mInternalSelectedConfig2 = std::nullopt;

    // Keep to simple power of two so we can have tight bounds on precision for quantized modes
    float const mExpertWDiag1{0.5};
    float const mExpertWDiagGated{1};
    float const mExpertWDiag2{2};

    float mMaxInput{};

    template <class T>
    T* allocBuffer(size_t size)
    {
        managed_buffers.emplace_back(mBufferManager->gpu(size * sizeof(T)));
        EXPECT_EQ(hipGetLastError(), hipSuccess) << "Error allocating buffer of size: " << size;
        T* ptr = static_cast<T*>(managed_buffers.back()->data());
        check_cuda_error(hipMemsetAsync(ptr, 0xD5, size * sizeof(T), mStream->get()));
        return ptr;
    }

    bool checkSufficientTestMemory(int64_t num_tokens, int64_t hidden_size, int64_t num_experts, int64_t k)
    {
        this->managed_buffers.clear();             // Make sure all the previous buffers are freed
        check_cuda_error(hipDeviceSynchronize()); // Sync to make sure all previous operations are resolved

        // Calculate the size contributions for all the large buffers to check if the GPU has enough space
        bool const is_gated = tensorrt_llm::isGatedActivation(mActType);
        size_t const num_gemms = 2 + is_gated;
        // Expert weights
        size_t const weight_size = hidden_size * (hidden_size * 4) * num_experts * sizeof(WeightStorage) * num_gemms;
        // Workspace size
        size_t const workspace_size = this->mMoERunner.getWorkspaceSize(
            num_tokens, hidden_size, hidden_size * 4, num_experts, k, this->mActType, mNormMode, {}, mUseLora);
        // The input/output buffers
        size_t const in_out_size = 2 * num_tokens * hidden_size * sizeof(DataType);

        // This should be correct to within 100MiB (on tests with 30GiB total)
        size_t const total_size = workspace_size + weight_size + in_out_size;

        size_t const memory_pool_free_mem_size = mBufferManager->memoryPoolFree();
        auto const [freeMem, totalMem] = tensorrt_llm::common::getDeviceMemoryInfo(false);
        float const freeMemBuffer = 0.9f; // Add some buffer so we aren't completely pushing the limits
        std::cout << "Free memory is: " << freeMem << ", memory pool size is: " << memory_pool_free_mem_size
                  << ", required memory is: " << total_size << ", device total memory capacity: " << totalMem
                  << std::endl;
        return (freeMem + memory_pool_free_mem_size) * freeMemBuffer >= total_size;
    }

    void initBuffersPermute(std::vector<std::vector<DataType>> h_hidden_states,
        std::vector<std::vector<float>> h_router_results, int64_t hidden_size, int64_t num_experts, int64_t k,
        std::vector<uint8_t> finished, MOEParallelismConfig parallelism_config)
    {
        managed_buffers.clear();

        mMoERunner.use_deterministic_hopper_reduce_ = k > 2 && mUseDeterminsiticHopperReduce;

        mHiddenSize = hidden_size;
        mInterSize = hidden_size * 4;
        mNumExperts = num_experts;
        mK = k;
        mIsGated = tensorrt_llm::isGatedActivation(mActType);
        mGatedMultiplier = mIsGated ? 2 : 1;
        auto const gated_inter = mInterSize * mGatedMultiplier;

        mTotalTokens = 0;

        std::vector<int64_t> h_seq_lens;
        h_seq_lens.push_back(0);
        for (auto& sequence : h_hidden_states)
        {
            assert(sequence.size() % hidden_size == 0);
            int64_t num_tokens = sequence.size() / hidden_size;
            h_seq_lens.emplace_back(h_seq_lens.back() + num_tokens);
            mTotalTokens += num_tokens;
        }

        size_t workspace_size = mMoERunner.getWorkspaceSize(
            mTotalTokens, mHiddenSize, mInterSize, mNumExperts, mK, mActType, mNormMode, parallelism_config, mUseLora);

        auto const stream = mStream->get();

        mWorkspace = allocBuffer<char>(workspace_size);
        // Memset to an obviously incorrect value, so we detect any issues with uninitialised fields
        check_cuda_error(hipMemsetAsync(mWorkspace, 0xD5, workspace_size, stream));
        size_t const expert_matrix_size = mNumExperts * mHiddenSize * mInterSize;

        mRawExpertWeight1 = allocBuffer<DataType>(expert_matrix_size * mGatedMultiplier);
        mRawExpertWeight2 = allocBuffer<DataType>(expert_matrix_size);

        size_t const experts_per_node = mNumExperts / parallelism_config.ep_size;
        int const moe_parallel_size = parallelism_config.tp_size * parallelism_config.ep_size;

        mTpExpertScratchSize = expert_matrix_size * mGatedMultiplier / moe_parallel_size;
        mTpExpertScratchSize += expert_matrix_size / moe_parallel_size;

        mExpertBias1 = nullptr;
        mExpertBias2 = nullptr;
        if (mUseBias)
        {
            // Allow space for the slice of bias1 in the scratch
            mTpExpertScratchSize += experts_per_node * gated_inter / parallelism_config.tp_size;
            mExpertBias1 = allocBuffer<DataType>(mNumExperts * gated_inter);
            mExpertBias2 = allocBuffer<DataType>(mNumExperts * mHiddenSize);

            check_cuda_error(hipMemsetAsync(mExpertBias1, 0x0, mNumExperts * gated_inter * sizeof(DataType), stream));
            check_cuda_error(hipMemsetAsync(mExpertBias2, 0x0, mNumExperts * mHiddenSize * sizeof(DataType), stream));
        }

        if constexpr (INT_QUANT)
        {
            mExpertWeight1 = allocBuffer<WeightStorage>(expert_matrix_size * mGatedMultiplier / WEIGHT_ELEM_PER_BYTE);
            mExpertWeight2 = allocBuffer<WeightStorage>(expert_matrix_size / WEIGHT_ELEM_PER_BYTE);

            mTpExpertScratchSize += experts_per_node * gated_inter / parallelism_config.tp_size;
            mExpertIntScale1 = allocBuffer<DataType>(mNumExperts * gated_inter);
            mExpertIntScale2 = allocBuffer<DataType>(mNumExperts * mHiddenSize);
        }
        else
        {
            mExpertWeight1 = mRawExpertWeight1;
            mExpertWeight2 = mRawExpertWeight2;
        }

        if constexpr (FP8)
        {
            mExpertFP8Scale1 = allocBuffer<float>(mNumExperts);
            mExpertFP8Scale2 = allocBuffer<float>(1);
            mExpertFP8Scale3 = allocBuffer<float>(mNumExperts);

            EXPECT_NE(mMaxInput, 0.0f);
            initFP8Scales(mMaxInput);
        }

        if (parallelism_config.tp_size > 1 || parallelism_config.ep_size > 1)
        {
            mTpExpertScratch = allocBuffer<DataType>(mTpExpertScratchSize);
        }

        mActiveRows = mTotalTokens;
        mFinished = nullptr;
        if (!finished.empty())
        {
            mFinished = allocBuffer<bool>(mTotalTokens);
            check_cuda_error(hipMemcpyAsync(
                mFinished, finished.data(), mTotalTokens * sizeof(bool), hipMemcpyHostToDevice, stream));
            static_assert(sizeof(bool) == sizeof(uint8_t), "Test assumes bool is interchangeable with uint8_t");
            mActiveRows = std::count(finished.begin(), finished.end(), 0);
        }

        mInputProbabilities = allocBuffer<float>(mTotalTokens * mNumExperts);
        mScaleProbs = allocBuffer<float>(mTotalTokens * mK);
        mInputTensor = allocBuffer<DataType>(mTotalTokens * mHiddenSize);
        mFinalOutput = allocBuffer<OutputType>(mTotalTokens * mHiddenSize);

        mSourceToExpandedMap = allocBuffer<int>(mTotalTokens * mK);
        mSelectedExpert = allocBuffer<int>(mTotalTokens * mK);

        auto* input_probs_ptr = mInputProbabilities;
        for (auto& sequence : h_router_results)
        {
            check_cuda_error(hipMemcpyAsync(
                input_probs_ptr, sequence.data(), sequence.size() * sizeof(float), hipMemcpyHostToDevice, stream));
            input_probs_ptr += sequence.size();
        }

        auto* hidden_states_ptr = mInputTensor;
        for (auto& sequence : h_hidden_states)
        {
            check_cuda_error(hipMemcpyAsync(hidden_states_ptr, sequence.data(), sequence.size() * sizeof(DataType),
                hipMemcpyHostToDevice, stream));
            hidden_states_ptr += sequence.size();
        }

        check_cuda_error(hipStreamSynchronize(stream));

        // Init the diagonals of our matrix, this will set to the scalar value
        initWeightsGated(
            mRawExpertWeight1, mHiddenSize, gated_inter, mExpertWDiag1, mExpertWDiagGated, mFP8WeightScalar1);
        initWeights(mRawExpertWeight2, mInterSize, mHiddenSize, mExpertWDiag2, mFP8WeightScalar2);

        if (mUseBias)
        {
            initBiasGated(mExpertBias1, gated_inter);
            initBias(mExpertBias2, mHiddenSize);
        }

        if constexpr (INT_QUANT)
        {
            cutlass_kernels::QuantType quant_type
                = INT4 ? cutlass_kernels::QuantType::W4_A16 : cutlass_kernels::QuantType::W8_A16;

            std::vector<size_t> shape1{(size_t) mNumExperts, (size_t) mHiddenSize, (size_t) gated_inter};
            std::vector<size_t> shape2{(size_t) mNumExperts, (size_t) mInterSize, (size_t) mHiddenSize};

            doIntQuant(quant_type, shape1, mRawExpertWeight1, mExpertIntScale1, mExpertWeight1);
            doIntQuant(quant_type, shape2, mRawExpertWeight2, mExpertIntScale2, mExpertWeight2);
        }

        check_cuda_error(hipStreamSynchronize(stream));
    }

    void doIntQuant(cutlass_kernels::QuantType quant_type, std::vector<size_t> shape, DataType* inputs,
        DataType* scales, uint8_t* outputs)
    {
        // Runs on the CPU, must be after stream sync
        if constexpr (INT_QUANT)
        {
            size_t elems = std::reduce(shape.begin(), shape.end(), 1, std::multiplies{});
            std::vector<int8_t> h_out(elems);
            std::vector<DataType> h_input(elems);
            std::vector<DataType> h_scales(shape[0] * shape[2]);

            check_cuda_error(hipMemcpy(h_input.data(), inputs, elems * sizeof(DataType), hipMemcpyDeviceToHost));

            cutlass_kernels::symmetric_quantize(h_out.data(), h_scales.data(), h_input.data(), shape, quant_type, true);

            check_cuda_error(hipMemcpy(
                outputs, h_out.data(), elems * sizeof(int8_t) / WEIGHT_ELEM_PER_BYTE, hipMemcpyHostToDevice));
            check_cuda_error(
                hipMemcpy(scales, h_scales.data(), h_scales.size() * sizeof(DataType), hipMemcpyHostToDevice));
        }
    }

    constexpr static float getFP8Scalar(float in)
    {
        return FP8_MAX / in;
    }

    void initFP8Scales(float max_input)
    {
        check_cuda_error(hipStreamSynchronize(mStream->get()));

        // Add shift to the max because we add an adjustment for each expert so they get different results.
        float max_shift = expertShift(mNumExperts - 1, mNumExperts);
        float maxW1 = max_shift + (mIsGated ? std::max(mExpertWDiag1, mExpertWDiagGated) : mExpertWDiag1);
        float maxW2 = max_shift + mExpertWDiag2;
        float scaleW1 = getFP8Scalar(maxW1);
        float scaleW2 = getFP8Scalar(maxW2);
        mFP8WeightScalar1 = scaleW1;
        mFP8WeightScalar2 = scaleW2;

        float scaleAct1 = getFP8Scalar(max_input);

        float maxFC1Output = calcMLPVal(max_input, mNumExperts - 1) / maxW2;
        float scaleAct2 = getFP8Scalar(maxFC1Output);

        ASSERT_NE(mExpertFP8Scale1, nullptr);
        ASSERT_NE(mExpertFP8Scale2, nullptr);
        ASSERT_NE(mExpertFP8Scale3, nullptr);

        // Dequant values for each expert are 1/(w_i*a_i) calculated above
        std::vector<float> scales_1(mNumExperts, 1.f / (scaleW1 * scaleAct1));
        std::vector<float> scales_2(1, scaleAct2);
        std::vector<float> scales_3(mNumExperts, 1.f / (scaleW2 * scaleAct2));

        check_cuda_error(hipMemcpyAsync(mExpertFP8Scale1, scales_1.data(), scales_1.size() * sizeof(float),
            hipMemcpyHostToDevice, mStream->get()));
        check_cuda_error(hipMemcpyAsync(mExpertFP8Scale2, scales_2.data(), scales_2.size() * sizeof(float),
            hipMemcpyHostToDevice, mStream->get()));
        check_cuda_error(hipMemcpyAsync(mExpertFP8Scale3, scales_3.data(), scales_3.size() * sizeof(float),
            hipMemcpyHostToDevice, mStream->get()));

        check_cuda_error(hipStreamSynchronize(mStream->get()));
    }

    void resetOutBuffers()
    {
        auto stream = mStream->get();

        if (mTpExpertScratch)
            check_cuda_error(hipMemsetAsync(mTpExpertScratch, 0x0, mTpExpertScratchSize, stream));
        check_cuda_error(hipMemsetAsync(mFinalOutput, 0x0, mTotalTokens * mHiddenSize * sizeof(DataType), stream));
        check_cuda_error(hipMemsetAsync(mSourceToExpandedMap, 0x0, sizeof(int) * mTotalTokens * mK, stream));
        check_cuda_error(hipMemsetAsync(mSelectedExpert, 0x0, sizeof(int) * mTotalTokens * mK, stream));
        check_cuda_error(hipMemsetAsync(mScaleProbs, 0x0, sizeof(float) * mTotalTokens * mK, stream));

        check_cuda_error(hipStreamSynchronize(stream));
    }

    void resizeRouterInputs(
        std::vector<std::vector<float>>& h_router_results, int64_t num_experts, int64_t num_tokens_per_seq)
    {
        for (int64_t i = 0; i < h_router_results.size(); i++)
        {
            auto& seq_routing = h_router_results[i];
            int64_t num_tokens = num_tokens_per_seq;
            auto hardcoded_experts = seq_routing.size() / num_tokens;
            ASSERT_EQ(seq_routing.size(), hardcoded_experts * num_tokens);
            if (num_experts > hardcoded_experts)
            {
                auto pos = seq_routing.begin() + hardcoded_experts;
                for (int64_t i = 0; i < num_tokens; i++, pos += num_experts)
                {
                    pos = seq_routing.insert(pos, num_experts - hardcoded_experts, 0);
                }
            }
            ASSERT_EQ(seq_routing.size(), num_experts * num_tokens);
        }
    }

    template <class T>
    auto populateTokens(std::vector<T>& hidden_states)
    {
        if constexpr (std::is_same_v<T, SafeFP8>)
        {
            std::vector<OutputType> internal_states(hidden_states.size());
            populateTokens(internal_states);

            mMaxInput = *std::max_element(internal_states.begin(), internal_states.end());
            float scalar = getFP8Scalar(mMaxInput);
            std::transform(internal_states.begin(), internal_states.end(), hidden_states.begin(),
                [scalar](OutputType in) -> T { return static_cast<T>((float) in * scalar); });
            // Do the reverse transformation since we only have so much precision and this is a pretty broad range
            std::transform(hidden_states.begin(), hidden_states.end(), internal_states.begin(),
                [scalar](T in) -> OutputType { return static_cast<OutputType>(((float) in) / scalar); });
            return internal_states;
        }
        else
        {
            std::vector<float> base(hidden_states.size());
            std::iota(base.begin(), base.end(), 0.0f);
            // Lambda subtracts a small value so we have some < 0 to test the activation for negatives
            std::transform(base.begin(), base.end(), hidden_states.begin(),
                [l = hidden_states.size()](auto a) { return T(a / l) - T(0.01f); });
            return hidden_states;
        }
    }

    void runMoEPermute(std::vector<std::vector<DataType>> h_hidden_states,
        std::vector<std::vector<float>> h_router_results, int64_t hidden_size, int64_t num_experts, int64_t k,
        std::vector<uint8_t> finished = {}, MOEParallelismConfig parallelism_config = {})
    {
        initBuffersPermute(std::move(h_hidden_states), std::move(h_router_results), hidden_size, num_experts, k,
            finished, parallelism_config);
        runMoEPermute(parallelism_config);
    }

    auto getWeights(MOEParallelismConfig parallelism_config)
    {
        void* ep_scale_1 = FP8 ? (void*) mExpertFP8Scale1 : (void*) mExpertIntScale1;
        void* ep_scale_2 = FP8 ? (void*) mExpertFP8Scale2 : (void*) mExpertIntScale2;
        void* ep_scale_3 = FP8 ? mExpertFP8Scale3 : nullptr;

        // Handle the case with no parallelism to not require the extra alloc
        if (parallelism_config.tp_size == 1 && parallelism_config.ep_size == 1)
        {
            return std::tuple{
                mExpertWeight1, mExpertWeight2, mExpertBias1, mExpertBias2, ep_scale_1, ep_scale_2, ep_scale_3};
        }

        // Slice weights for EP
        size_t const gated_inter = mInterSize * mGatedMultiplier;
        size_t const experts_per_node = mNumExperts / parallelism_config.ep_size;
        size_t const weight_matrix_size = mHiddenSize * mInterSize * experts_per_node / WEIGHT_ELEM_PER_BYTE;
        size_t const bias_fc1_size = gated_inter * experts_per_node;
        size_t const bias_fc2_size = mHiddenSize * experts_per_node;
        size_t const scale1_size = gated_inter * experts_per_node;
        size_t const scale2_size = mHiddenSize * experts_per_node;
        auto* weight1_ptr = mExpertWeight1 + weight_matrix_size * mGatedMultiplier * parallelism_config.ep_rank;
        auto* weight2_ptr = mExpertWeight2 + weight_matrix_size * parallelism_config.ep_rank;
        auto* bias1_ptr = mUseBias ? mExpertBias1 + bias_fc1_size * parallelism_config.ep_rank : nullptr;
        auto* bias2_ptr = mUseBias ? mExpertBias2 + bias_fc2_size * parallelism_config.ep_rank : nullptr;

        if (INT_QUANT)
        {
            ep_scale_1 = mExpertIntScale1 + scale1_size * parallelism_config.ep_rank;
            ep_scale_2 = mExpertIntScale2 + scale2_size * parallelism_config.ep_rank;
        }
        if constexpr (FP8)
        {
            ep_scale_1 = mExpertFP8Scale1 + experts_per_node * parallelism_config.ep_rank;
            ep_scale_3 = mExpertFP8Scale3 + experts_per_node * parallelism_config.ep_rank;
        }

        // Slice weights for TP
        void* scale_1 = ep_scale_1;
        void* scale_2 = ep_scale_2;
        void* scale_3 = ep_scale_3;

        int const tp_size = parallelism_config.tp_size;
        int const tp_rank = parallelism_config.tp_rank;

        size_t const matrix_size = mHiddenSize * mInterSize / tp_size;
        size_t const gated_matrix_size = mHiddenSize * mInterSize * mGatedMultiplier / tp_size;
        size_t const row_size_inter = mInterSize / tp_size;

        auto* weight_1 = reinterpret_cast<WeightStorage*>(mTpExpertScratch);
        auto* weight_2 = weight_1 + experts_per_node * gated_matrix_size;
        auto* bias_1 = reinterpret_cast<DataType*>(weight_2 + experts_per_node * matrix_size);

        // 2D memcpy just the slices we care about
        // TODO Re-quantize here with matrices divided
        size_t const row_size_1 = matrix_size * sizeof(WeightStorage) / WEIGHT_ELEM_PER_BYTE;
        check_cuda_error(
            hipMemcpy2DAsync(weight_1, row_size_1, (uint8_t*) weight1_ptr + row_size_1 * tp_rank, row_size_1 * tp_size,
                row_size_1, experts_per_node * mGatedMultiplier, hipMemcpyDeviceToDevice, mStream->get()));

        size_t const row_size_2 = row_size_inter * sizeof(WeightStorage) / WEIGHT_ELEM_PER_BYTE;
        check_cuda_error(
            hipMemcpy2DAsync(weight_2, row_size_2, (uint8_t*) weight2_ptr + row_size_2 * tp_rank, row_size_2 * tp_size,
                row_size_2, experts_per_node * mHiddenSize, hipMemcpyDeviceToDevice, mStream->get()));

        if (mUseBias)
        {
            size_t const row_size_bias = row_size_inter * sizeof(DataType);
            check_cuda_error(hipMemcpy2DAsync(bias_1, row_size_bias, (uint8_t*) bias1_ptr + row_size_bias * tp_rank,
                row_size_bias * tp_size, row_size_bias, experts_per_node * mGatedMultiplier, hipMemcpyDeviceToDevice,
                mStream->get()));
        }

        if constexpr (INT_QUANT)
        {
            scale_2 = ep_scale_2;
            size_t const row_size_scale = row_size_inter * sizeof(DataType);
            check_cuda_error(hipMemcpy2DAsync(scale_1, row_size_scale,
                (uint8_t*) ep_scale_1 + row_size_scale * tp_rank, row_size_scale * tp_size, row_size_scale,
                experts_per_node * mGatedMultiplier, hipMemcpyDeviceToDevice, mStream->get()));
        }

        bias_1 = mUseBias ? bias_1 : nullptr;

        return std::tuple{weight_1, weight_2, bias_1, bias2_ptr, scale_1, scale_2, scale_3};
    }

    auto getFilteredConfigs(int sm)
    {
        auto tactics = mMoERunner.getTactics();
        if (sm == 89)
        {
            // Filter some unsupported configs for L40S
            auto it = std::remove_if(tactics.begin(), tactics.end(),
                [&](auto conf)
                {
                    using tensorrt_llm::cutlass_extensions::CutlassTileConfig;
                    auto checks = std::vector{
                        // Fail for BF16/FP16
                        conf.tile_config == CutlassTileConfig::CtaShape128x128x64_WarpShape64x32x64,
                        conf.tile_config == CutlassTileConfig::CtaShape64x128x64_WarpShape32x64x64 && conf.stages == 4,
                        // Fail for FP8
                        FP8 && conf.tile_config == CutlassTileConfig::CtaShape16x256x128_WarpShape16x64x128
                            && conf.stages >= 3,
                    };

                    return std::any_of(checks.begin(), checks.end(), [](auto v) { return v; });
                });
            tactics.erase(it, tactics.end());
        }

        EXPECT_FALSE(tactics.empty());

        return tactics;
    }

    auto selectTacticsForArch(int sm)
    {
        bool is_sm90 = sm >= 90 && !INT_QUANT;
        auto tactics = getFilteredConfigs(sm);
        auto it = std::find_if(tactics.begin(), tactics.end(), [is_sm90](auto& c) { return c.is_sm90 == is_sm90; });
        if (it == tactics.end())
        {
            // Fall back to any tactic
            std::cout << "WARNING: Could not find config for sm version " << sm << std::endl;
            return std::pair{tactics[0], tactics[0]};
        }

        return std::pair(*it, *it);
    }

    using ConfigsToTestVec = std::vector<std::pair<tensorrt_llm::cutlass_extensions::CutlassGemmConfig,
        tensorrt_llm::cutlass_extensions::CutlassGemmConfig>>;

    auto getAllTileConfigsToTest()
    {
        if (mOverrideSelectedConfig1 && mOverrideSelectedConfig2)
        {
            return ConfigsToTestVec{std::pair{*mOverrideSelectedConfig1, *mOverrideSelectedConfig2}};
        }

        int sm = getSMVersion();
        ConfigsToTestVec tactics = {selectTacticsForArch(sm)};
        if (sm >= 90)
        {
            // SM90 should also grab some configs for SM80 to test them
            tactics.push_back(selectTacticsForArch(80));
        }
        return tactics;
    }

    void runMoEPermute(MOEParallelismConfig parallelism_config)
    {
        // Clear the buffers to blank so we can assume zero if not written
        resetOutBuffers();

        auto const [weight1_ptr, weight2_ptr, bias1_ptr, bias2_ptr, scale1_ptr, scale2_ptr, scale3_ptr]
            = getWeights(parallelism_config);

        auto stream = mStream->get();
        auto tactic1 = mInternalSelectedConfig1;
        auto tactic2 = mInternalSelectedConfig2;
        if (!tactic1)
        {
            int sm = getSMVersion();
            std::tie(tactic1, tactic2) = selectTacticsForArch(sm);
        }
        ASSERT_TRUE(tactic1.has_value());
        ASSERT_TRUE(tactic2.has_value());

        QuantParams quant_params;
        if constexpr (INT_QUANT)
        {
            quant_params = QuantParams::Int(scale1_ptr, scale2_ptr);
        }
        else
        {
            quant_params = QuantParams::FP8(static_cast<float const*>(scale1_ptr),
                static_cast<float const*>(scale2_ptr), static_cast<float const*>(scale3_ptr));
        }

        LoraParams lora_params;

        mMoERunner.setTactic(tactic1, tactic2);
        mMoERunner.runMoe(mInputTensor, mInputProbabilities, weight1_ptr, bias1_ptr, mActType, weight2_ptr, bias2_ptr,
            quant_params, mTotalTokens, mHiddenSize, mInterSize / parallelism_config.tp_size, mNumExperts, mK,
            mWorkspace, mFinalOutput, mFinished, mActiveRows, mScaleProbs, mSourceToExpandedMap, mSelectedExpert,
            mSparseMixerEpsilon, parallelism_config, mNormMode, mUseLora, lora_params, stream);

        check_cuda_error(hipStreamSynchronize(stream));
    }

    template <class T>
    std::vector<T> getDataFromDevice(T const* in, size_t length)
    {
        std::vector<T> data(length);

        auto const stream = mStream->get();
        check_cuda_error(hipMemcpyAsync(data.data(), in, length * sizeof(T), hipMemcpyDeviceToHost, stream));
        check_cuda_error(hipStreamSynchronize(mStream->get()));

        return data;
    }

    auto maskSelectedExpertsForTP(std::vector<int> const& vector, int tp_size, int tp_rank)
    {
        std::vector<int> result;
        int num_experts_per_node = mNumExperts / tp_size;
        std::transform(vector.begin(), vector.end(), std::back_inserter(result),
            [=](int entry)
            {
                if (entry >= num_experts_per_node * tp_rank && entry < num_experts_per_node * (tp_rank + 1))
                    return entry;
                return (int) mNumExperts + entry;
            });
        return result;
    }

    void debugPrint()
    {
#define PRINT_CAST(array, size, cast)                                                                                  \
    do                                                                                                                 \
        if (array)                                                                                                     \
        {                                                                                                              \
            auto data = getDataFromDevice(array, size);                                                                \
            std::cout << #array << ": ";                                                                               \
            for (auto v : data)                                                                                        \
            {                                                                                                          \
                if (cast(v))                                                                                           \
                    std::cout << cast(v) << ", ";                                                                      \
                else                                                                                                   \
                    std::cout << "., ";                                                                                \
            }                                                                                                          \
            std::cout << std::endl;                                                                                    \
        }                                                                                                              \
    while (0)
#define PRINT(array, size) PRINT_CAST(array, size, )

        using WeightPrintType = std::conditional_t<INT_QUANT, uint8_t, WeightStorage>;
        PRINT_CAST((WeightPrintType*) mExpertWeight1,
            mNumExperts * mHiddenSize * mInterSize * mGatedMultiplier / WEIGHT_ELEM_PER_BYTE, float);
        PRINT_CAST(
            (WeightPrintType*) mExpertWeight2, mNumExperts * mHiddenSize * mInterSize / WEIGHT_ELEM_PER_BYTE, float);
        // PRINT_CAST(mRawExpertWeight1, mNumExperts * mHiddenSize * mInterSize * mGatedMultiplier, float);
        // PRINT_CAST(mRawExpertWeight2, mNumExperts * mHiddenSize * mInterSize, float);
        PRINT_CAST(mExpertBias1, mNumExperts * mInterSize * mGatedMultiplier, float);
        PRINT_CAST(mExpertBias2, mNumExperts * mHiddenSize, float);
        PRINT_CAST(mExpertIntScale1, mNumExperts * mInterSize * mGatedMultiplier, float);
        PRINT_CAST(mExpertIntScale2, mNumExperts * mHiddenSize, float);
        PRINT(mFinalOutput, mTotalTokens * mHiddenSize);
        PRINT_CAST((uint8_t*) mFinished, mTotalTokens, (int) );
        PRINT(mInputProbabilities, mTotalTokens * mNumExperts);
        PRINT(mScaleProbs, mTotalTokens * mK);
        PRINT(mInputProbabilities, mTotalTokens * mNumExperts);
        PRINT_CAST(mInputTensor, mTotalTokens * mHiddenSize, float);
        PRINT(mSourceToExpandedMap, mTotalTokens * mK);
        PRINT(mSelectedExpert, mTotalTokens * mK);

#undef PRINT_CAST
#undef PRINT
    }

    template <class T>
    T actfn(T in)
    {
        if (mActType == tensorrt_llm::ActivationType::Identity)
            return in;
        if (mActType == tensorrt_llm::ActivationType::Relu)
            return std::max(in, T(0.0f));
        if (mActType == tensorrt_llm::ActivationType::Gelu || mActType == tensorrt_llm::ActivationType::Geglu)
            return (std::erf(float(in) * float(sqrt(0.5))) + 1) * 0.5f * float(in);
        if (mActType == tensorrt_llm::ActivationType::Silu || mActType == tensorrt_llm::ActivationType::Swiglu)
        {
            return (float(in) / (1.f + std::exp(-(in))));
        }
        assert(false);
        return in;
    }

    float calcMLPVal(float input, int expert_id, bool final_bias = false)
    {
        if (expert_id >= mNumExperts)
            return 0;

        float expert_shift = expertShift(expert_id, mNumExperts);
        float w1_bias = mUseBias ? expert_id : 0.f;
        float activated = 0;
        if (mIsGated)
        {
            float scalar = mExpertWDiag1 + expert_shift;
            float fc1 = (float) input * scalar + (float) w1_bias;

            float gated_scalar = mExpertWDiagGated + expert_shift;
            float gated_bias = mUseBias ? (float) w1_bias + 1.f : 0.f;
            float gate = (float) input * gated_scalar + gated_bias;

            activated = fc1 * actfn(gate);
        }
        else
        {
            float scalar = mExpertWDiag1 + expert_shift;
            float fc1 = input * scalar + w1_bias;
            activated = actfn(fc1);
        }

        EXPECT_TRUE(mUseBias || !final_bias);
        float result = activated * (mExpertWDiag2 + expert_shift) + (float) (final_bias ? expert_id : 0);
        return result;
    }

    float calcMLPValWithFinalBias(float input, int expert_id)
    {
        return calcMLPVal(input, expert_id, mUseBias);
    }

    // NOTE This is a useful function for debugging routing failures. But you need to know the exact offset of
    //   this info in the workspace so having a test depend on something so internal is suboptimal
    //
    // void comparePermuted(const std::vector<int>& expected_experts, const std::vector<int>& expected_permutation,
    //     const std::vector<DataType>& input_data)
    //{
    //     auto states = getDataFromDevice(magic incantation into workspace, mTotalTokens * mK * mHiddenSize);
    //
    //    // Loop for the number of times each token is duplicated
    //    for (int k_idx = 0; k_idx < mK; k_idx++)
    //    {
    //        for (int64_t token_id = 0; token_id < mTotalTokens; token_id++)
    //        {
    //            // Permutation has the position of the first copy of all token,
    //            // followed by the position of the second copy of all tokens etc.
    //            const int64_t permuted_position = expected_permutation[k_idx * mTotalTokens + token_id];
    //
    //            // Expected experts has all the selected experts for token one,
    //            // followed by all the selected experts for token two etc.
    //            const int64_t expert_id = expected_experts[token_id * mK + k_idx];
    //
    //            // Compare the copied tokens with the projection applied
    //            for (int64_t hidden_id = 0; hidden_id < mHiddenSize; hidden_id++)
    //            {
    //                auto ref = calcMLPVal(input_data[token_id * mHiddenSize + hidden_id], expert_id);
    //                auto actual = states[permuted_position * mHiddenSize + hidden_id];
    //                ASSERT_NEAR(ref, actual, getTolerance(ref))
    //                    << "Incorrect value at position: mK: " << k_idx << ", token: " << token_id
    //                    << ", permuted dest: " << permuted_position << ", expert id: " << expert_id
    //                    << ", hidden id: " << hidden_id;
    //            }
    //        }
    //    }
    //}

    std::vector<float> softmax(std::vector<float> const& expected_probs)
    {
        std::vector<float> softmax;
        // All values we test are 0-1 so we can skip the normalization step
        std::transform(expected_probs.begin(), expected_probs.end(), std::back_inserter(softmax),
            [&](float const in) -> float
            {
                auto res = exp(in);
                return res;
            });

        for (int64_t token = 0; token < softmax.size(); token += mNumExperts)
        {
            auto start = softmax.begin() + token;
            auto end = start + mNumExperts;
            auto sum = std::accumulate(start, end, 0.f);
            std::transform(start, end, start, [=](auto in) { return in / sum; });
        }

        return softmax;
    }

    void renormScales(float* probs, int const* experts)
    {
        if (mNormMode != MOEExpertScaleNormalizationMode::RENORMALIZE)
            return;
        float sum = 0;
        for (int k_idx = 0; k_idx < mK; k_idx++)
        {
            sum += probs[experts[k_idx]];
        }
        float norm_factor = 1.0f / sum;
        for (int k_idx = 0; k_idx < mK; k_idx++)
        {
            probs[experts[k_idx]] *= norm_factor;
        }
    }

    float sparseMixer(std::vector<float> logits, int token_idx, int k_idx, int expected_expert)
    {
        EXPECT_LE(mK, 2);
        EXPECT_LT(k_idx, mK);
        EXPECT_LT(token_idx * mNumExperts, logits.size());
        EXPECT_LE((token_idx + 1) * mNumExperts, logits.size());

        auto start_it = logits.begin() + token_idx * mNumExperts;
        auto end_it = logits.begin() + (token_idx + 1) * mNumExperts;

        // Mask old maxes and get the kth largest
        auto max_it = end_it;
        for (int i = 0; i <= k_idx; i++)
        {
            max_it = std::max_element(start_it, end_it);
            if (i != k_idx)
            {
                EXPECT_NE(max_it, end_it);
                *max_it = -INFINITY;
            }
        }

        EXPECT_EQ((max_it - start_it), expected_expert)
            << "Expected token " << token_idx << " k_idx " << k_idx << " to select expert " << expected_expert;

        std::vector<float> masked;
        std::transform(start_it, end_it, std::back_inserter(masked),
            [this, max_it](auto val)
            {
                float mask_value = (*max_it - val) / max(abs(val), *max_it);
                return (mask_value > 2 * mSparseMixerEpsilon) ? -INFINITY : val;
            });
        auto output_probs = softmax(masked);
        return output_probs[expected_expert];
    }

    void compareSoftmax(std::vector<int> const& expected_experts, std::vector<float> const& expected_probs,
        std::vector<float> scale_probs = {})
    {
        if (scale_probs.empty())
            scale_probs = getDataFromDevice(mScaleProbs, mTotalTokens * mK);
        auto softmax_probs = softmax(expected_probs);

        for (int64_t token_id = 0; token_id < mTotalTokens; token_id++)
        {
            renormScales(&softmax_probs[token_id * mNumExperts], &expected_experts[token_id * mK]);

            for (int k_idx = 0; k_idx < mK; k_idx++)
            {
                int selected_expert = expected_experts[token_id * mK + k_idx];
                if (selected_expert < mNumExperts) // Ignore 'finished' values
                {
                    float expected_value = softmax_probs[token_id * mNumExperts + selected_expert];
                    if (mNormMode == tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::SPARSE_MIXER)
                    {
                        expected_value = sparseMixer(expected_probs, token_id, k_idx, selected_expert);
                    }

                    ASSERT_NEAR(expected_value, scale_probs[token_id * mK + k_idx], getTolerance())
                        << "Scales mismatched for token: " << token_id << " k: " << k_idx
                        << " selected_expert: " << selected_expert;
                }
            }
        }
    }

    void compareFinal(std::vector<int> const& expected_experts, std::vector<float> const& expected_probs,
        std::vector<OutputType> const& input_data, std::vector<OutputType> final_results = {})
    {
        if (final_results.empty())
            final_results = getDataFromDevice(mFinalOutput, mTotalTokens * mHiddenSize);

        auto softmax_probs = softmax(expected_probs);
        for (int64_t token_id = 0; token_id < mTotalTokens; token_id++)
        {
            renormScales(&softmax_probs[token_id * mNumExperts], &expected_experts[token_id * mK]);

            for (int64_t hidden_id = 0; hidden_id < mHiddenSize; hidden_id++)
            {
                float sum = 0.0f;
                // Loop for the number of times each token is duplicated
                for (int k_idx = 0; k_idx < mK; k_idx++)
                {
                    int selected_expert = expected_experts[token_id * mK + k_idx];

                    float scale_value = softmax_probs[token_id * mNumExperts + selected_expert];
                    if (mNormMode == tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::SPARSE_MIXER)
                    {
                        scale_value = sparseMixer(expected_probs, token_id, k_idx, selected_expert);
                    }

                    sum += float(calcMLPValWithFinalBias(
                               static_cast<float>(input_data[token_id * mHiddenSize + hidden_id]), selected_expert))
                        * scale_value;
                }

                ASSERT_NEAR(OutputType{sum}, final_results[token_id * mHiddenSize + hidden_id], getTolerance(sum))
                    << "Incorrect final value at for token: " << token_id << " offset: " << hidden_id;
            }
        }
    }

    void BasicPermuteTest(int k = 1, int64_t hidden_size = DEFAULT_HIDDEN_SIZE, int64_t num_experts = 4);

    std::vector<int> calcPermuteMapExpertParallel(std::vector<int> const& expected_experts);
    void ExpertParallelTest(int k = 1);

    void TensorParallelTest(int k = 1);

    void MixedParallelTest(int k = 1);
};

template <class WeightParams>
using LargeMixtureOfExpertsTest = MixtureOfExpertsTest<WeightParams>;

template <class DataType_, class WeightType_ = DataType_, class OutputType_ = DataType_>
struct WeightParams
{
    using DataType = DataType_;
    using WeightType = WeightType_;
    using OutputType = OutputType_;
};

// TODO Fix int quantized
using Types = ::testing::Types<
#ifdef ENABLE_BF16
    WeightParams<__hip_bfloat16>,
#endif
#ifdef ENABLE_FP8
    WeightParams<SafeFP8, SafeFP8, half>,
#endif
    WeightParams<half>, WeightParams<float>

    //, WeightParams<half, uint8_t>, WeightParams<half, cutlass::uint4b_t>

    >;
TYPED_TEST_SUITE(MixtureOfExpertsTest, Types);

// Have a separate test with only FP8 and half data type because this test is long
using LargeTestTypes = ::testing::Types<
#ifdef ENABLE_FP8
    WeightParams<SafeFP8, SafeFP8, half>,
#endif
    WeightParams<half>>;
TYPED_TEST_SUITE(LargeMixtureOfExpertsTest, LargeTestTypes);

template <class TypeParam_>
BufferManager::CudaStreamPtr MixtureOfExpertsTest<TypeParam_>::mStream{};
template <class TypeParam_>
std::unique_ptr<BufferManager> MixtureOfExpertsTest<TypeParam_>::mBufferManager{};
template <class TypeParam_>
int MixtureOfExpertsTest<TypeParam_>::mDeviceCount{};

template <class TypeParam_>
void MixtureOfExpertsTest<TypeParam_>::BasicPermuteTest(int k, int64_t hidden_size, int64_t num_experts)
{
    if constexpr (FP8)
    {
        // TODO Remove this when bias + FP8 is supported
        mUseBias = false;
    }
    auto test_archs = getAllTileConfigsToTest();
    for (auto [gemm1, gemm2] : test_archs)
    {
        mInternalSelectedConfig1 = gemm1;
        mInternalSelectedConfig2 = gemm2;

        //    int64_t num_experts = 4;
        int64_t num_tokens = 3;

        std::vector<DataType> hidden_states(hidden_size * num_tokens);
        auto raw_unquant_input = populateTokens(hidden_states);

        std::vector<float> probs = {
            0.5, 0.1, 0.25, 0.15,   //
            0.03, 0.2, 0.07, 0.7,   //
            0.25, 0.21, 0.35, 0.19, //
        };

        std::vector<std::vector<DataType>> hidden_input = {hidden_states};
        std::vector<std::vector<float>> router_input = {probs};
        resizeRouterInputs(router_input, num_experts, num_tokens);

        runMoEPermute(hidden_input, router_input, hidden_size, num_experts, k);

        std::vector<int> expected_experts{0, 3, 2};
        if (k == 2)
            expected_experts = {0, 2, 3, 1, 2, 0};
        else if (k == 3)
            expected_experts = {0, 2, 3, 3, 1, 2, 2, 0, 1};

        auto selected_expert = getDataFromDevice(mSelectedExpert, num_tokens * k);
        EXPECT_EQ(selected_expert, expected_experts);

        auto proj_map = getDataFromDevice(mSourceToExpandedMap, num_tokens * k);
        // This is the final position of:
        // Token 1 Expert 1, T2E1, T3E1, T1E2, T2E2, T3E2
        std::vector<int> permute_map{0, 2, 1};
        if (k == 2)
            permute_map = {0, 5, 4, 3, 2, 1};
        if (k == 3)
            permute_map = {0, 8, 6, 4, 2, 1, 7, 5, 3};
        ASSERT_EQ(permute_map, proj_map);
        compareSoftmax(selected_expert, router_input[0]);
        compareFinal(selected_expert, router_input[0], raw_unquant_input);
    }
}

TYPED_TEST(MixtureOfExpertsTest, Permute)
{
    this->BasicPermuteTest();
}

TYPED_TEST(MixtureOfExpertsTest, PermuteK2)
{
    this->BasicPermuteTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteK3)
{
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteNoBias)
{
    this->mUseBias = false;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteRenormalization)
{
    this->mNormMode = tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::RENORMALIZE;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteSparseMixer)
{
    this->mNormMode = tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::SPARSE_MIXER;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteGeglu)
{
    this->mActType = tensorrt_llm::ActivationType::Geglu;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteSwiglu)
{
    this->mActType = tensorrt_llm::ActivationType::Swiglu;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteNonDeterministic)
{
    this->mUseDeterminsiticHopperReduce = false;
    // Just test case 3, cases 1&2 always use the fused paths
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteVerySmall)
{
    for (int i = 1; i <= 3; i++)
    {
        this->BasicPermuteTest(1, this->MINIMUM_ALIGNMENT * i);
        this->BasicPermuteTest(2, this->MINIMUM_ALIGNMENT * i);
        this->BasicPermuteTest(3, this->MINIMUM_ALIGNMENT * i);
    }
}

TYPED_TEST(MixtureOfExpertsTest, PermuteSwigluVerySmall)
{
    this->mActType = tensorrt_llm::ActivationType::Swiglu;
    for (int i = 1; i <= 3; i++)
    {
        this->BasicPermuteTest(1, this->MINIMUM_ALIGNMENT * i);
        this->BasicPermuteTest(2, this->MINIMUM_ALIGNMENT * i);
        this->BasicPermuteTest(3, this->MINIMUM_ALIGNMENT * i);
    }
}

TYPED_TEST(MixtureOfExpertsTest, PermuteMixtral8x7b)
{
    this->mUseBias = false;
    this->mActType = tensorrt_llm::ActivationType::Swiglu;
    this->BasicPermuteTest(2, 4096, 8);
}

template <class TypeParam_>
std::vector<int> MixtureOfExpertsTest<TypeParam_>::calcPermuteMapExpertParallel(
    std::vector<int> const& expected_experts)
{
    std::vector<int> map(expected_experts.size());
    auto getInterleavedIndex = [this](int i) { return (i % mK) * mTotalTokens + i / mK; };
    int map_idx = 0;
    for (int expert = 0; expert < mNumExperts * 2; expert++)
    {
        for (int i = 0; i < map.size(); i++)
        {
            if (expected_experts[i] == expert)
                map[getInterleavedIndex(i)] = map_idx++;
        }
    }

    return map;
}

template <class TypeParam_>
void MixtureOfExpertsTest<TypeParam_>::ExpertParallelTest(int k)
{
    if (FP8)
    {
        // TODO Remove this when bias + FP8 is supported
        mUseBias = false;
    }

    auto test_archs = getAllTileConfigsToTest();
    for (auto [gemm1, gemm2] : test_archs)
    {
        mInternalSelectedConfig1 = gemm1;
        mInternalSelectedConfig2 = gemm2;

        int64_t hidden_size = DEFAULT_HIDDEN_SIZE;
        int parallelism = 2;
        int64_t num_experts = 4;
        int64_t num_tokens = 3;

        std::vector<DataType> hidden_states(hidden_size * num_tokens);
        auto raw_unquant_input = populateTokens(hidden_states);

        std::vector<float> probs = {
            0.5, 0.1, 0.25, 0.15,   //
            0.03, 0.2, 0.07, 0.7,   //
            0.25, 0.21, 0.35, 0.19, //
        };

        std::vector<int> expected_experts{0, 3, 2};
        if (k == 2)
            expected_experts = {0, 2, 3, 1, 2, 0};
        else if (k == 3)
            expected_experts = {0, 2, 3, 3, 1, 2, 2, 0, 1};
        std::vector<OutputType> results(hidden_states.size(), 0);
        for (int i = 0; i < parallelism; i++)
        {
            if (i == 0)
            {
                // Only need to init the inputs on the first iteration
                runMoEPermute({hidden_states}, {probs}, hidden_size, num_experts, k, {},
                    MOEParallelismConfig{1, 0, parallelism, i});
            }
            else
            {
                runMoEPermute(MOEParallelismConfig{1, 0, parallelism, i});
            }

            auto selected_expert = getDataFromDevice(mSelectedExpert, num_tokens * k);
            // Experts should only be selected when we are on the right node
            // Note the index is [0,num_experts_per_node), so we offset the experts by the start for this node
            int const start_expert = i * (mNumExperts / parallelism);
            std::transform(selected_expert.begin(), selected_expert.end(), selected_expert.begin(),
                [&](int val) { return val >= mNumExperts ? val : val + start_expert; });
            auto masked_expected_experts = maskSelectedExpertsForTP(expected_experts, parallelism, i);
            ASSERT_EQ(selected_expert, masked_expected_experts);

            auto proj_map = getDataFromDevice(mSourceToExpandedMap, num_tokens * k);
            auto permute_map = calcPermuteMapExpertParallel(masked_expected_experts);
            ASSERT_EQ(permute_map, proj_map) << "Iteration " << i;
            compareSoftmax(expected_experts, probs);

            // Do the final reduce
            auto iter_results = getDataFromDevice(mFinalOutput, num_tokens * hidden_size);
            std::transform(
                iter_results.cbegin(), iter_results.cend(), results.cbegin(), results.begin(), std::plus<>{});
        }

        compareFinal(expected_experts, probs, raw_unquant_input, results);
    }
}

TYPED_TEST(MixtureOfExpertsTest, ExpertParallel)
{
    this->ExpertParallelTest();
}

TYPED_TEST(MixtureOfExpertsTest, ExpertParallelK2)
{
    this->ExpertParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, ExpertParallelNoBias)
{
    this->mUseBias = false;
    this->ExpertParallelTest();
    this->ExpertParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, ExpertParallelRenorm)
{
    this->mNormMode = MOEExpertScaleNormalizationMode::RENORMALIZE;
    this->ExpertParallelTest();
    this->ExpertParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, ExpertParallelSparseMixer)
{
    this->mNormMode = MOEExpertScaleNormalizationMode::SPARSE_MIXER;
    this->ExpertParallelTest();
    this->ExpertParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, ExpertParallelGeglu)
{
    this->mActType = tensorrt_llm::ActivationType::Geglu;
    this->ExpertParallelTest();
    this->ExpertParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, ExpertParallelSwiglu)
{
    this->mActType = tensorrt_llm::ActivationType::Swiglu;
    this->ExpertParallelTest();
    this->ExpertParallelTest(2);
}

template <class TypeParam_>
void MixtureOfExpertsTest<TypeParam_>::TensorParallelTest(int k)
{
    if (FP8)
    {
        // TODO Remove this when bias + FP8 is supported
        mUseBias = false;
    }

    auto test_archs = getAllTileConfigsToTest();
    for (auto [gemm1, gemm2] : test_archs)
    {
        mInternalSelectedConfig1 = gemm1;
        mInternalSelectedConfig2 = gemm2;

        int64_t hidden_size = DEFAULT_HIDDEN_SIZE;
        int parallelism = 8;
        int64_t num_experts = 4;
        int64_t num_tokens = 3;

        std::vector<DataType> hidden_states(hidden_size * num_tokens);
        auto raw_unquant_input = populateTokens(hidden_states);

        std::vector<float> probs = {
            0.5, 0.1, 0.25, 0.15,   //
            0.03, 0.2, 0.07, 0.7,   //
            0.25, 0.21, 0.35, 0.19, //
        };

        std::vector<int> expected_experts{0, 3, 2};
        if (k == 2)
            expected_experts = {0, 2, 3, 1, 2, 0};
        else if (k == 3)
            expected_experts = {0, 2, 3, 3, 1, 2, 2, 0, 1};
        std::vector<OutputType> results(hidden_states.size(), 0);
        for (int i = 0; i < parallelism; i++)
        {
            if (i == 0)
            {
                // Only need to init the inputs on the first iteration
                runMoEPermute({hidden_states}, {probs}, hidden_size, num_experts, k, {},
                    MOEParallelismConfig{parallelism, i, 1, 0});
            }
            else
            {
                runMoEPermute(MOEParallelismConfig{parallelism, i, 1, 0});
            }

            auto selected_expert = getDataFromDevice(mSelectedExpert, num_tokens * k);
            EXPECT_EQ(selected_expert, expected_experts);

            auto proj_map = getDataFromDevice(mSourceToExpandedMap, num_tokens * k);
            std::vector<int> permute_map{0, 2, 1};
            if (k == 2)
                permute_map = {0, 5, 4, 3, 2, 1};
            if (k == 3)
                permute_map = {0, 8, 6, 4, 2, 1, 7, 5, 3};

            ASSERT_EQ(permute_map, proj_map) << "Iteration " << i;

            // Do the final reduce
            auto iter_results = getDataFromDevice(mFinalOutput, num_tokens * hidden_size);
            std::transform(
                iter_results.cbegin(), iter_results.cend(), results.cbegin(), results.begin(), std::plus<>{});
        }

        compareFinal(expected_experts, probs, raw_unquant_input, results);
    }
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallel)
{
    this->TensorParallelTest();
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallelK2)
{
    this->TensorParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallelK3)
{
    this->TensorParallelTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallelNoBias)
{
    this->mUseBias = false;
    this->TensorParallelTest();
    this->TensorParallelTest(2);
    this->TensorParallelTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallelRenorm)
{
    this->mNormMode = MOEExpertScaleNormalizationMode::RENORMALIZE;
    this->TensorParallelTest();
    this->TensorParallelTest(2);
    this->TensorParallelTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallelSparseMixer)
{
    this->mNormMode = MOEExpertScaleNormalizationMode::SPARSE_MIXER;
    this->TensorParallelTest();
    this->TensorParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallelGeglu)
{
    this->mActType = tensorrt_llm::ActivationType::Geglu;
    this->TensorParallelTest();
    this->TensorParallelTest(2);
    this->TensorParallelTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, TensorParallelSwiglu)
{
    this->mActType = tensorrt_llm::ActivationType::Swiglu;
    this->TensorParallelTest();
    this->TensorParallelTest(2);
    this->TensorParallelTest(3);
}

template <class TypeParam_>
void MixtureOfExpertsTest<TypeParam_>::MixedParallelTest(int k)
{
    if (FP8)
    {
        // TODO Remove this when bias + FP8 is supported
        mUseBias = false;
    }

    auto test_archs = getAllTileConfigsToTest();
    for (auto [gemm1, gemm2] : test_archs)
    {
        mInternalSelectedConfig1 = gemm1;
        mInternalSelectedConfig2 = gemm2;

        int64_t hidden_size = DEFAULT_HIDDEN_SIZE;
        int tp_parallelism = 2;
        int ep_parallelism = 2;
        int64_t num_experts = 4;
        int64_t num_tokens = 3;

        std::vector<DataType> hidden_states(hidden_size * num_tokens);
        auto raw_unquant_input = populateTokens(hidden_states);

        std::vector<float> probs = {
            0.5, 0.1, 0.25, 0.15,   //
            0.03, 0.2, 0.07, 0.7,   //
            0.25, 0.21, 0.35, 0.19, //
        };

        std::vector<int> expected_experts{0, 3, 2};
        if (k == 2)
            expected_experts = {0, 2, 3, 1, 2, 0};
        else if (k == 3)
            expected_experts = {0, 2, 3, 3, 1, 2, 2, 0, 1};
        std::vector<OutputType> results(hidden_states.size(), 0);
        for (int i = 0; i < tp_parallelism; i++)
        {
            for (int j = 0; j < ep_parallelism; j++)
            {
                if (i == 0 && j == 0)
                {
                    // Only need to init the inputs on the first iteration
                    runMoEPermute({hidden_states}, {probs}, hidden_size, num_experts, k, {},
                        MOEParallelismConfig{tp_parallelism, i, ep_parallelism, j});
                }
                else
                {
                    runMoEPermute(MOEParallelismConfig{tp_parallelism, i, ep_parallelism, j});
                }

                auto selected_expert = getDataFromDevice(mSelectedExpert, num_tokens * k);
                // Experts should only be selected when we are on the right node
                // Note the index is [0,num_experts_per_node), so we offset the experts by the start for this node
                int const start_expert = j * (mNumExperts / ep_parallelism);
                std::transform(selected_expert.begin(), selected_expert.end(), selected_expert.begin(),
                    [&](int val) { return val >= mNumExperts ? val : val + start_expert; });
                auto masked_expected_experts = maskSelectedExpertsForTP(expected_experts, ep_parallelism, j);
                ASSERT_EQ(selected_expert, masked_expected_experts);

                auto proj_map = getDataFromDevice(mSourceToExpandedMap, num_tokens * k);
                auto permute_map = calcPermuteMapExpertParallel(masked_expected_experts);
                ASSERT_EQ(permute_map, proj_map) << "Iteration " << i << " " << j;
                compareSoftmax(expected_experts, probs);

                // Do the final reduce
                auto iter_results = getDataFromDevice(mFinalOutput, num_tokens * hidden_size);
                std::transform(
                    iter_results.cbegin(), iter_results.cend(), results.cbegin(), results.begin(), std::plus<>{});
            }
        }

        compareFinal(expected_experts, probs, raw_unquant_input, results);
    }
}

TYPED_TEST(MixtureOfExpertsTest, MixedParallel)
{
    this->MixedParallelTest();
}

TYPED_TEST(MixtureOfExpertsTest, MixedParallelK2)
{
    this->MixedParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, MixedParallelNoBias)
{
    this->mUseBias = false;
    this->MixedParallelTest();
    this->MixedParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, MixedParallelRenorm)
{
    this->mNormMode = MOEExpertScaleNormalizationMode::RENORMALIZE;
    this->MixedParallelTest();
    this->MixedParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, MixedParallelSparseMixer)
{
    this->mNormMode = MOEExpertScaleNormalizationMode::SPARSE_MIXER;
    this->MixedParallelTest();
    this->MixedParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, MixedParallelGeglu)
{
    this->mActType = tensorrt_llm::ActivationType::Geglu;
    this->MixedParallelTest();
    this->MixedParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, MixedParallelSwiglu)
{
    this->mActType = tensorrt_llm::ActivationType::Swiglu;
    this->MixedParallelTest();
    this->MixedParallelTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, ConfigSweep)
{
    auto genConfigName = [](auto conf) -> std::string
    {
        using namespace tensorrt_llm::cutlass_extensions;
        std::stringstream tactic;
        tactic << (conf.is_sm90 ? "SM90+" : "<SM90") << " tactic with tile shape ";
        if (conf.tile_config_sm90 != CutlassTileConfigSM90::ChooseWithHeuristic)
        {
            tactic << (int) conf.tile_config_sm90 << " and cluster shape " << (int) conf.cluster_shape
                   << " mainloop sched " << (int) conf.mainloop_schedule << " epi sched "
                   << (int) conf.epilogue_schedule;
        }
        else if (conf.tile_config != CutlassTileConfig::ChooseWithHeuristic)
        {
            tactic << (int) conf.tile_config << " and stages " << (int) conf.stages << " split k "
                   << (int) conf.split_k_factor;
        }
        else
        {
            return {};
        }
        return tactic.str();
    };

    auto const actiavtion_pool = {
        tensorrt_llm::ActivationType::Relu, tensorrt_llm::ActivationType::Swiglu, tensorrt_llm::ActivationType::Geglu};
    auto configs = this->getFilteredConfigs(getSMVersion());
    for (auto const activation_type : actiavtion_pool)
    {
        for (auto conf1 : configs)
        {
            for (auto conf2 : configs)
            {
                auto name1 = genConfigName(conf1);
                auto name2 = genConfigName(conf2);
                if (name1.empty() || name2.empty())
                {
                    FAIL() << "Uninitialised tactic encountered";
                }
                ASSERT_NO_THROW({
                    this->mActType = activation_type;
                    for (int k = 1; k <= 3; k++)
                    {
                        this->mOverrideSelectedConfig1 = conf1;
                        this->mOverrideSelectedConfig2 = conf2;
                        this->BasicPermuteTest(k);
                    }
                    if (::testing::Test::HasFailure()) // Throw on test failure so we get the print message
                        throw std::runtime_error("Test Failed");
                }) << "Failed\nTactic 1: "
                   << name1 << "\nTactic 2: " << name2 << " and activation type: " << static_cast<int>(activation_type);
            }
        }
    }
}

TYPED_TEST(LargeMixtureOfExpertsTest, PermuteVeryLargeExperts)
{
    // Chosen so that hidden_size * inter_size * num_experts >> 2^32, but we can still fit in 80GB for `half`
    // Uses a non-power of two so any integer overflow will have bad alignment
    int64_t hidden_size = 31 * 1024;
    ASSERT_GT(hidden_size * hidden_size * 4, (int64_t) std::numeric_limits<int>::max() + 1ull);

    int64_t k = 2; // Use k=2 so all experts get a value
    // 3 tokens 4 experts are the defaults for BasicPermuteTest
    if (!this->checkSufficientTestMemory(3, hidden_size, 4, k))
    {
        GTEST_SKIP() << "Insufficient free memory for test";
    }

    this->BasicPermuteTest(k, hidden_size); // 4 x 32k x 128K experts
}

TYPED_TEST(LargeMixtureOfExpertsTest, PermuteVeryLongSequence)
{
    this->mUseBias = !this->FP8;

    using DataType = typename TypeParam::DataType;
    // Sequence * hidden size > INT32_MAX
    int64_t hidden_size = 2048ll;
    int64_t num_experts = 4;
    int64_t k = 1;
    int64_t tokens_to_test = 100;
    int64_t num_tokens = 2ull * 1024ll * 1024ll + tokens_to_test + 1ll;
    ASSERT_GT(hidden_size * (num_tokens - tokens_to_test), (uint64_t) std::numeric_limits<uint32_t>::max() + 1ull);

    if (!this->checkSufficientTestMemory(num_tokens, hidden_size, num_experts, k))
    {
        GTEST_SKIP() << "Insufficient free memory for test";
    }

    std::vector<DataType> hidden_states(hidden_size * num_tokens);
    this->mMaxInput = 1.f; // Any arbitrary non-zero value

    // All tokens to expert 0, so we catch the case where an expert has more than 2^32 tokens
    float const token_probs[] = {1.f, 0.5f, 0.f, 0.f};
    std::vector<float> probs;
    probs.reserve(num_tokens * num_experts);
    for (size_t i = 0; i < num_tokens; i++)
    {
        probs.insert(probs.cend(), std::begin(token_probs), std::end(token_probs));
    }
    // Override the first few tokens to go to different experts.
    // This covers the regression case where an overflow only impacts one of the last experts
    // In particular the case when there are more than 2^32 elements before the last expert
    for (int i = 1; i < tokens_to_test; i++)
    {
        probs[i * num_experts + i % num_experts] = 2.f;
    }

    this->runMoEPermute({hidden_states}, {probs}, hidden_size, num_experts, k);

    // Just look at the first few tokens
    this->mTotalTokens = tokens_to_test;

    probs.resize(num_experts * this->mTotalTokens);
    hidden_states.resize(hidden_size * this->mTotalTokens);

    auto selected_expert = this->getDataFromDevice(this->mSelectedExpert, k * this->mTotalTokens);
    // We set the first few tokens to go to the corresponding i-th expert
    for (int i = 0; i < tokens_to_test; i++)
    {
        ASSERT_EQ(selected_expert[i], i % num_experts);
    }

    this->compareSoftmax(selected_expert, probs);
    // Create a default vector for the reference outputs of the correct type for FP8
    std::vector<typename TypeParam::OutputType> unquant_states(this->mTotalTokens * hidden_size);
    this->compareFinal(selected_expert, probs, unquant_states);
}

using MixtureOfExpertsProfilerTest = MixtureOfExpertsTest<WeightParams<half, half>>;

TEST_F(MixtureOfExpertsProfilerTest, TestGeneratedProfilerDistribution)
{
    //    int64_t num_tokens = 128;
    int64_t num_experts = 8;
    int64_t k = 2;

    GemmProfilerBackend backend;

    // We need to test different EP values to ensure the tokens are properly assigned
    for (int64_t num_tokens : {1, 128})
    {
        int64_t expanded_num_tokens = num_tokens * k;
        for (int ep : {1, 4, 8})
        {
            backend.init(this->mMoERunner, GemmProfilerBackend::GemmToProfile::GEMM_1, nvinfer1::DataType::kHALF,
                nvinfer1::DataType::kHALF, nvinfer1::DataType::kHALF, num_experts, k, 1024, 4096, {}, false, mUseLora,
                MOEParallelismConfig{1, 0, ep, ep - 1});

            auto ws_size = backend.getWorkspaceSize(num_tokens);
            auto workspace = this->allocBuffer<char>(ws_size);

            int64_t num_experts_per_node = num_experts / ep;

            backend.prepare(num_tokens, workspace, mStream->get());

            auto getNext = backend.getWorkspacePointerGenerator(workspace, num_tokens, getSMVersion() >= 90);
            auto const* expert_first_token_offset_size = reinterpret_cast<int64_t*>(getNext());
            auto const* source_to_dest_map = reinterpret_cast<int*>(getNext());
            auto const* dest_to_source_map = reinterpret_cast<int*>(getNext());
            auto const* token_selected_experts = reinterpret_cast<int*>(getNext());

            for (int sample = 0; sample < backend.NUM_ROUTING_SAMPLES; sample++)
            {
                auto host_expert_first_token_offset_size = getDataFromDevice(
                    expert_first_token_offset_size + sample * (num_experts_per_node + 1), num_experts_per_node + 1);
                auto host_source_to_dest_map
                    = getDataFromDevice(source_to_dest_map + sample * expanded_num_tokens, expanded_num_tokens);
                auto host_dest_to_source_map
                    = getDataFromDevice(dest_to_source_map + sample * expanded_num_tokens, expanded_num_tokens);
                auto host_token_selected_experts
                    = getDataFromDevice(token_selected_experts + sample * expanded_num_tokens, expanded_num_tokens);

                std::vector<int64_t> calculated_routing_values(num_experts_per_node + 1, 0);
                int skipped = 0;
                for (auto v : host_token_selected_experts)
                {
                    ASSERT_TRUE(v < num_experts_per_node || (v == num_experts && ep > 1));
                    skipped += (v == num_experts);
                    if (v < num_experts_per_node)
                    {
                        calculated_routing_values[v]++;
                    }
                }

                if (num_tokens > 1)
                {
                    // Check tokens are distributed between all EP ranks
                    // Statistically possible, but so unlikely that it should be considered a bug
                    ASSERT_TRUE(ep == 1 || skipped > 0);
                    // Check all experts get some tokens
                    ASSERT_EQ(std::count(calculated_routing_values.begin(), calculated_routing_values.end() - 1, 0), 0);

                    float p = 1.f / num_experts;
                    float variance = expanded_num_tokens * p * (1 - p);
                    float stddev = sqrt(variance);
                    float mean = expanded_num_tokens * p;
                    for (int i = 0; i < num_experts_per_node; i++)
                    {
                        // All values should be within three standard deviations of the mean
                        // 99.7% of values should fall within this range.
                        // We have NUM_ROUTING_SAMPLES * (8 + 2 + 1) = 176 cases so this is unlikely
                        // If the test changes to have a much larger number of cases this will need revisited
                        EXPECT_LE(abs(calculated_routing_values[i] - mean), 3 * stddev)
                            << "Expert " << i << " for sample " << sample << " has unbalanced token count "
                            << calculated_routing_values[i] << " vs mean value " << mean << " with standard deviation "
                            << stddev;
                    }
                }
                ASSERT_EQ(host_expert_first_token_offset_size.back(), expanded_num_tokens - skipped);

                std::exclusive_scan(calculated_routing_values.begin(), calculated_routing_values.end(),
                    calculated_routing_values.begin(), 0);
                ASSERT_TRUE(std::equal(calculated_routing_values.begin(), calculated_routing_values.end(),
                    host_expert_first_token_offset_size.begin()));

                std::fill(calculated_routing_values.begin(), calculated_routing_values.end(), 0);
                for (int64_t token_idx = 0; token_idx < num_tokens; token_idx++)
                {
                    for (int64_t k_idx = 0; k_idx < k; k_idx++)
                    {
                        int64_t idx = token_idx * k + k_idx;
                        int64_t expert_idx = host_token_selected_experts[idx];

                        if (expert_idx < num_experts)
                        {
                            int64_t source_location = k_idx * num_tokens + token_idx;
                            int64_t dest_location = host_expert_first_token_offset_size[expert_idx]
                                + calculated_routing_values[expert_idx];

                            ASSERT_EQ(host_source_to_dest_map[source_location], dest_location);
                            ASSERT_EQ(host_dest_to_source_map[dest_location], source_location);

                            calculated_routing_values[expert_idx]++;
                        }
                    }
                }
            }
        }
    }
}

using MixtureOfExpertsUnitTests = MixtureOfExpertsTest<WeightParams<half, half>>;

TEST_F(MixtureOfExpertsUnitTests, SparseMixerReferenceTest)
{
    // Test the sparse mixer reference implementation is doing the correct thing
    // This makes sure we are testing the correct behaviour
    this->mNumExperts = 4;
    this->mK = 2;
    auto res = this->sparseMixer({1.0f, 1.0f, -INFINITY, -INFINITY}, 0, 0, 0);
    ASSERT_FLOAT_EQ(res, 0.5f);
    res = this->sparseMixer({1.0f, 1.0f, -INFINITY, -INFINITY}, 0, 1, 1);
    ASSERT_FLOAT_EQ(res, 1.0f);

    res = this->sparseMixer({2.0f, 0.0f, -INFINITY, -INFINITY}, 0, 0, 0);
    ASSERT_FLOAT_EQ(res, 1.0f);
    res = this->sparseMixer({2.0f, 0.0f, -INFINITY, -INFINITY}, 0, 1, 1);
    ASSERT_FLOAT_EQ(res, 1.0f);

    res = this->sparseMixer({0.0f, 2.0f, -INFINITY, -INFINITY}, 0, 0, 1);
    ASSERT_FLOAT_EQ(res, 1.0f);
    res = this->sparseMixer({0.0f, 2.0f, -INFINITY, -INFINITY}, 0, 1, 0);
    ASSERT_FLOAT_EQ(res, 1.0f);

    res = this->sparseMixer({1.0f, 1.0f, 1.0f, -INFINITY}, 0, 0, 0);
    ASSERT_FLOAT_EQ(res, 1.f / 3.f);
    res = this->sparseMixer({1.0f, 1.0f, 1.0f, -INFINITY}, 0, 1, 1);
    ASSERT_FLOAT_EQ(res, 0.5f);
}
